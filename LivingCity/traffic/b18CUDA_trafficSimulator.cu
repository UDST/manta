#include "hip/hip_runtime.h"
//CUDA CODE
#include <assert.h>
#include <stdio.h>
#include <vector>
#include <iostream>

#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""

#include "b18TrafficPerson.h"
#include "b18EdgeData.h"

#ifndef ushort
#define ushort uint16_t
#endif
#ifndef uint
#define uint uint32_t
#endif
#ifndef uchar
#define uchar uint8_t
#endif

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}
inline void printMemoryUsage() {
  // show memory usage of GPU
  size_t free_byte;
  size_t total_byte;
  hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);
  if (hipSuccess != cuda_status) {
    printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status));
    exit(1);
  }
  double free_db = (double) free_byte;
  double total_db = (double) total_byte;
  double used_db = total_db - free_db;
  printf("GPU memory usage: used = %.0f, free = %.0f MB, total = %.0f MB\n", used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}


/* Constants */

__constant__ float intersectionClearance = 7.8f;


// `s_0` refers to the minimum spacing distance used in the Intelligent Driver Model (IDM)
__constant__ float s_0 = 7.0f;


// Distance from which stopping before an intersection will be considered as having stopped for the
// intersection itself.
// This value should take into account the intersection cleareance (and be greater than it).
__constant__ float relevantStopDistance = 10.0f;


__constant__ bool calculatePollution = true;

// Half a second happens between each iteration of the simulation
__constant__ float DELTA_TIME = 0.5f;

const float DELTA_TIME_HOST = 0.5f;
const uint numStepsPerSample = 30.0f / DELTA_TIME_HOST; //each min
const uint numStepsTogether = 12; //change also in density (10 per hour)


/* Variables */

bool readFirstMapC=true;
uint mapToReadShift;
uint mapToWriteShift;
uint halfLaneMap;
float startTime;

LC::B18TrafficPerson *trafficPersonVec_d;
uint *indexPathVec_d;
LC::B18EdgeData *edgesData_d;
uchar *laneMap_d;
LC::B18IntersectionData *intersections_d;
uchar *trafficLights_d;
float* accSpeedPerLinePerTimeInterval_d;
float* numVehPerLinePerTimeInterval_d;

uint* deviceInLanesIndexes;
LC::Connection *deviceConnections;
size_t amountOfConnections;
uint *deviceConnectionsBlocking;

LC::Intersection *deviceIntersections;
size_t amountOfIntersections;

LC::TrafficLightScheduleEntry *deviceTrafficLightSchedules;

void b18InitCUDA(
    bool fistInitialization,
    std::vector<LC::B18TrafficPerson>& trafficPersonVec,
    std::vector<uint> &indexPathVec,
    std::vector<LC::B18EdgeData>& edgesData,
    std::vector<uchar>& laneMap,
    std::vector<uchar>& trafficLights,
    std::vector<LC::B18IntersectionData>& b18Intersections,
    float startTimeH,
    float endTimeH,
    std::vector<float>& accSpeedPerLinePerTimeInterval,
    std::vector<float>& numVehPerLinePerTimeInterval,
    const std::vector<LC::Connection> & hostConnections,
    const std::vector<uint> & hostConnectionsBlocking,
    const std::vector<LC::Intersection> & hostIntersections,
    const std::vector<LC::TrafficLightScheduleEntry> & hostTrafficLightSchedules,
    const std::vector<uint> & hostInLanesIndexes) {

  { // In lanes indexes
    size_t size = hostInLanesIndexes.size() * sizeof(uint);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &deviceInLanesIndexes, size));   // Allocate array on device
    gpuErrchk(hipMemcpy(deviceInLanesIndexes, hostInLanesIndexes.data(), size, hipMemcpyHostToDevice));
  }

  { // Connections
    amountOfConnections = hostConnections.size();
    size_t size = hostConnections.size() * sizeof(LC::Connection);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &deviceConnections, size));   // Allocate array on device
    gpuErrchk(hipMemcpy(deviceConnections, hostConnections.data(), size, hipMemcpyHostToDevice));
  }

  { // Connections blocking
    size_t size = hostConnectionsBlocking.size() * sizeof(uint);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &deviceConnectionsBlocking, size));   // Allocate array on device
    gpuErrchk(hipMemcpy(deviceConnectionsBlocking, hostConnectionsBlocking.data(), size, hipMemcpyHostToDevice));
  }

  { // Intersections
    amountOfIntersections = hostIntersections.size();
    size_t size = hostIntersections.size() * sizeof(LC::Intersection);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &deviceIntersections, size));   // Allocate array on device
    gpuErrchk(hipMemcpy(deviceIntersections, hostIntersections.data(), size, hipMemcpyHostToDevice));
  }

  { // Traffic light schedules
    size_t size = hostTrafficLightSchedules.size() * sizeof(LC::TrafficLightScheduleEntry);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &deviceTrafficLightSchedules, size));   // Allocate array on device
    gpuErrchk(hipMemcpy(deviceTrafficLightSchedules, hostTrafficLightSchedules.data(), size, hipMemcpyHostToDevice));
  }

  { // people
    size_t size = trafficPersonVec.size() * sizeof(LC::B18TrafficPerson);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &trafficPersonVec_d, size));   // Allocate array on device
    gpuErrchk(hipMemcpy(trafficPersonVec_d, trafficPersonVec.data(), size, hipMemcpyHostToDevice));
  }

  { // indexPathVec
    size_t sizeIn = indexPathVec.size() * sizeof(uint);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &indexPathVec_d, sizeIn));   // Allocate array on device
    gpuErrchk(hipMemcpy(indexPathVec_d, indexPathVec.data(), sizeIn, hipMemcpyHostToDevice));
  }
  {//edgeData
    size_t sizeD = edgesData.size() * sizeof(LC::B18EdgeData);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &edgesData_d, sizeD));   // Allocate array on device
    gpuErrchk(hipMemcpy(edgesData_d, edgesData.data(), sizeD, hipMemcpyHostToDevice));
  }
  {//laneMap
    size_t sizeL = laneMap.size() * sizeof(uchar);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &laneMap_d, sizeL));   // Allocate array on device
    gpuErrchk(hipMemcpy(laneMap_d, laneMap.data(), sizeL, hipMemcpyHostToDevice));
    halfLaneMap = laneMap.size() / 2;
  }
  {// b18Intersections
    size_t sizeI = b18Intersections.size() * sizeof(LC::B18IntersectionData);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &intersections_d, sizeI));   // Allocate array on device
    gpuErrchk(hipMemcpy(intersections_d, b18Intersections.data(), sizeI, hipMemcpyHostToDevice));
    size_t sizeT = trafficLights.size() * sizeof(uchar);//total number of lanes
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &trafficLights_d, sizeT));   // Allocate array on device
    gpuErrchk(hipMemcpy(trafficLights_d, trafficLights.data(), sizeT, hipMemcpyHostToDevice));
  }
  {
    startTime = startTimeH * 3600.0f;
    uint numSamples = ceil(((endTimeH*3600.0f - startTimeH*3600.0f) / (DELTA_TIME_HOST * numStepsPerSample * numStepsTogether))) + 1; //!!!
    accSpeedPerLinePerTimeInterval.clear();
    numVehPerLinePerTimeInterval.clear();
    accSpeedPerLinePerTimeInterval.resize(numSamples * trafficLights.size());
    numVehPerLinePerTimeInterval.resize(numSamples * trafficLights.size());
    size_t sizeAcc = accSpeedPerLinePerTimeInterval.size() * sizeof(float);
    if (fistInitialization)gpuErrchk(hipMalloc((void **) &accSpeedPerLinePerTimeInterval_d, sizeAcc));   // Allocate array on device
    if (fistInitialization)gpuErrchk(hipMalloc((void **) &numVehPerLinePerTimeInterval_d, sizeAcc));   // Allocate array on device
    gpuErrchk(hipMemset(&accSpeedPerLinePerTimeInterval_d[0], 0, sizeAcc));
    gpuErrchk(hipMemset(&numVehPerLinePerTimeInterval_d[0], 0, sizeAcc));
  }
}//

void b18FinishCUDA(void){
  hipFree(deviceConnections);
  hipFree(deviceInLanesIndexes);
  hipFree(deviceConnectionsBlocking);
  hipFree(deviceIntersections);
  hipFree(deviceTrafficLightSchedules);
  hipFree(trafficPersonVec_d);
  hipFree(indexPathVec_d);
  hipFree(edgesData_d);
  hipFree(laneMap_d);
  hipFree(intersections_d);
  hipFree(trafficLights_d);
  hipFree(accSpeedPerLinePerTimeInterval_d);
  hipFree(numVehPerLinePerTimeInterval_d);
}//

void b18GetDataCUDA(std::vector<LC::B18TrafficPerson>& trafficPersonVec) {
  // copy back people
  size_t size = trafficPersonVec.size() * sizeof(LC::B18TrafficPerson);
  hipMemcpy(trafficPersonVec.data(),trafficPersonVec_d,size,hipMemcpyDeviceToHost);//hipMemcpyHostToDevice
}


__device__ void calculateGapsLC(
    uint mapToReadShift,
    uchar* laneMap,
    /*uchar trafficLightState,*/
    bool intersectionEnabled,
    uint laneToCheck,
    ushort numLinesEdge,
    float posInMToCheck,
    float length,
    uchar &v_a,
    uchar &v_b,
    float &gap_a,
    float &gap_b) {
  ushort currentLaneMaximumPosition = ceil(length);
  ushort initShift = ceil(posInMToCheck);
  bool found = false;

  // CHECK FORWARD
  for (ushort b = initShift - 1; (b < currentLaneMaximumPosition) && (found == false); b++) { //NOTE -1 to make sure there is none in at the same level
    const uint posToSample = mapToReadShift + kMaxMapWidthM * (laneToCheck + (((int) (b / kMaxMapWidthM)) * numLinesEdge)) + b % kMaxMapWidthM;
    const uchar laneChar = laneMap[posToSample];

    if (laneChar != 0xFF) {
      gap_a = ((float) b - initShift); //m
      v_a = laneChar; //laneChar is in 3*ms (to save space in array)
      found = true;
      break;
    }
  }

  if (found == false) {
    if (!intersectionEnabled) {
      gap_a = gap_b = 1000.0f; //force to change to the line without vehicle
      v_a = v_b = 0xFF;
      return;
    }
  }

  if (found == false) {
    gap_a = 1000.0f;
  }

  // CHECK BACKWARDS
  found = false;

  for (int b = initShift + 1; (b >= 0) && (found == false); b--) {  // NOTE +1 to make sure there is none in at the same level
    const uint posToSample = mapToReadShift + kMaxMapWidthM * (laneToCheck + (((int) (b / kMaxMapWidthM)) * numLinesEdge)) + b % kMaxMapWidthM;
    const uchar laneChar = laneMap[posToSample];
    if (laneChar != 0xFF) {
      gap_b = ((float) initShift - b); //m
      v_b = laneChar; //laneChar is in 3*ms (to save space in array)
      found = true;
      break;
    }
  }

  if (found == false) {
    gap_b = 1000.0f;
  }
}

__device__ void calculateLaneCarShouldBe(
    uint curEdgeLane,
    uint nextEdge,
    const LC::B18IntersectionData* b18Intersections,
    uint edgeNextInters,
    ushort edgeNumLanes,
    ushort &initOKLanes,
    ushort &endOKLanes) {
  initOKLanes = 0;
  endOKLanes = edgeNumLanes;
  bool currentEdgeFound = false;
  bool exitFound = false;
  ushort numExitToTake = 0;
  ushort numExists = 0;

  for (int eN = b18Intersections[edgeNextInters].totalInOutEdges - 1; eN >= 0; eN--) {  // clockwise
    // retrieve
    uint procEdge = b18Intersections[edgeNextInters].edge[eN];

    if ((procEdge & kMaskLaneMap) == curEdgeLane) { //current edge 0xFFFFF
      currentEdgeFound = true;
      if (exitFound == false) {
        numExitToTake = 0;
      }
      continue;
    }

    if ((procEdge & kMaskInEdge) == 0x0) { //out edge 0x800000
      numExists++;
      if (currentEdgeFound == true) {
        numExitToTake++;
      }
      if (currentEdgeFound == false && exitFound == false) {
        numExitToTake++;
      }
    }
    if ((procEdge & kMaskInEdge) == nextEdge) {
      exitFound = true;
      currentEdgeFound = false;
    }
  }

  if (edgeNumLanes == 0) {
    printf("ERRRROR\n");
  }

  switch (edgeNumLanes) {
    /// ONE LANE
  case 1:
    initOKLanes = 0;
    endOKLanes = 1;
    break;

    /// TWO LANE
  case 2:
    switch (numExists) {
    case 1:
    case 2://all okay
      initOKLanes = 0;
      endOKLanes = 2;
      break;

    case 3:
      if (numExitToTake > 2) { //left
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 2;
      break;

    default:

      if (numExitToTake >= numExists - 1) {
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 2;
      break;
    }

    break;

    /// THREE LANE
  case 3:
    switch (numExists) {
    case 1:
    case 2://all okay
      initOKLanes = 0;
      endOKLanes = 3;
      break;

    case 3:
      if (numExitToTake > 2) { //left
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 3;
      break;

    default:
      if (numExitToTake >= numExists - 1) {
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 2;
      break;
    }

    break;

  case 4:
    switch (numExists) {
    case 1:
    case 2://all okay
      initOKLanes = 0;
      endOKLanes = 4;
      break;

    case 3:
      if (numExitToTake == 1) { //right
        initOKLanes = 3;
        endOKLanes = 4;
      }

      if (numExitToTake > 3) { //left
        initOKLanes = 0;
        endOKLanes = 1;
        break;
      }

      initOKLanes = 1;
      endOKLanes = 4;
      break;

    default:
      if (numExitToTake == 1) { //right
        initOKLanes = edgeNumLanes - 1;
        endOKLanes = edgeNumLanes;
      }

      if (numExitToTake >= numExists - 2) {
        initOKLanes = 0;
        endOKLanes = 2;
        break;
      }

      initOKLanes = 1; //also lane 2
      endOKLanes = edgeNumLanes;
    }

    break;

  default:
    switch (numExists) {
    case 1:
    case 2://all okay
      initOKLanes = 0;
      endOKLanes = edgeNumLanes;
      break;

    case 3:
      if (numExitToTake == 1) { //right
        initOKLanes = edgeNumLanes - 1;
        endOKLanes = edgeNumLanes;
      }

      if (numExitToTake > edgeNumLanes - 2) { //left
        initOKLanes = 0;
        endOKLanes = 2;
        break;
      }

      initOKLanes = 1;
      endOKLanes = edgeNumLanes;
      break;

    default:
      if (numExitToTake < 2) { //right
        initOKLanes = edgeNumLanes - 2;
        endOKLanes = edgeNumLanes;
      }

      if (numExitToTake >= numExists - 2) {
        initOKLanes = 0;
        endOKLanes = 2;
        break;
      }

      initOKLanes = 1; //also lane 2
      endOKLanes = edgeNumLanes - 1;
    }

    break;
  }
}

__global__ void kernel_updatePersonsCars(
    const int numPeople,
    float currentTime,
    uint mapToReadShift,
    uint mapToWriteShift,
    uint halfLaneMap,
    LC::B18TrafficPerson *trafficPersonVec,
    uint *indexPathVec,
    LC::B18EdgeData* edgesData,
    uchar *laneMap,
    LC::B18IntersectionData *b18Intersections,
    LC::Connection *connections,
    size_t amountOfConnections,
    uint *connectionsBlocking,
    LC::Intersection *intersections,
    size_t amountOfIntersections,
    LC::TrafficLightScheduleEntry *trafficLightSchedules) {
  const int p = blockIdx.x * blockDim.x + threadIdx.x;
  // Only proceed if the computed index `p` is valid
  if (p < numPeople) {
    /**
     * First ensure this person's car's info is initialized and whether is it active or not
     */
    if (trafficPersonVec[p].active == 2) {
      // Return if this person has reached its destiny
      return;
    }

    if (trafficPersonVec[p].active == 0){
      if (trafficPersonVec[p].time_departure > currentTime) {
        // Return if it's not yet the time for this person
        return;
      }

      const uint firstEdge = indexPathVec[trafficPersonVec[p].indexPathInit];
      if (firstEdge == -1) {
        // Return if this person's path has length zero
        trafficPersonVec[p].active = 2;
        return;
      }

      // Else initialize this person's data
      trafficPersonVec[p].indexPathCurr = trafficPersonVec[p].indexPathInit;
      trafficPersonVec[p].edgeNumLanes = edgesData[firstEdge].numLines;
      trafficPersonVec[p].edgeNextInters = edgesData[firstEdge].nextInters;
      trafficPersonVec[p].length = edgesData[firstEdge].length;
      trafficPersonVec[p].maxSpeedMperSec = edgesData[firstEdge].maxSpeedMperSec;

      // Find the starting position of the current person
      // At least `requiredAmountOfEmptyCells` are needed before the position where the car will be
      // placed
      const ushort requiredAmountOfEmptyCells = s_0;
      const ushort startingRoadAmountOfCells = ceil(trafficPersonVec[p].length);
      // We will start to search from the middle of the starting road
      const ushort initShift = static_cast<ushort>(0.5f * startingRoadAmountOfCells);
      bool placed = false;
      ushort amountOfEmptyCells = 0;
      for (
          ushort position = initShift;
          position < startingRoadAmountOfCells && !placed;
          position++) {
        const ushort numberOfRightLane = trafficPersonVec[p].edgeNumLanes - 1;
        size_t posToSample = mapToReadShift + kMaxMapWidthM * (firstEdge + numberOfRightLane) + position;
        const uchar laneChar = laneMap[posToSample];
        if (laneChar != 0xFF) {
          // If the cell is not empty reset the empty-cells counter
          amountOfEmptyCells = 0;
          continue;
        }

        // Keep advancing until enough empty cells have been found
        amountOfEmptyCells++;
        if (amountOfEmptyCells < requiredAmountOfEmptyCells) { continue; }

        // If we get to this point we can place the car
        trafficPersonVec[p].numOfLaneInEdge = numberOfRightLane;
        trafficPersonVec[p].posInLaneM = position;
        const uchar vInMpS = static_cast<uchar>(trafficPersonVec[p].v * 3);
        posToSample = mapToWriteShift + kMaxMapWidthM * (firstEdge + numberOfRightLane) + position;
        laneMap[posToSample] = vInMpS;
        placed = true;
        break;
      }

      if (!placed) {
        // Return if the current road is too busy
        return;
      }

      trafficPersonVec[p].v = 0;
      trafficPersonVec[p].LC_stateofLaneChanging = 0;
      trafficPersonVec[p].active = 1;
      trafficPersonVec[p].num_steps = 1;
      trafficPersonVec[p].co = 0.0f;
      trafficPersonVec[p].gas = 0.0f;
      trafficPersonVec[p].isApproachingStopJunction = false;
      trafficPersonVec[p].distanceUntilIntersection = INT_MAX >> 2;
      trafficPersonVec[p].stoppedBeforeNextIntersection = false;

      const uint nextEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];
      if (nextEdge != -1) {
        trafficPersonVec[p].nextEdgemaxSpeedMperSec = edgesData[nextEdge].maxSpeedMperSec;
        trafficPersonVec[p].nextEdgeNumLanes = edgesData[nextEdge].numLines;
        trafficPersonVec[p].nextEdgeNextInters = edgesData[nextEdge].nextInters;
        trafficPersonVec[p].nextEdgeLength = edgesData[nextEdge].length;
        trafficPersonVec[p].LC_initOKLanes = 0xFF;
        trafficPersonVec[p].LC_endOKLanes = 0xFF;
      }
      return;
    }

    // At this point we can assume the current person is already active
    if (float(currentTime) == int(currentTime)) { // assuming deltatime = 0.5f --> each second
      trafficPersonVec[p].num_steps++;
    }

    /**
     * Gather enough information to know how the current car should be updated, using the
     * Intelligent Driver Model (IDM).
     */
    const uint currentEdge = indexPathVec[trafficPersonVec[p].indexPathCurr];
    const uint nextEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];
    const ushort currentPositionInLane = static_cast<ushort>(floor(trafficPersonVec[p].posInLaneM));
    const ushort currentLaneMaximumPosition = ceil(trafficPersonVec[p].length - intersectionClearance);

    printf(
      "{currentTime: %.2f, id: %d, speed: %5.2f, edgeId: %d, currentPositionInLane: %02d, maximumPositionInLane: %02d}\n",
      currentTime,
      p,
      trafficPersonVec[p].v,
      currentEdge,
      currentPositionInLane,
      currentLaneMaximumPosition);


    bool nextVehicleIsATrafficLight = false;
    int remainingCellsToCheck = max(30.0f, trafficPersonVec[p].v * DELTA_TIME * 2);

    bool obstacleFound = false;
    float distanceUntilObstacle = static_cast<float>(remainingCellsToCheck);
    float speedDifferenceWithNextObstacle = 0;

    // Check if there is another car in the same lane
    for (
        ushort b = currentPositionInLane + 2;
        b < currentLaneMaximumPosition && !obstacleFound && remainingCellsToCheck > 0;
        b++, remainingCellsToCheck--) {
      const uint posToSample =
         mapToReadShift
         + kMaxMapWidthM * (
           currentEdge
           + static_cast<int>(currentPositionInLane / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
           + trafficPersonVec[p].numOfLaneInEdge)
         + b % kMaxMapWidthM;
      const uchar laneChar = laneMap[posToSample];

      if (laneChar != 0xFF) {
        distanceUntilObstacle = static_cast<float>(b - currentPositionInLane);
        speedDifferenceWithNextObstacle = trafficPersonVec[p].v - (laneChar / 3.0f);  //laneChar is in 3*ms (to save space in array)
        obstacleFound = true;
        break;
      }
    }

    // At this point we found an obstacle or we reached the end of the current edge
    // If we are at the end of the current edge, check if this car's lane's connections are enabled
    bool atLeastOneEnabledConnection = false;
    int nextEdgeChosenLane = -1;
    if (
        currentPositionInLane < currentLaneMaximumPosition
        && !obstacleFound
        && remainingCellsToCheck > 0
        && nextEdge != -1) {
      const int dstVertexNumber = edgesData[currentEdge].targetVertexIndex;
      const ushort currentLaneNumber = currentEdge + trafficPersonVec[p].numOfLaneInEdge;

      const int distanceUntilIntersection = currentLaneMaximumPosition - currentPositionInLane;
      trafficPersonVec[p].isApproachingStopJunction = intersections[dstVertexNumber].isStopIntersection;
      trafficPersonVec[p].distanceUntilIntersection = distanceUntilIntersection;

      // If the car is approaching a stop intersection and it has not yet stopped then the
      // intersection must be treated as an obstacle
      bool mustTreatIntersectionAsObstacle =
        trafficPersonVec[p].isApproachingStopJunction
        && !trafficPersonVec[p].stoppedBeforeNextIntersection;

      if (!mustTreatIntersectionAsObstacle) {
        // Check if a least one connection is enabled between the current edge and the following one
        for (
            int connectionIdx = intersections[dstVertexNumber].connectionGraphStart;
            connectionIdx < intersections[dstVertexNumber].connectionGraphEnd;
            ++connectionIdx) {
          const LC::Connection & connection = connections[connectionIdx];
          const bool isRelevant =
            connection.inLaneNumber == currentLaneNumber
            && connection.outEdgeNumber == nextEdge;
          if (!isRelevant) continue;

          if (connection.enabled) {
            atLeastOneEnabledConnection = true;
            nextEdgeChosenLane = connection.outLaneNumber - connection.outEdgeNumber;
            break;
          }
        }
      }

      // If no connection to the needed edge is enabled, then that intersection will be treated as
      // an obstacle
      mustTreatIntersectionAsObstacle |= !atLeastOneEnabledConnection;

      if (mustTreatIntersectionAsObstacle) {
        distanceUntilObstacle = (float) distanceUntilIntersection;
        speedDifferenceWithNextObstacle = trafficPersonVec[p].v - 0;
        nextVehicleIsATrafficLight = true;
        obstacleFound = true;
      }
    }

    // If we still need it, check if there is an obstacle in next edge's chosen lane
    const ushort chosenLaneMaximumPosition =
      static_cast<ushort>(ceil(trafficPersonVec[p].nextEdgeLength - intersectionClearance));
    for (
        ushort b = 0;
        atLeastOneEnabledConnection
          && b < chosenLaneMaximumPosition
          && !obstacleFound
          && remainingCellsToCheck > 0;
        b++, remainingCellsToCheck--) {
      const uint posToSample =
        mapToReadShift
        + kMaxMapWidthM * (
          nextEdge
          + static_cast<int>(b / kMaxMapWidthM) * trafficPersonVec[p].nextEdgeNumLanes
          + nextEdgeChosenLane)
        + b % kMaxMapWidthM;
      const uchar laneChar = laneMap[posToSample];

      if (laneChar != 0xFF) {
        obstacleFound = true;
        distanceUntilObstacle =
          static_cast<float>(b + currentLaneMaximumPosition - currentPositionInLane); //m
        speedDifferenceWithNextObstacle =
          trafficPersonVec[p].v - (laneChar / 3.0f); //laneChar is in 3*ms (to save space in array)
        break;
      }
    }

    /**
     * Update car's information
     */
    float thirdTerm = 0;
    if (obstacleFound) {
      const float s_star = s_0 + max(
        0.0f,
        trafficPersonVec[p].v * trafficPersonVec[p].T + (trafficPersonVec[p].v * speedDifferenceWithNextObstacle) / (2 * sqrtf(trafficPersonVec[p].a * trafficPersonVec[p].b))
      );
      thirdTerm = powf(s_star / distanceUntilObstacle, 2);
    }
    float dv_dt = trafficPersonVec[p].a * (1.0f - std::pow((trafficPersonVec[p].v / trafficPersonVec[p].maxSpeedMperSec), 4) - thirdTerm);

    float numToMove = max(0.0f, trafficPersonVec[p].v * DELTA_TIME + 0.5f * (dv_dt) * DELTA_TIME * DELTA_TIME);
    trafficPersonVec[p].v += dv_dt * DELTA_TIME;
    if (trafficPersonVec[p].v < 0) {
      trafficPersonVec[p].v = 0;
      dv_dt = 0.0f;
    }

    trafficPersonVec[p].stoppedBeforeNextIntersection |=
      trafficPersonVec[p].distanceUntilIntersection < relevantStopDistance
      && trafficPersonVec[p].v < 0.0001;

    trafficPersonVec[p].cumulative_velocity += trafficPersonVec[p].v;

    // Note: compute CO and Gas values each second
    if (calculatePollution && ((float(currentTime) == int(currentTime)))) { // enabled and each second (assuming DELTA_TIME 0.5f)

      // CO Calculation
      const float speedMph = trafficPersonVec[p].v * 2.2369362920544; //mps to mph
      const float COStepPerSecond = -0.064 + 0.0056 * speedMph + 0.00026 * (speedMph - 50.0f) * (speedMph - 50.0f);
      if (COStepPerSecond > 0) { trafficPersonVec[p].co += COStepPerSecond; }

      // Gas Consumption
      const float a = dv_dt;
      const float v = trafficPersonVec[p].v; // in mps
      const float Pea = a > 0.0f ? (0.472f*1.680f*a*a*v) : 0.0f;
      const float gasStepPerSecond = 0.666f + 0.072f*(0.269f*v + 0.000672f*(v*v*v) + 0.0171f*(v*v) + 1.680f*a*v + Pea);
      trafficPersonVec[p].gas += gasStepPerSecond;
    }

    if (trafficPersonVec[p].v == 0) {
      const ushort posInLineCells = static_cast<ushort>(trafficPersonVec[p].posInLaneM);
      const uint posToSample =
        mapToWriteShift
        + kMaxMapWidthM * (
            currentEdge
            + static_cast<int>(posInLineCells / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
            + trafficPersonVec[p].numOfLaneInEdge)
        + posInLineCells % kMaxMapWidthM;
      laneMap[posToSample] = 0;

      return;
    }

    trafficPersonVec[p].color = p << 8;
    trafficPersonVec[p].posInLaneM = trafficPersonVec[p].posInLaneM + numToMove;

    const bool reachedIntersection =
      static_cast<ushort>(ceil(trafficPersonVec[p].posInLaneM)) > currentLaneMaximumPosition ;
    if (reachedIntersection) { //reach intersection
      numToMove = trafficPersonVec[p].posInLaneM - trafficPersonVec[p].length;
    } else { //does not research next intersection
      // If the intersection has not been reached try to changed lane if:
      //   - The car is going at least 10 km per hour
      //   - 5 seconds have happened since the last lane change
      if (trafficPersonVec[p].v > 3.0f && trafficPersonVec[p].num_steps % 5 == 0) {
        // next thing is not a traffic light
        // skip if there is one lane (avoid to do this)
        // skip if it is the last edge
        if (nextVehicleIsATrafficLight == false && trafficPersonVec[p].edgeNumLanes > 1 && nextEdge != -1) {
          ////////////////////////////////////////////////////
          // LC 1 update lane changing status
          if (trafficPersonVec[p].LC_stateofLaneChanging == 0) {
            // 2.2-exp((x-1)^2)
            const float x = trafficPersonVec[p].posInLaneM / trafficPersonVec[p].length;
            if (x > 0.4f) { //just after 40% of the road
              float probabiltyMandatoryState = 2.2 - exp((x - 1) * (x - 1));
              if ((((int) (x * 100) % 100) / 100.0f) < probabiltyMandatoryState) { // pseudo random number
                trafficPersonVec[p].LC_stateofLaneChanging = 1;
              }
            }
          }

          ////////////////////////////////////////////////////
          // LC 2 NOT MANDATORY STATE
          if (trafficPersonVec[p].LC_stateofLaneChanging == 0) {
            // discretionary change: v slower than the current road limit and deccelerating and moving
            if ((trafficPersonVec[p].v < (trafficPersonVec[p].maxSpeedMperSec * 0.7f)) &&
              (dv_dt < 0) && trafficPersonVec[p].v > 3.0f) {
              bool leftLane = trafficPersonVec[p].numOfLaneInEdge > 0; //at least one lane on the left
              bool rightLane = trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].edgeNumLanes - 1; //at least one lane

              if (leftLane == true && rightLane == true) {
                if (int(trafficPersonVec[p].v) % 2 == 0) { // pseudo random
                  leftLane = false;
                } else {
                  rightLane = false;
                }
              }
              ushort laneToCheck;
              if (leftLane == true) {
                laneToCheck = trafficPersonVec[p].numOfLaneInEdge - 1;
              } else {
                laneToCheck = trafficPersonVec[p].numOfLaneInEdge + 1;
              }

              uchar v_a, v_b;
              float gap_a, gap_b;
              calculateGapsLC(mapToReadShift, laneMap, atLeastOneEnabledConnection,
                currentEdge + laneToCheck, trafficPersonVec[p].edgeNumLanes, trafficPersonVec[p].posInLaneM,
                trafficPersonVec[p].length, v_a, v_b, gap_a, gap_b);

              if (gap_a == 1000.0f && gap_b == 1000.0f) { //lag and lead car very far
                trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
              } else { // NOT ALONE
                float b1A = 0.05f, b2A = 0.15f;
                float b1B = 0.15f, b2B = 0.40f;
                // s_0-> critical lead gap
                float g_na_D, g_bn_D;
                bool acceptLC = true;

                if (gap_a != 1000.0f) {
                  g_na_D = max(s_0, s_0 + b1A * trafficPersonVec[p].v + b2A *
                    (trafficPersonVec[p].v - v_a * 3.0f));

                  if (gap_a < g_na_D) { //gap smaller than critical gap
                    acceptLC = false;
                  }
                }

                if (acceptLC == true && gap_b != 1000.0f) {
                  g_bn_D = max(s_0, s_0 + b1B * v_b * 3.0f + b2B * (v_b * 3.0f - trafficPersonVec[p].v));

                  if (gap_b < g_bn_D) { //gap smaller than critical gap
                    acceptLC = false;
                  }
                }

                if (acceptLC == true) {
                  trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
                }
              }
            }
          }// Discretionary

          ////////////////////////////////////////////////////
          // LC 3 *MANDATORY* STATE
          if (trafficPersonVec[p].LC_stateofLaneChanging == 1) {
            // LC 3.1 Calculate the correct lanes
            if (trafficPersonVec[p].LC_endOKLanes == 0xFF) {
              calculateLaneCarShouldBe(currentEdge, nextEdge, b18Intersections,
                trafficPersonVec[p].edgeNextInters, trafficPersonVec[p].edgeNumLanes,
                trafficPersonVec[p].LC_initOKLanes, trafficPersonVec[p].LC_endOKLanes);

              if (trafficPersonVec[p].LC_initOKLanes == 0 &&
                trafficPersonVec[p].LC_endOKLanes == 0) {
                //exit(0);
              }
            }


            bool leftLane = false, rightLane = false;

            // LC 3.2 CORRECT LANES--> DICRETIONARY LC WITHIN
            if (trafficPersonVec[p].numOfLaneInEdge >= trafficPersonVec[p].LC_initOKLanes &&
              trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].LC_endOKLanes) {
              // for discretionary it should be under some circustances
              if ((trafficPersonVec[p].v < (trafficPersonVec[p].maxSpeedMperSec * 0.7f)) &&
                (dv_dt < 0) && trafficPersonVec[p].v > 3.0f) {
                leftLane =
                  (trafficPersonVec[p].numOfLaneInEdge > 0) && //at least one lane on the left
                  (trafficPersonVec[p].numOfLaneInEdge - 1 >= trafficPersonVec[p].LC_initOKLanes)
                  &&
                  (trafficPersonVec[p].numOfLaneInEdge - 1 < trafficPersonVec[p].LC_endOKLanes);
                rightLane =
                  (trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].edgeNumLanes - 1) &&
                  //at least one lane
                  (trafficPersonVec[p].numOfLaneInEdge + 1 >= trafficPersonVec[p].LC_initOKLanes)
                  &&
                  (trafficPersonVec[p].numOfLaneInEdge + 1 < trafficPersonVec[p].LC_endOKLanes);
              }
            }
            // LC 3.3 INCORRECT LANES--> MANDATORY LC
            else {
              if (trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].LC_initOKLanes) {
                rightLane = true;
              } else {
                leftLane = true;
              }

              if (rightLane == true &&
                trafficPersonVec[p].numOfLaneInEdge + 1 >= trafficPersonVec[p].edgeNumLanes) {
                printf("ERROR: RT laneToCheck>=trafficPersonVec[p].edgeNumLanes\n");
              }

              if (leftLane == true && trafficPersonVec[p].numOfLaneInEdge == 0) {
                printf("ERROR %u: LT laneToCheck>=trafficPersonVec[p].edgeNumLanes OK %u-%u NE %u\n",
                  p, trafficPersonVec[p].LC_initOKLanes, trafficPersonVec[p].LC_endOKLanes,
                  nextEdge);
                //exit(0);
              }
            }

            if (leftLane == true || rightLane == true) {
              // choose lane (if necessary)
              if (leftLane == true && rightLane == true) {
                if ((int) (trafficPersonVec[p].posInLaneM) % 2 == 0) { //pseudo random
                  leftLane = false;
                } else {
                  rightLane = false;
                }
              }
              ushort laneToCheck;
              if (leftLane == true) {
                laneToCheck = trafficPersonVec[p].numOfLaneInEdge - 1;
              } else {
                laneToCheck = trafficPersonVec[p].numOfLaneInEdge + 1;
              }

              if (laneToCheck >= trafficPersonVec[p].edgeNumLanes) {
                printf("ERROR: laneToCheck>=trafficPersonVec[p].edgeNumLanes %u %u\n",
                  laneToCheck, trafficPersonVec[p].edgeNumLanes);
              }

              uchar v_a, v_b;
              float gap_a, gap_b;
              calculateGapsLC(mapToReadShift, laneMap, atLeastOneEnabledConnection,
                currentEdge + laneToCheck, trafficPersonVec[p].edgeNumLanes, trafficPersonVec[p].posInLaneM,
                trafficPersonVec[p].length, v_a, v_b, gap_a, gap_b);

              if (gap_a == 1000.0f && gap_b == 1000.0f) { //lag and lead car very far
                trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
              } else { // NOT ALONE
                float b1A = 0.05f, b2A = 0.15f;
                float b1B = 0.15f, b2B = 0.40f;
                float gamma = 0.000025;
                // s_0-> critical lead gap
                float distEnd = trafficPersonVec[p].length - trafficPersonVec[p].posInLaneM;
                float expTerm = (1 - exp(-gamma * distEnd * distEnd));

                float g_na_M, g_bn_M;
                bool acceptLC = true;

                if (gap_a != 1000.0f) {
                  g_na_M = max(s_0, s_0 + (b1A * trafficPersonVec[p].v + b2A *
                    (trafficPersonVec[p].v - v_a * 3.0f)));

                  if (gap_a < g_na_M) { //gap smaller than critical gap
                    acceptLC = false;
                  }
                }

                if (acceptLC == true && gap_b != 1000.0f) {
                  g_bn_M = max(s_0, s_0 + (b1B * v_b * 3.0f + b2B * (v_b * 3.0f -
                    trafficPersonVec[p].v)));

                  if (gap_b < g_bn_M) { //gap smaller than critical gap
                    acceptLC = false;
                  }
                }

                if (acceptLC == true) {
                  trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
                }
              }
            }
          }// Mandatory
        }//at least two lanes and not stopped by traffic light
      }

      // Update person' speed
      const uchar vInMpS = (uchar) (trafficPersonVec[p].v * 3); //speed in m/s to fit in uchar
      ushort posInLineCells = (ushort) (trafficPersonVec[p].posInLaneM);
      if (posInLineCells >= currentLaneMaximumPosition)
        posInLineCells = currentLaneMaximumPosition - 1;
      const uint posToSample =
        mapToWriteShift
        + kMaxMapWidthM * (
            currentEdge
            + static_cast<int>(posInLineCells / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
            + trafficPersonVec[p].numOfLaneInEdge)
        + posInLineCells % kMaxMapWidthM;
      laneMap[posToSample] = vInMpS;
      return;
    }

    if (nextEdge == -1) {
      trafficPersonVec[p].active = 2;
      return;
    }

    assert(reachedIntersection);
    if (nextEdgeChosenLane < 0 || nextEdgeChosenLane >= trafficPersonVec[p].nextEdgeNumLanes)
      nextEdgeChosenLane = 0;

    // Update current edge information
    trafficPersonVec[p].indexPathCurr++;
    trafficPersonVec[p].maxSpeedMperSec = trafficPersonVec[p].nextEdgemaxSpeedMperSec;
    trafficPersonVec[p].edgeNumLanes = trafficPersonVec[p].nextEdgeNumLanes;
    trafficPersonVec[p].edgeNextInters = trafficPersonVec[p].nextEdgeNextInters;
    trafficPersonVec[p].length = trafficPersonVec[p].nextEdgeLength;
    trafficPersonVec[p].posInLaneM = numToMove;
    trafficPersonVec[p].numOfLaneInEdge = nextEdgeChosenLane;
    trafficPersonVec[p].isApproachingStopJunction = false;
    trafficPersonVec[p].distanceUntilIntersection = INT_MAX >> 2;
    trafficPersonVec[p].stoppedBeforeNextIntersection = false;

    // Update person's next edge
    const uint nextEdgeIdx = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];

    if (nextEdgeIdx != -1) {
      trafficPersonVec[p].LC_initOKLanes = 0xFF;
      trafficPersonVec[p].LC_endOKLanes = 0xFF;

      trafficPersonVec[p].nextEdgemaxSpeedMperSec = edgesData[nextEdgeIdx].maxSpeedMperSec;
      trafficPersonVec[p].nextEdgeNumLanes = edgesData[nextEdgeIdx].numLines;
      trafficPersonVec[p].nextEdgeNextInters = edgesData[nextEdgeIdx].nextInters;
      trafficPersonVec[p].nextEdgeLength = edgesData[nextEdgeIdx].length;
    }

    trafficPersonVec[p].LC_stateofLaneChanging = 0;
    const uchar vInMpS = static_cast<uchar>(trafficPersonVec[p].v * 3); //speed in m/s to fit in uchar
    const ushort posInLineCells = static_cast<ushort>(trafficPersonVec[p].posInLaneM);

    const uint posToSample =
      mapToWriteShift
      + kMaxMapWidthM * (
          nextEdge
          + static_cast<int>(posInLineCells / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
          + trafficPersonVec[p].numOfLaneInEdge)
      + posInLineCells % kMaxMapWidthM;  // note the last % should not happen
    laneMap[posToSample] = vInMpS;
  }
}

__device__ void updateTrafficLight(
    const int intersectionIdx,
    float currentTime,
    LC::Intersection *intersections,
    LC::Connection *connections,
    LC::TrafficLightScheduleEntry *trafficLightSchedules) {
  LC::Intersection & intersection = intersections[intersectionIdx];
  const bool hasSchedule =
    intersection.trafficLightSchedulesEnd - intersection.trafficLightSchedulesStart > 0;
  if (!hasSchedule)
    return;

  // First disable all intersection's connections
  for (
      uint connectionIdx = intersection.connectionGraphStart;
      connectionIdx < intersection.connectionGraphEnd;
      ++connectionIdx) {
    connections[connectionIdx].enabled = false;
  }

  // Then enables the connections corresponding to the current schedule position
  const uint startingScheduleGroup = intersection.currentScheduleGroup;
  int count = 0;
  do {
    count++;
    const LC::TrafficLightScheduleEntry & scheduleEntry =
      trafficLightSchedules[intersection.scheduleIdx];

    assert(
      startingScheduleGroup == scheduleEntry.scheduleGroup
      && "Incoherent traffic schedules info");

    connections[scheduleEntry.connectionIdx].enabled = true;

    ++intersection.scheduleIdx;
  } while (
      intersection.scheduleIdx < intersection.trafficLightSchedulesEnd
      && trafficLightSchedules[intersection.scheduleIdx].scheduleGroup == startingScheduleGroup);

  // Update indexes
  if (intersection.scheduleIdx < intersection.trafficLightSchedulesEnd) {
    ++intersection.currentScheduleGroup;
  } else {
    intersection.currentScheduleGroup = 0;
    intersection.scheduleIdx = intersection.trafficLightSchedulesStart;
  }

  // Update next event
  intersection.timeOfNextUpdate =
    currentTime + trafficLightSchedules[intersection.scheduleIdx].scheduledTime;
}

/*
 * Compute the score of the input lane. Lower score means the lane has more priority.
 * */
__device__ float inLaneScore(
    const uint laneIdx,
    const LC::Intersection & intersection,
    const LC::Connection * connections,
    const LC::B18EdgeData * edgesData,
    const uint mapToReadShift,
    const uchar * laneMap) {
  const size_t maxDistanceToCheck = 30;
  uint correspondingEdgeIdx = -1;
  for (
      uint connectionIdx = intersection.connectionGraphStart;
      connectionIdx < intersection.connectionGraphEnd;
      ++connectionIdx) {
    if (connections[connectionIdx].inLaneNumber == laneIdx) {
      correspondingEdgeIdx = connections[connectionIdx].inEdgeNumber;
      break;
    }
  }

  if (correspondingEdgeIdx == -1)
    return maxDistanceToCheck;

  const ushort laneMaximumPosition =
    ceil(edgesData[correspondingEdgeIdx].length - intersectionClearance);

  // Find the closest car in the lane
  int currentDistance = 0;
  int currentPosition = laneMaximumPosition;
  while (currentDistance < maxDistanceToCheck && currentPosition > 0) {
    size_t posToSample = mapToReadShift + kMaxMapWidthM * laneIdx + currentPosition;
    const uchar laneChar = laneMap[posToSample];
    if (laneChar != 0xFF) {
      // A car was found in the lane
      break;
    }
    --currentPosition;
    ++currentDistance;
  }
  float score = currentDistance;
  if (edgesData[correspondingEdgeIdx].startsAtHighway)
    score /= 2;
  return score;
}

/*
 * Sort the corresponding chunk of the entering lanes indexes using insertion sort and using the
 * in lanes scores.
 */
__device__ void sortByInLaneScore(
    float currentTime,
    uint * inLanesIndexes,
    const LC::Intersection & intersection,
    const uchar *laneMap,
    const LC::Connection * connections,
    const LC::B18EdgeData * edgesData,
    uint mapToReadShift) {
  const auto score = [&] (const uint laneIdx) {
    return inLaneScore(laneIdx, intersection, connections, edgesData, mapToReadShift, laneMap);
  };

  const uint from = intersection.inLanesIndexesStart;
  const uint to = intersection.inLanesIndexesEnd;

  uint mainIdx, secondIdx, tmpValue;
  mainIdx = from + 1;
  while (mainIdx < to) {
    secondIdx = mainIdx;
    while (
        secondIdx > 0
        && score(inLanesIndexes[secondIdx - 1]) > score(inLanesIndexes[secondIdx])) {
      tmpValue = inLanesIndexes[secondIdx];
      inLanesIndexes[secondIdx] = inLanesIndexes[secondIdx - 1];
      inLanesIndexes[secondIdx - 1] = tmpValue;
      --secondIdx;
    }
    ++mainIdx;
  }
}

__device__ void updateUnsupervised(
    const int intersectionIdx,
    float currentTime,
    uint mapToReadShift,
    const uchar *laneMap,
    LC::Intersection *intersections,
    LC::Connection *connections,
    LC::B18EdgeData *edgesData,
    uint *connectionsBlocking,
    uint *inLanesIndexes) {
  LC::Intersection & intersection = intersections[intersectionIdx];

  sortByInLaneScore(
    currentTime,
    inLanesIndexes,
    intersection,
    laneMap,
    connections,
    edgesData,
    mapToReadShift
  );

  // Enable the intersection's connections
  for (
      uint connectionIdx = intersection.connectionGraphStart;
      connectionIdx < intersection.connectionGraphEnd;
      ++connectionIdx) {
    connections[connectionIdx].enabled = true;
  }

  for (
      uint inLaneIdx = intersection.inLanesIndexesStart;
      inLaneIdx < intersection.inLanesIndexesEnd;
      ++inLaneIdx) {
    uint targetLane = inLanesIndexes[inLaneIdx];
    // For each entering lane...
    for (
        uint connectionIdx = intersection.connectionGraphStart;
        connectionIdx < intersection.connectionGraphEnd;
        ++connectionIdx) {
      // ..for each connection starting from that lane...
      const LC::Connection & connection = connections[connectionIdx];
      if (connection.inLaneNumber != targetLane)
        continue;

      if (!connection.enabled)
        continue;

      // ..keep it enabled and disable the corresponding blocked connections
      for (
          uint i = connection.connectionsBlockingStart;
          i < connection.connectionsBlockingEnd;
          ++i) {
        const uint blockedConnectionIdx = connectionsBlocking[i];
        assert(connectionIdx != blockedConnectionIdx);
        connections[blockedConnectionIdx].enabled = false;
      }
    }
  }

  // Update every 6 seconds
  intersection.timeOfNextUpdate = currentTime + 6;
}

__global__ void kernel_updateIntersectionConnections(
    float currentTime,
    uint mapToReadShift,
    LC::Intersection *intersections,
    size_t amountOfIntersections,
    LC::Connection *connections,
    uint *connectionsBlocking,
    LC::TrafficLightScheduleEntry *trafficLightSchedules,
    uint *inLanesIndexes,
    LC::B18EdgeData * edgesData,
    const uchar *laneMap) {
  const int intersectionIdx = blockIdx.x * blockDim.x + threadIdx.x;
  if (intersectionIdx < amountOfIntersections) {
    LC::Intersection & intersection = intersections[intersectionIdx];

    const int amountOfConnections =
      intersection.connectionGraphEnd - intersection.connectionGraphStart;
    const bool hasConnections = amountOfConnections > 0;
    const bool needsUpdate = currentTime >= intersection.timeOfNextUpdate;

    if (!needsUpdate || !hasConnections)
      return;

    if (intersection.trafficControl == TrafficControl::TrafficLight) {
      updateTrafficLight(
          intersectionIdx,
          currentTime,
          intersections,
          connections,
          trafficLightSchedules);
    } else if (intersection.trafficControl == TrafficControl::Unsupervised) {
      updateUnsupervised(
          intersectionIdx,
          currentTime,
          mapToReadShift,
          laneMap,
          intersections,
          connections,
          edgesData,
          connectionsBlocking,
          inLanesIndexes);
    }
  }
}

__global__ void kernel_sampleTraffic(
  int numPeople,
  LC::B18TrafficPerson *trafficPersonVec,
  uint *indexPathVec,
  float *accSpeedPerLinePerTimeInterval,
  float *numVehPerLinePerTimeInterval, //this could have been int
  uint offset
  ) {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p < numPeople) {//CUDA check (inside margins)
    if (trafficPersonVec[p].active == 1) { // just active
      int edgeNum = indexPathVec[trafficPersonVec[p].indexPathCurr];
      accSpeedPerLinePerTimeInterval[edgeNum + offset] += trafficPersonVec[p].v / 3.0f;
      numVehPerLinePerTimeInterval[edgeNum + offset]++;
    }
  }
}
__global__ void kernel_resetPeople(
  int numPeople,
  LC::B18TrafficPerson *trafficPersonVec) {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p < numPeople) {//CUDA check (inside margins)
    trafficPersonVec[p].active = 0;
  }
}

void b18GetSampleTrafficCUDA(std::vector<float>& accSpeedPerLinePerTimeInterval, std::vector<float>& numVehPerLinePerTimeInterval) {
  // copy back people
  const size_t size = accSpeedPerLinePerTimeInterval.size() * sizeof(float);
  hipMemcpy(accSpeedPerLinePerTimeInterval.data(), accSpeedPerLinePerTimeInterval_d, size, hipMemcpyDeviceToHost);

  const size_t sizeI = numVehPerLinePerTimeInterval.size() * sizeof(uchar);
  hipMemcpy(numVehPerLinePerTimeInterval.data(), numVehPerLinePerTimeInterval_d, sizeI, hipMemcpyDeviceToHost);
}

void b18ResetPeopleLanesCUDA(uint numPeople) {
  kernel_resetPeople<<<ceil(numPeople / 1024.0f), 1024>>>(numPeople, trafficPersonVec_d);
  hipMemset(&laneMap_d[0], -1, halfLaneMap*sizeof(unsigned char));
  hipMemset(&laneMap_d[halfLaneMap], -1, halfLaneMap*sizeof(unsigned char));
}

void b18SimulateTrafficCUDA(const float currentTime, uint numPeople, uint numIntersections) {
  ////////////////////////////////////////////////////////////
  // 1. CHANGE MAP: set map to use and clean the other
  if(readFirstMapC==true){
    mapToReadShift=0;
    mapToWriteShift=halfLaneMap;
    gpuErrchk(hipMemset(&laneMap_d[halfLaneMap], -1, halfLaneMap*sizeof(unsigned char)));//clean second half
  }else{
    mapToReadShift=halfLaneMap;
    mapToWriteShift=0;
    gpuErrchk(hipMemset(&laneMap_d[0], -1, halfLaneMap*sizeof(unsigned char)));//clean first half
  }
  readFirstMapC=!readFirstMapC;//next iteration invert use

  // Update intersections.
  kernel_updateIntersectionConnections<<<ceil(numIntersections / 512.0f), 512>>>(
    currentTime,
    mapToReadShift,
    deviceIntersections,
    amountOfIntersections,
    deviceConnections,
    deviceConnectionsBlocking,
    deviceTrafficLightSchedules,
    deviceInLanesIndexes,
    edgesData_d,
    laneMap_d);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  // Simulate people.
  kernel_updatePersonsCars<<<ceil(numPeople / 384.0f), 384>>>(
    numPeople,
    currentTime,
    mapToReadShift,
    mapToWriteShift,
    halfLaneMap,
    trafficPersonVec_d,
    indexPathVec_d,
    edgesData_d,
    laneMap_d,
    intersections_d,
    deviceConnections,
    amountOfConnections,
    deviceConnectionsBlocking,
    deviceIntersections,
    amountOfIntersections,
    deviceTrafficLightSchedules);
  gpuErrchk(hipPeekAtLastError());
  gpuErrchk(hipDeviceSynchronize());

  // Sample if necessary.
  if ((((float) ((int) currentTime)) == (currentTime)) &&
    ((int) currentTime % ((int) 30)) == 0) { //3min //(sample double each 3min)
    int samplingNumber = (currentTime - startTime) / (30 * numStepsTogether);
    uint offset = numIntersections * samplingNumber;
    kernel_sampleTraffic<<<ceil(numPeople / 1024.0f), 1024>>>(numPeople, trafficPersonVec_d, indexPathVec_d, accSpeedPerLinePerTimeInterval_d, numVehPerLinePerTimeInterval_d, offset);
    gpuErrchk(hipPeekAtLastError());
  }
}//
