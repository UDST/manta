#include "hip/hip_runtime.h"
//CUDA CODE
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""
#include "assert.h"

#include "b18TrafficPerson.h"
#include "b18EdgeData.h"
#include <vector>
#include <iostream>

#include "../../src/benchmarker.h"
#include "sp/config.h"

#ifndef ushort
#define ushort uint16_t
#endif
#ifndef uint
#define uint uint32_t
#endif
#ifndef uchar
#define uchar uint8_t
#endif

///////////////////////////////
// CONSTANTS

__constant__ float intersectionClearance = 7.8f; //TODO(pavan): WHAT IS THIS?

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}
inline void printMemoryUsage() {
  // show memory usage of GPU
  size_t free_byte;
  size_t total_byte;
  hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);
  if (hipSuccess != cuda_status) {
    printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status));
    exit(1);
  }
  double free_db = (double) free_byte;
  double total_db = (double) total_byte;
  double used_db = total_db - free_db;
  printf("GPU memory usage: used = %.0f, free = %.0f MB, total = %.0f MB\n", used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}
////////////////////////////////
// VARIABLES
LC::B18TrafficPerson *trafficPersonVec_d;
uint *indexPathVec_d;
LC::B18EdgeData *edgesData_d;

__constant__ bool calculatePollution = true;
__constant__ float cellSize = 1.0f;

//__constant__ float deltaTime = 0.5f;
//const float deltaTimeH = 0.5f;

//const uint numStepsPerSample = 30.0f / deltaTimeH; //each min
//const uint numStepsTogether = 12; //change also in density (10 per hour)

uchar *laneMap_d;
bool readFirstMapC=true;
uint mapToReadShift;
uint mapToWriteShift;
uint halfLaneMap;
float startTime;


LC::B18IntersectionData *intersections_d;
uchar *trafficLights_d;

float* accSpeedPerLinePerTimeInterval_d;
float* numVehPerLinePerTimeInterval_d;

void b18InitCUDA(
  bool fistInitialization,
  std::vector<LC::B18TrafficPerson>& trafficPersonVec, 
  std::vector<uint> &indexPathVec, 
  std::vector<LC::B18EdgeData>& edgesData, 
  std::vector<uchar>& laneMap, 
  std::vector<uchar>& trafficLights, 
  std::vector<LC::B18IntersectionData>& intersections,
  float startTimeH, float endTimeH,
  std::vector<float>& accSpeedPerLinePerTimeInterval,
  std::vector<float>& numVehPerLinePerTimeInterval,
  float deltaTime) {
  //printf(">>b18InitCUDA firstInitialization %s\n", (fistInitialization?"INIT":"ALREADY INIT"));
  //printMemoryUsage();

  const uint numStepsPerSample = 30.0f / deltaTime; //each min
  const uint numStepsTogether = 12; //change also in density (10 per hour)
  { // people
    size_t size = trafficPersonVec.size() * sizeof(LC::B18TrafficPerson);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &trafficPersonVec_d, size));   // Allocate array on device
    gpuErrchk(hipMemcpy(trafficPersonVec_d, trafficPersonVec.data(), size, hipMemcpyHostToDevice));
  }
  
  { // indexPathVec
    size_t sizeIn = indexPathVec.size() * sizeof(uint);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &indexPathVec_d, sizeIn));   // Allocate array on device
    gpuErrchk(hipMemcpy(indexPathVec_d, indexPathVec.data(), sizeIn, hipMemcpyHostToDevice));
  }
  {//edgeData
    size_t sizeD = edgesData.size() * sizeof(LC::B18EdgeData);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &edgesData_d, sizeD));   // Allocate array on device
    gpuErrchk(hipMemcpy(edgesData_d, edgesData.data(), sizeD, hipMemcpyHostToDevice));
  }
  {//laneMap
    size_t sizeL = laneMap.size() * sizeof(uchar);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &laneMap_d, sizeL));   // Allocate array on device
    gpuErrchk(hipMemcpy(laneMap_d, laneMap.data(), sizeL, hipMemcpyHostToDevice));
    halfLaneMap = laneMap.size() / 2;
  }
  {// intersections
    size_t sizeI = intersections.size() * sizeof(LC::B18IntersectionData);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &intersections_d, sizeI));   // Allocate array on device
    gpuErrchk(hipMemcpy(intersections_d, intersections.data(), sizeI, hipMemcpyHostToDevice));
    size_t sizeT = trafficLights.size() * sizeof(uchar);//total number of lanes
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &trafficLights_d, sizeT));   // Allocate array on device
    gpuErrchk(hipMemcpy(trafficLights_d, trafficLights.data(), sizeT, hipMemcpyHostToDevice));
  }
  {
    startTime = startTimeH * 3600.0f;
    uint numSamples = ceil(((endTimeH*3600.0f - startTimeH*3600.0f) / (deltaTime * numStepsPerSample * numStepsTogether))) + 1; //!!!
    accSpeedPerLinePerTimeInterval.clear();
    numVehPerLinePerTimeInterval.clear();
    accSpeedPerLinePerTimeInterval.resize(numSamples * trafficLights.size());
    numVehPerLinePerTimeInterval.resize(numSamples * trafficLights.size());
    size_t sizeAcc = accSpeedPerLinePerTimeInterval.size() * sizeof(float);
    if (fistInitialization)gpuErrchk(hipMalloc((void **) &accSpeedPerLinePerTimeInterval_d, sizeAcc));   // Allocate array on device
    if (fistInitialization)gpuErrchk(hipMalloc((void **) &numVehPerLinePerTimeInterval_d, sizeAcc));   // Allocate array on device
    gpuErrchk(hipMemset(&accSpeedPerLinePerTimeInterval_d[0], 0, sizeAcc));
    gpuErrchk(hipMemset(&numVehPerLinePerTimeInterval_d[0], 0, sizeAcc));
  }
  printMemoryUsage();
}//

void b18FinishCUDA(void){
  //////////////////////////////
  // FINISH
  hipFree(trafficPersonVec_d);
  hipFree(indexPathVec_d);
  hipFree(edgesData_d);
  hipFree(laneMap_d);
  hipFree(intersections_d);
  hipFree(trafficLights_d);

  hipFree(accSpeedPerLinePerTimeInterval_d);
  hipFree(numVehPerLinePerTimeInterval_d);
}//

 void b18GetDataCUDA(std::vector<LC::B18TrafficPerson>& trafficPersonVec, std::vector<LC::B18EdgeData> &edgesData){
   // copy back people
   size_t size = trafficPersonVec.size() * sizeof(LC::B18TrafficPerson);
   size_t size_edges = edgesData.size() * sizeof(LC::B18EdgeData);
   hipMemcpy(trafficPersonVec.data(),trafficPersonVec_d,size,hipMemcpyDeviceToHost);//hipMemcpyHostToDevice
   hipMemcpy(edgesData.data(),edgesData_d,size_edges,hipMemcpyDeviceToHost);//hipMemcpyHostToDevice
 }


 __device__ void calculateGapsLC(
   uint mapToReadShift,
   uchar* laneMap,
   uchar trafficLightState,
   uint laneToCheck,
   ushort numLinesEdge,
   float posInMToCheck,
   float length,
   uchar &v_a,
   uchar &v_b,
   float &gap_a,
   float &gap_b) {

   ushort numOfCells = ceil(length);
   ushort initShift = ceil(posInMToCheck);
   uchar laneChar;
   bool found = false;

   // CHECK FORWARD
   //printf("initShift %u numOfCells %u\n",initShift,numOfCells);
   for (ushort b = initShift - 1; (b < numOfCells) && (found == false); b++) { //NOTE -1 to make sure there is none in at the same level
     // laneChar = laneMap[mapToReadShift + maxWidth * (laneToCheck) + b];
     const uint posToSample = mapToReadShift + kMaxMapWidthM * (laneToCheck + (((int) (b / kMaxMapWidthM)) * numLinesEdge)) + b % kMaxMapWidthM;
     laneChar = laneMap[posToSample];

     if (laneChar != 0xFF) {
       gap_a = ((float) b - initShift); //m
       v_a = laneChar; //laneChar is in 3*ms (to save space in array)
       found = true;
       break;
     }
   }

   if (found == false) {
     if (trafficLightState == 0x00) { //red
       //found=true;
       gap_a = gap_b = 1000.0f; //force to change to the line without vehicle
       v_a = v_b = 0xFF;
       return;
     }
   }

   if (found == false) {
     gap_a = 1000.0f;
   }

   // CHECK BACKWARDS
   found = false;

   //printf("2initShift %u numOfCells %u\n",initShift,numOfCells);
   for (int b = initShift + 1; (b >= 0) && (found == false); b--) {  // NOTE +1 to make sure there is none in at the same level
     //laneChar = laneMap[mapToReadShift + maxWidth * (laneToCheck) + b];
     const uint posToSample = mapToReadShift + kMaxMapWidthM * (laneToCheck + (((int) (b / kMaxMapWidthM)) * numLinesEdge)) + b % kMaxMapWidthM;
     laneChar = laneMap[posToSample];
     if (laneChar != 0xFF) {
       gap_b = ((float) initShift - b); //m
       v_b = laneChar; //laneChar is in 3*ms (to save space in array)
       found = true;
       break;
     }
   }

   //printf("3initShift %u numOfCells %u\n",initShift,numOfCells);
   if (found == false) {
     gap_b = 1000.0f;
   }

  }//

__device__ void calculateLaneCarShouldBe(
  uint curEdgeLane,
  uint nextEdge,
  LC::B18IntersectionData* intersections,
  uint edgeNextInters,
  ushort edgeNumLanes,
  ushort &initOKLanes,
  ushort &endOKLanes) {

  initOKLanes = 0;
  endOKLanes = edgeNumLanes;
  bool currentEdgeFound = false;
  bool exitFound = false;
  ushort numExitToTake = 0;
  ushort numExists = 0;

  for (int eN = intersections[edgeNextInters].totalInOutEdges - 1; eN >= 0; eN--) {  // clockwise
    uint procEdge = intersections[edgeNextInters].edge[eN];

    if ((procEdge & kMaskLaneMap) == curEdgeLane) { //current edge 0xFFFFF
      currentEdgeFound = true;
      if (exitFound == false) {
        numExitToTake = 0;
      }
      continue;
    }

    if ((procEdge & kMaskInEdge) == 0x0) { //out edge 0x800000
      numExists++;
      if (currentEdgeFound == true) {
        numExitToTake++;
      }
      if (currentEdgeFound == false && exitFound == false) {
        numExitToTake++;
      }
    }
    if ((procEdge & kMaskInEdge) == nextEdge) {
      exitFound = true;
      currentEdgeFound = false;
    }
  }

  if (edgeNumLanes == 0) {
    printf("ERRRROR\n");
  }

  switch (edgeNumLanes) {
    /// ONE LANE
    case 1:
      initOKLanes = 0;
      endOKLanes = 1;
      break;

    /// TWO LANE
    case 2:
      switch (numExists) {
      case 1:
      case 2://all okay
        initOKLanes = 0;
        endOKLanes = 2;
        break;

      case 3:
        if (numExitToTake > 2) { //left
          initOKLanes = 0;
          endOKLanes = 1;
          break;
        }

        initOKLanes = 1;
        endOKLanes = 2;
        break;

      default:
        if (numExitToTake >= numExists - 1) {
          initOKLanes = 0;
          endOKLanes = 1;
          break;
        }

        initOKLanes = 1;
        endOKLanes = 2;
        break;
      }

      break;

      /// THREE LANE
    case 3:
      switch (numExists) {
      case 1:
      case 2://all okay
        initOKLanes = 0;
        endOKLanes = 3;
        break;

      case 3:
        if (numExitToTake > 2) { //left
          initOKLanes = 0;
          endOKLanes = 1;
          break;
        }

        initOKLanes = 1;
        endOKLanes = 3;
        break;

      default:
        if (numExitToTake >= numExists - 1) {
          initOKLanes = 0;
          endOKLanes = 1;
          break;
        }

        initOKLanes = 1;
        endOKLanes = 2;
        break;
      }

      break;

    case 4:
      switch (numExists) {
      case 1:
      case 2://all okay
        initOKLanes = 0;
        endOKLanes = 4;
        break;

      case 3:
        if (numExitToTake == 1) { //right
          initOKLanes = 3;
          endOKLanes = 4;
        }

        if (numExitToTake > 3) { //left
          initOKLanes = 0;
          endOKLanes = 1;
          break;
        }

        initOKLanes = 1;
        endOKLanes = 4;
        break;

      default:
        if (numExitToTake == 1) { //right
          initOKLanes = edgeNumLanes - 1;
          endOKLanes = edgeNumLanes;
        }

        if (numExitToTake >= numExists - 2) {
          initOKLanes = 0;
          endOKLanes = 2;
          break;
        }

        initOKLanes = 1; //also lane 2
        endOKLanes = edgeNumLanes;
      }

      break;

    default:
      switch (numExists) {
      case 1:
      case 2://all okay
        initOKLanes = 0;
        endOKLanes = edgeNumLanes;
        break;

      case 3:
        if (numExitToTake == 1) { //right
          initOKLanes = edgeNumLanes - 1;
          endOKLanes = edgeNumLanes;
        }

        if (numExitToTake > edgeNumLanes - 2) { //left
          initOKLanes = 0;
          endOKLanes = 2;
          break;
        }

        initOKLanes = 1;
        endOKLanes = edgeNumLanes;
        break;

      default:
        if (numExitToTake < 2) { //right
          initOKLanes = edgeNumLanes - 2;
          endOKLanes = edgeNumLanes;
        }

        if (numExitToTake >= numExists - 2) {
          initOKLanes = 0;
          endOKLanes = 2;
          break;
        }

        initOKLanes = 1; //also lane 2
        endOKLanes = edgeNumLanes - 1;
      }

      break;
  }
}//

 // Kernel that executes on the CUDA device
__global__ void kernel_trafficSimulation(
  int numPeople,
  float currentTime,
  uint mapToReadShift,
  uint mapToWriteShift,
  LC::B18TrafficPerson *trafficPersonVec,
  uint *indexPathVec,
  LC::B18EdgeData* edgesData,
  uchar *laneMap,
  LC::B18IntersectionData *intersections,
  uchar *trafficLights,
  float deltaTime,
  const parameters simParameters)
  {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  //printf("p %d Numpe %d\n",p,numPeople);
  if (p < numPeople) {//CUDA check (inside margins)
    if (trafficPersonVec[p].active == 2) {
      return;
    }
    // set up next edge info
    uint nextEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];

    ///////////////////////////////
    //2.1. check if person should still wait or should start
    if (trafficPersonVec[p].active == 0) {

      //printf("  1. Person: %d active==0\n",p);
      if (trafficPersonVec[p].time_departure > currentTime) { //wait
        //1.1 just continue waiting
        //printf("   1.1 Person: %d wait\n",p);
        return;
      } else { //start
        //printf("p %d edge = %u\n", p, trafficPersonVec[p].indexPathInit);
        //1.2 find first edge
        trafficPersonVec[p].indexPathCurr = trafficPersonVec[p].indexPathInit; // reset index.
        uint firstEdge = indexPathVec[trafficPersonVec[p].indexPathCurr];
        //printf("indexPathVec %d = %u nextEdge = %u\n", p, indexPathVec[trafficPersonVec[p].indexPathCurr], indexPathVec[trafficPersonVec[p].indexPathCurr + 1]);
      

        if (firstEdge == -1) {
          trafficPersonVec[p].active = 2;
          //printf("0xFFFF\n");
          return;
        }

        //1.3 update person edgeData

        // COPY DATA FROM EDGE TO PERSON
        trafficPersonVec[p].edgeNumLanes = edgesData[firstEdge].numLines;
        trafficPersonVec[p].edgeNextInters = edgesData[firstEdge].nextIntersMapped;
        //printf("edgeNextInters %u = %u\n", firstEdge, edgesData[firstEdge].nextIntersMapped);

        trafficPersonVec[p].length = edgesData[firstEdge].length;
              
        //printf("edgesData length %f\n",edgesData[firstEdge].length);
        trafficPersonVec[p].maxSpeedMperSec = edgesData[firstEdge].maxSpeedMperSec;
        //printf("edgesData %.10f\n",edgesData[firstEdge].maxSpeedMperSec);
        //1.4 try to place it in middle of edge
        ushort numOfCells = ceil(trafficPersonVec[p].length);
        ushort initShift = (ushort) (0.5f * numOfCells); //number of cells it should be placed (half of road)

        uchar laneChar;
        bool placed = false;

        ushort numCellsEmptyToBePlaced = simParameters.s_0;
        ushort countEmptyCells = 0;
        //printf("b = %d, numOfCells = %d\n", initShift, numOfCells);
        for (ushort b = initShift; (b < numOfCells) && (placed == false); b++) {
          ushort lN = trafficPersonVec[p].edgeNumLanes - 1; //just right LANE !!!!!!!
          laneChar = laneMap[mapToReadShift + kMaxMapWidthM * (firstEdge + lN) +
            b]; //get byte of edge (proper line)

          if (laneChar != 0xFF) {
            countEmptyCells = 0;
            continue;
          }

          countEmptyCells++;// ensure there is enough room to place the car

          if (countEmptyCells < numCellsEmptyToBePlaced) {
            continue;
          }

          trafficPersonVec[p].numOfLaneInEdge = lN;
          trafficPersonVec[p].posInLaneM = b; //m
          uchar vInMpS = (uchar) (trafficPersonVec[p].v *
            3); //speed in m/s *3 (to keep more precision
          laneMap[mapToWriteShift + kMaxMapWidthM * (firstEdge + lN) + b] = vInMpS; //TODO(pavan): WHAT IS THIS?
          placed = true;
          //printf("Placed\n");
          break;
        }

        if (placed == false) { //not posible to start now
          return;
        }

        trafficPersonVec[p].v = 0;
        trafficPersonVec[p].LC_stateofLaneChanging = 0;

        //1.5 active car

        trafficPersonVec[p].active = 1;
        trafficPersonVec[p].isInIntersection = 0;
        trafficPersonVec[p].num_steps = 1;
        trafficPersonVec[p].co = 0.0f;
        trafficPersonVec[p].gas = 0.0f;
        //trafficPersonVec[p].nextPathEdge++;//incremet so it continues in next edge

        //trafficPersonVec[p].nextEdge=nextEdge;
        if (nextEdge != -1) {
          trafficPersonVec[p].nextEdgemaxSpeedMperSec =
            edgesData[nextEdge].maxSpeedMperSec;
          trafficPersonVec[p].nextEdgeNumLanes = edgesData[nextEdge].numLines;
          trafficPersonVec[p].nextEdgeNextInters = edgesData[nextEdge].nextIntersMapped;
          trafficPersonVec[p].nextEdgeLength = edgesData[nextEdge].length;
          //trafficPersonVec[p].nextPathEdge++;
          trafficPersonVec[p].LC_initOKLanes = 0xFF;
          trafficPersonVec[p].LC_endOKLanes = 0xFF;
        }
        return;
      }
    }
    

    ///////////////////////////////
    //2. it is moving
    //if (float(currentTime) == int(currentTime)) { // assuming deltatime = 0.5f --> each second
    trafficPersonVec[p].num_steps++;
    //}
    //2.1 try to move
    float numMToMove;
    bool getToNextEdge = false;
    bool nextVehicleIsATrafficLight = false;
    uint currentEdge = indexPathVec[trafficPersonVec[p].indexPathCurr];

    //when we're on a new edge for the first time
    if (currentEdge == trafficPersonVec[p].nextEdge) {
      trafficPersonVec[p].end_time_on_prev_edge = currentTime - deltaTime;
      float elapsed_s = (trafficPersonVec[p].end_time_on_prev_edge - trafficPersonVec[p].start_time_on_prev_edge); //multiply by delta_time to get seconds elapsed (not half seconds)

      // We filter whenever elapsed_s == 0, which means the time granularity was not enough to measure the speed
      // We also filter whenever 0 > elapsed_s > 5, because it causes manual_v to turn extraordinarily high
      if (elapsed_s > 5) {
        trafficPersonVec[p].manual_v = edgesData[trafficPersonVec[p].prevEdge].length / elapsed_s;
        edgesData[trafficPersonVec[p].prevEdge].curr_iter_num_cars += 1;
        edgesData[trafficPersonVec[p].prevEdge].curr_cum_vel += trafficPersonVec[p].manual_v;
      }

      trafficPersonVec[p].start_time_on_prev_edge = currentTime;
      trafficPersonVec[p].prevEdge = currentEdge;
    }
    trafficPersonVec[p].nextEdge = nextEdge;
    

    // www.vwi.tu-dresden.de/~treiber/MicroApplet/IDM.html
    // IDM
    float thirdTerm = 0;
    ///////////////////////////////////////////////////
    // 2.1.1 Find front car
    int numCellsCheck = max(30.0f, trafficPersonVec[p].v * deltaTime * 2); //30 or double of the speed*time
    
    // a) SAME LINE (BEFORE SIGNALING)
    bool found = false;
    bool noFirstInLaneBeforeSign = false; //use for stop control (just let 1st to pass) TODO(pavan): I DON'T GET THIS
    bool noFirstInLaneAfterSign = false; //use for stop control (just let 1st to pass)
    float s;
    float delta_v;
    uchar laneChar;
    ushort byteInLine = (ushort) floor(trafficPersonVec[p].posInLaneM);
    ushort numOfCells = ceil((trafficPersonVec[p].length - intersectionClearance)); //intersectionClearance hardcoded to 7.8f - why?

    for (ushort b = byteInLine + 2; (b < numOfCells) && (found == false) && (numCellsCheck > 0); b++, numCellsCheck--) {
      // ShiftRead + WIDTH * (width number * # lanes + # laneInEdge) + b  TODO(pavan): WHAT IS THIS?
      //TODO(pavan): double check what mapToReadShift is printing out
      const uint posToSample = mapToReadShift + kMaxMapWidthM * (indexPathVec[trafficPersonVec[p].indexPathCurr] + (((int) (byteInLine / kMaxMapWidthM)) * trafficPersonVec[p].edgeNumLanes) + trafficPersonVec[p].numOfLaneInEdge) + b % kMaxMapWidthM;
      laneChar = laneMap[posToSample];

      //TODO(pavan): Is this clause for when it is not at the intersection yet but it has found a car in front of it?
      if (laneChar != 0xFF) {
        s = ((float) (b - byteInLine)); //m
        delta_v = trafficPersonVec[p].v - (laneChar / 3.0f); //laneChar is in 3*ms (to save space in array)
        found = true;
        noFirstInLaneBeforeSign = true; 
        break;
      }
    } 

  /*
    // b) TRAFFIC LIGHT
    if (byteInLine < numOfCells && found == false && numCellsCheck > 0) { //before traffic signaling (and not cell limited) TODO(pavan): Is this clause for when it is now at the intersection?
      if (trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge] == 0x00) { //red
        s = ((float) (numOfCells - byteInLine)); //m
        delta_v = trafficPersonVec[p].v - 0; //it should be treated as an obstacle

        //uncomment the following 2 lines if we want only red lights; comment them out if we want only green lights
        nextVehicleIsATrafficLight = true;
        //printf("\nFOUND TL\n",s,delta_v);
        found = true;
      }
    }
    
  // c) SAME LINE (AFTER SIGNALING)
    for (ushort b = byteInLine + 2; (b < numOfCells) && (found == false) && (numCellsCheck > 0); b++, numCellsCheck--) {
      // laneChar = laneMap[mapToReadShift + maxWidth * t(indexPathVec[rafficPersonVec[p].indexPathCurr] + trafficPersonVec[p].numOfLaneInEdge) + b];
      const uint posToSample = mapToReadShift + kMaxMapWidthM * (indexPathVec[trafficPersonVec[p].indexPathCurr] + (((int) (byteInLine / kMaxMapWidthM)) * trafficPersonVec[p].edgeNumLanes) + trafficPersonVec[p].numOfLaneInEdge) + b % kMaxMapWidthM;
      laneChar = laneMap[posToSample];

      if (laneChar != 0xFF) {
        s = ((float) (b - byteInLine)); //m
        delta_v = trafficPersonVec[p].v - (laneChar /
          3.0f); //laneChar is in 3*ms (to save space in array)
        found = true;
        noFirstInLaneAfterSign = true;
        break;
      }
    }

  // d) IF IT REACHES A STOP SIGN
  //TODO(pavan): This never happens because all the traffic lights are set as 0x00 (red light) (b18TrafficLaneMap.cpp)
    if (trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge] == 0x0F && numCellsCheck > 0) { //stop 
    if (trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge] == 0x0F && numCellsCheck > 0) { //stop 
  if (trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge] == 0x0F && numCellsCheck > 0) { //stop 
      //check
      if (noFirstInLaneBeforeSign == false && byteInLine < numOfCells && //first before traffic
        trafficPersonVec[p].v == 0 && //stopped
        noFirstInLaneAfterSign == false) { // noone after the traffic light (otherwise wait before stop) !! TODO also check the beginning of next edge

        trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge] = 0x00; //reset stop
        trafficPersonVec[p].posInLaneM = ceilf(numOfCells) + 1; //move magicly after stop

      } else { //stop before STOP
        if (noFirstInLaneBeforeSign == false) { //just update this if it was the first one before sign
          s = ((float) (numOfCells - byteInLine)); //m
          delta_v = trafficPersonVec[p].v - 0; //it should be treated as an obstacle
          nextVehicleIsATrafficLight = true;
          found = true;
        }
      }
    }
  */
    // NEXT LINE
  // e) MOVING ALONG IN THE NEXT EDGE
    if (found == false && numCellsCheck > 0) { //check if in next line
      if ((nextEdge != -1) &&
        (trafficPersonVec[p].edgeNextInters != trafficPersonVec[p].end_intersection)) { // we haven't arrived to destination (check next line)
        ushort nextEdgeLaneToBe = trafficPersonVec[p].numOfLaneInEdge; //same lane

        //printf("trafficPersonVec[p].numOfLaneInEdge %u\n",trafficPersonVec[p].numOfLaneInEdge);
        if (nextEdgeLaneToBe >= trafficPersonVec[p].nextEdgeNumLanes) {
          nextEdgeLaneToBe = trafficPersonVec[p].nextEdgeNumLanes -
            1; //change line if there are less roads
        }

        //printf("2trafficPersonVec[p].numOfLaneInEdge %u\n",trafficPersonVec[p].numOfLaneInEdge);
        ushort numOfCells = ceil(trafficPersonVec[p].nextEdgeLength);

        for (ushort b = 0; (b < numOfCells) && (found == false) && (numCellsCheck > 0); b++, numCellsCheck--) {
          //laneChar = laneMap[mapToReadShift + maxWidth * (nextEdge + nextEdgeLaneToBe) + b];
          const uint posToSample = mapToReadShift + kMaxMapWidthM * (nextEdge + nextEdgeLaneToBe) + b; // b18 not changed since we check first width
          laneChar = laneMap[posToSample];

          if (laneChar != 0xFF) {
            s = ((float) (b)); //m
            delta_v = trafficPersonVec[p].v - (laneChar / 3.0f);  // laneChar is in 3*ms (to save space in array)
            found = true;
            break;
          }
        }
      }
    }


    float s_star;
    //if (p == 13) {
    //        printf("delta_v[%d] = %f\n", p, delta_v);
    //}

    if (found == true && delta_v > 0) { //car in front and slower than us
    //if (found == true) { //car in front and slower than us
      // 2.1.2 calculate dv_dt
      s_star = simParameters.s_0 + max(0.0f,
        (trafficPersonVec[p].v * trafficPersonVec[p].T + (trafficPersonVec[p].v *
        delta_v) / (2 * sqrtf(trafficPersonVec[p].a * trafficPersonVec[p].b))));
      thirdTerm =powf(((s_star) / (s)), 2);
      //printf("s_star[%d] = %f\n", p, s_star);
      //printf(">FOUND s_star %f thirdTerm %f!!!!\n",s_star,thirdTerm);
    }

    float dv_dt = trafficPersonVec[p].a * (1.0f - std::pow((
      trafficPersonVec[p].v / trafficPersonVec[p].maxSpeedMperSec), 4) - thirdTerm);

    // 2.1.3 update values
    numMToMove = max(0.0f,
      trafficPersonVec[p].v * deltaTime + 0.5f * (dv_dt) * deltaTime * deltaTime);

    //printf("v %.10f v d %.10f\n",trafficPersonVec[p].v,trafficPersonVec[p].v+((dv_dt/(deltaTime)/deltaTime)));
    trafficPersonVec[p].v += dv_dt * deltaTime;

    if (trafficPersonVec[p].v < 0) {
      //printf("p %d v %f v0 %f a %f dv_dt %f s %f s_star %f MOVE %f\n",p,trafficPersonVec[p].v,trafficPersonVec[p].maxSpeedMperSec,trafficPersonVec[p].a,dv_dt,s,s_star,numMToMove);
      trafficPersonVec[p].v = 0;
      dv_dt = 0.0f;
    }

    //if (p == 87) {
            //printf("%d,%f,%f\n", trafficPersonVec[p].indexPathCurr, trafficPersonVec[p].maxSpeedMperSec, trafficPersonVec[p].v);
            //printf("thirdTerm[%d] = %f\n", p, thirdTerm);
            //printf("a [%d] = %f\n", p, trafficPersonVec[p].a);
            //printf("p = %d\n", p);
            //printf("edge index = %d\n", trafficPersonVec[p].indexPathCurr);
            //printf("speed limit [%d] = %f\n", p, trafficPersonVec[p].maxSpeedMperSec);
            //printf("v [%d] = %f\n", p, trafficPersonVec[p].v);
            //printf("velocity = %f\n", trafficPersonVec[p].v);
            //printf("dv_dt[%d] = %f\n", p, dv_dt);

    //}

    trafficPersonVec[p].cum_v += trafficPersonVec[p].v;
    //printf("vel person %d = %f\n", p, trafficPersonVec[p].cum_v);

    //calculate per edge metrics (velocity, cumulative velocity)
    //edgesData[currentEdge].curr_cum_vel += trafficPersonVec[p].manual_v;
    
    //printf("currentEdge = %u\n, num_cars = %d\n, curr_iter_cum_vel = %f\n, curr_cum_vel = %f\n", currentEdge, edgesData[currentEdge].curr_iter_num_cars, edgesData[currentEdge].curr_iter_cum_vel, edgesData[currentEdge].curr_cum_vel);


    if (calculatePollution && ((float(currentTime) == int(currentTime)))) { // enabled and each second (assuming deltaTime 0.5f)
      // CO Calculation
      const float speedMph = trafficPersonVec[p].v * 2.2369362920544; //mps to mph
      const float coStep = -0.064 + 0.0056 * speedMph + 0.00026 * (speedMph - 50.0f) * (speedMph - 50.0f);

      if (coStep > 0) {
        // coStep *= deltaTime; // we just compute it each second
        trafficPersonVec[p].co += coStep;
      }
      // Gas Consumption
      const float a = dv_dt;
      const float v = trafficPersonVec[p].v; // in mps
      const float Pea = a > 0.0f ? (0.472f*1.680f*a*a*v) : 0.0f;
      const float gasStep = 0.666f + 0.072f*(0.269f*v + 0.000672f*(v*v*v) + 0.0171f*(v*v) + 1.680f*a*v + Pea);
      /*if (p == 0) {
      printf("Time %f --> a %.6f v %.6f\n", currentTime, a, v);
      printf("Time %f --> Consumption %.6f %.6f %.6f %.6f\n", currentTime, (0.269f*v + 0.000672f*(v*v*v)), (0.0171f*(v*v)), 1680.0f*a*v, Pea);
      printf("Time %f --> Consumption %f+0.072*%f --> %f\n\n", currentTime, 0.666f, (0.269f*v + 0.000672f*(v*v*v) + 0.0171f*(v*v) + 1680.0f*a*v + Pea), gasStep);
      }*/
      trafficPersonVec[p].gas += gasStep; // *= deltaTime // we just compute it each second

    }

    //////////////////////////////////////////////

    if (trafficPersonVec[p].v == 0) { //if not moving not do anything else
      ushort posInLineCells = (ushort) (trafficPersonVec[p].posInLaneM);
      //laneMap[mapToWriteShift + maxWidth * (currentEdge + trafficPersonVec[p].numOfLaneInEdge) + posInLineCells] = 0;
      const uint posToSample = mapToWriteShift + kMaxMapWidthM * (currentEdge + (((int) (posInLineCells / kMaxMapWidthM)) * trafficPersonVec[p].edgeNumLanes) + trafficPersonVec[p].numOfLaneInEdge) + posInLineCells % kMaxMapWidthM;
      laneMap[posToSample] = 0;

      return;
    }

    //////////

    ///////////////////////////////
    // COLOR
    trafficPersonVec[p].color = p << 8;
    //if (clientMain->ui.b18RenderSimulationCheckBox->isChecked()) {
    //if(G::global().getInt("cuda_carInfoRendering_type")==0){
    //qsrand(p);

    /*}
    if(G::global().getInt("cuda_carInfoRendering_type")==1){
    uchar c=(uchar)(255*trafficPersonVec[p].v/15.0f);//84m/s is more than 300km/h
    trafficPersonVec[p].color=(c<<24)|(c<<16)|(c<<8);
    }
    if(G::global().getInt("cuda_carInfoRendering_type")==2){
    uchar c=255*trafficPersonVec[p].LC_stateofLaneChanging;
    trafficPersonVec[p].color=(c<<24)|(c<<16)|(c<<8);

    }*/
    //}

    ////////////////////////////////

    // STOP (check if it is a stop if it can go through)

    trafficPersonVec[p].posInLaneM = trafficPersonVec[p].posInLaneM + numMToMove;

    if (trafficPersonVec[p].posInLaneM >
      trafficPersonVec[p].length) { //reach intersection
      numMToMove = trafficPersonVec[p].posInLaneM - trafficPersonVec[p].length;
      getToNextEdge = true;
      trafficPersonVec[p].dist_traveled += trafficPersonVec[p].length;
    } else { //does not reach an intersection
      ////////////////////////////////////////////////////////
      // LANE CHANGING (happens when we are not reached the intersection)
      //printf("first pass\n");
      if (trafficPersonVec[p].v > 3.0f && //at least 10km/h to try to change lane
        trafficPersonVec[p].num_steps % 5 == 0 //just check every (5 steps) 5 seconds
        ) {
        //next thing is not a traffic light
        // skip if there is one lane (avoid to do this)
        // skip if it is the last edge
        if (nextVehicleIsATrafficLight == false &&
          trafficPersonVec[p].edgeNumLanes > 1 && nextEdge != -1) {
          //printf("second pass\n");

          ////////////////////////////////////////////////////
          // LC 1 update lane changing status
          if (trafficPersonVec[p].LC_stateofLaneChanging == 0) {
            // 2.2-exp((x-1)^2)
            float x = trafficPersonVec[p].posInLaneM / trafficPersonVec[p].length;

            if (x > 0.4f) { //just after 40% of the road
              float probabiltyMandatoryState = 2.2 - exp((x - 1) * (x - 1));

              //if (((float) qrand() / RAND_MAX) < probabiltyMandatoryState) {
              if ((((int) (x * 100) % 100) / 100.0f) < probabiltyMandatoryState) { // pseudo random number
                trafficPersonVec[p].LC_stateofLaneChanging = 1;
              }
            }

          }

          ////////////////////////////////////////////////////
          // LC 2 NOT MANDATORY STATE
          if (trafficPersonVec[p].LC_stateofLaneChanging == 0) {
            //if(p==40)printf("LC v %f v0 %f a %f\n",trafficPersonVec[p].v,trafficPersonVec[p].maxSpeedMperSec*0.5f,dv_dt);
            // discretionary change: v slower than the current road limit and deccelerating and moving
            if ((trafficPersonVec[p].v < (trafficPersonVec[p].maxSpeedMperSec * 0.7f)) &&
              (dv_dt < 0) && trafficPersonVec[p].v > 3.0f) {
              //printf(">>LANE CHANGE\n");

              //printf("LC 0 %u\n",trafficPersonVec[p].numOfLaneInEdge);
              bool leftLane = trafficPersonVec[p].numOfLaneInEdge >
                0; //at least one lane on the left
              bool rightLane = trafficPersonVec[p].numOfLaneInEdge <
                trafficPersonVec[p].edgeNumLanes - 1; //at least one lane

              if (leftLane == true && rightLane == true) {
                if (int(trafficPersonVec[p].v) % 2 == 0) { // pseudo random
                  leftLane = false;
                } else {
                  rightLane = false;
                }
              }
              ushort laneToCheck;
              if (leftLane == true) {
                laneToCheck = trafficPersonVec[p].numOfLaneInEdge - 1;
              } else {
                laneToCheck = trafficPersonVec[p].numOfLaneInEdge + 1;
              }

              uchar v_a, v_b;
              float gap_a, gap_b;
              //printf("p %u LC 1 %u\n",p,laneToCheck);
              uchar trafficLightState = trafficLights[currentEdge +
                trafficPersonVec[p].numOfLaneInEdge];
              calculateGapsLC(mapToReadShift, laneMap, trafficLightState,
                currentEdge + laneToCheck, trafficPersonVec[p].edgeNumLanes, trafficPersonVec[p].posInLaneM,
                trafficPersonVec[p].length, v_a, v_b, gap_a, gap_b);

              //printf("LC 2 %u %u %f %f\n",v_a,v_b,gap_a,gap_b);
              if (gap_a == 1000.0f && gap_b == 1000.0f) { //lag and lead car very far
                trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE

              } else { // NOT ALONE
                float b1A = 0.05f, b2A = 0.15f;
                float b1B = 0.15f, b2B = 0.40f;
                // simParameters.s_0-> critical lead gap
                float g_na_D, g_bn_D;
                bool acceptLC = true;

                if (gap_a != 1000.0f) {
                  g_na_D = max(simParameters.s_0, simParameters.s_0 + b1A * trafficPersonVec[p].v + b2A *
                    (trafficPersonVec[p].v - v_a * 3.0f));

                  if (gap_a < g_na_D) { //gap smaller than critical gap
                    acceptLC = false;
                  }
                }

                if (acceptLC == true && gap_b != 1000.0f) {
                  g_bn_D = max(simParameters.s_0, simParameters.s_0 + b1B * v_b * 3.0f + b2B * (v_b * 3.0f - trafficPersonVec[p].v));

                  if (gap_b < g_bn_D) { //gap smaller than critical gap
                    acceptLC = false;
                  }
                }

                if (acceptLC == true) {
                  trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
                }
              }

              //printf("<<LANE CHANGE\n");
            }


          }// Discretionary

          ////////////////////////////////////////////////////
          // LC 3 *MANDATORY* STATE
          if (trafficPersonVec[p].LC_stateofLaneChanging == 1) {
          //printf("state of lange changing = mandatory\n");
            // LC 3.1 Calculate the correct lanes
            if (trafficPersonVec[p].LC_endOKLanes == 0xFF) {
  //printf("currentEdge = %u, nextEdge = %u, edgeNextInters = %u, edgeNumLanes = %u\n", currentEdge, nextEdge, trafficPersonVec[p].edgeNextInters, trafficPersonVec[p].edgeNumLanes);
              calculateLaneCarShouldBe(currentEdge, nextEdge, intersections,
                trafficPersonVec[p].edgeNextInters, trafficPersonVec[p].edgeNumLanes,
                trafficPersonVec[p].LC_initOKLanes, trafficPersonVec[p].LC_endOKLanes);

              //printf("p%u num lanes %u min %u max %u\n",p,trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);
              if (trafficPersonVec[p].LC_initOKLanes == 0 &&
                trafficPersonVec[p].LC_endOKLanes == 0) {
                //exit(0);
              }
            }


            //printf(">>LANE CHANGE\n");
            //printf("LC 0 %u\n",trafficPersonVec[p].numOfLaneInEdge);
            bool leftLane = false, rightLane = false;

            // LC 3.2 CORRECT LANES--> DICRETIONARY LC WITHIN
            if (trafficPersonVec[p].numOfLaneInEdge >= trafficPersonVec[p].LC_initOKLanes &&
              trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].LC_endOKLanes) {
              // for discretionary it should be under some circustances
              if ((trafficPersonVec[p].v < (trafficPersonVec[p].maxSpeedMperSec * 0.7f)) &&
                (dv_dt < 0) && trafficPersonVec[p].v > 3.0f) {
                leftLane =
                  (trafficPersonVec[p].numOfLaneInEdge > 0) && //at least one lane on the left
                  (trafficPersonVec[p].numOfLaneInEdge - 1 >= trafficPersonVec[p].LC_initOKLanes)
                  &&
                  (trafficPersonVec[p].numOfLaneInEdge - 1 < trafficPersonVec[p].LC_endOKLanes);
                rightLane =
                  (trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].edgeNumLanes - 1) &&
                  //at least one lane
                  (trafficPersonVec[p].numOfLaneInEdge + 1 >= trafficPersonVec[p].LC_initOKLanes)
                  &&
                  (trafficPersonVec[p].numOfLaneInEdge + 1 < trafficPersonVec[p].LC_endOKLanes);
                //printf("D\n");
              }
            }
            // LC 3.3 INCORRECT LANES--> MANDATORY LC
            else {
              //printf("num lanes %u min %u max %u\n",trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);
              //printf("p%u num lanes %u min %u max %u\n",p,trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);

              if (trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].LC_initOKLanes) {
                rightLane = true;
              } else {
                leftLane = true;
              }

              if (rightLane == true &&
                trafficPersonVec[p].numOfLaneInEdge + 1 >= trafficPersonVec[p].edgeNumLanes) {
                printf("ERROR: RT laneToCheck>=trafficPersonVec[p].edgeNumLanes\n");
              }

              if (leftLane == true && trafficPersonVec[p].numOfLaneInEdge == 0) {
                printf("ERROR %u: LT laneToCheck>=trafficPersonVec[p].edgeNumLanes OK %u-%u NE %u\n",
                  p, trafficPersonVec[p].LC_initOKLanes, trafficPersonVec[p].LC_endOKLanes,
                  nextEdge);
                //exit(0);
              }

              //printf("M L %d R %d nL %u\n",leftLane,rightLane,trafficPersonVec[p].numOfLaneInEdge);
            }

            if (leftLane == true || rightLane == true) {

              // choose lane (if necessary)
              if (leftLane == true && rightLane == true) {
                if ((int) (trafficPersonVec[p].posInLaneM) % 2 == 0) { //pseudo random
                  leftLane = false;
                } else {
                  rightLane = false;
                }
              }
              ushort laneToCheck;
              if (leftLane == true) {
                laneToCheck = trafficPersonVec[p].numOfLaneInEdge - 1;
              } else {
                laneToCheck = trafficPersonVec[p].numOfLaneInEdge + 1;
              }

              if (laneToCheck >= trafficPersonVec[p].edgeNumLanes) {
                printf("ERROR: laneToCheck>=trafficPersonVec[p].edgeNumLanes %u %u\n",
                  laneToCheck, trafficPersonVec[p].edgeNumLanes);
              }

              uchar v_a, v_b;
              float gap_a, gap_b;
              //printf("p %u LC 1 %u\n",p,laneToCheck);
              uchar trafficLightState = trafficLights[currentEdge +
                trafficPersonVec[p].numOfLaneInEdge];
              calculateGapsLC(mapToReadShift, laneMap, trafficLightState,
                currentEdge + laneToCheck, trafficPersonVec[p].edgeNumLanes, trafficPersonVec[p].posInLaneM,
                trafficPersonVec[p].length, v_a, v_b, gap_a, gap_b);

              //printf("LC 2 %u %u %f %f\n",v_a,v_b,gap_a,gap_b);
              if (gap_a == 1000.0f && gap_b == 1000.0f) { //lag and lead car very far
                trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE

              } else { // NOT ALONE
                float b1A = 0.05f, b2A = 0.15f;
                float b1B = 0.15f, b2B = 0.40f;
                float gamma = 0.000025;
                // simParameters.s_0-> critical lead gap
                float distEnd = trafficPersonVec[p].length - trafficPersonVec[p].posInLaneM;
                float expTerm = (1 - exp(-gamma * distEnd * distEnd));

                float g_na_M, g_bn_M;
                bool acceptLC = true;

                if (gap_a != 1000.0f) {
                  g_na_M = max(simParameters.s_0, simParameters.s_0 + (b1A * trafficPersonVec[p].v + b2A *
                    (trafficPersonVec[p].v - v_a * 3.0f)));

                  if (gap_a < g_na_M) { //gap smaller than critical gap
                    acceptLC = false;
                  }
                }

                if (acceptLC == true && gap_b != 1000.0f) {
                  g_bn_M = max(simParameters.s_0, simParameters.s_0 + (b1B * v_b * 3.0f + b2B * (v_b * 3.0f -
                    trafficPersonVec[p].v)));

                  if (gap_b < g_bn_M) { //gap smaller than critical gap
                    acceptLC = false;
                  }
                }

                if (acceptLC == true) {
                  trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
                }
              }


            }

          }// Mandatory

        }//at least two lanes and not stopped by traffic light

      }

      ///////////////////////////////////////////////////////

      uchar vInMpS = (uchar) (trafficPersonVec[p].v * 3); //speed in m/s to fit in uchar
      ushort posInLineCells = (ushort) (trafficPersonVec[p].posInLaneM);
      //laneMap[mapToWriteShift + maxWidth * (currentEdge + trafficPersonVec[p].numOfLaneInEdge) + posInLineCells] = vInMpS;
      //printf("numeoflaneinedge %d calculated edge %d\n", trafficPersonVec[p].numOfLaneInEdge, (currentEdge + (((int) (posInLineCells / kMaxMapWidthM)) * trafficPersonVec[p].edgeNumLanes) + trafficPersonVec[p].numOfLaneInEdge));
      const uint posToSample = mapToWriteShift + kMaxMapWidthM * (currentEdge + (((int) (posInLineCells / kMaxMapWidthM)) * trafficPersonVec[p].edgeNumLanes) + trafficPersonVec[p].numOfLaneInEdge) + posInLineCells % kMaxMapWidthM;
      laneMap[posToSample] = vInMpS;
      //printf("2<<LANE CHANGE\n");
      return;
    }

    //2.2 close to intersection

    //2.2 check if change intersection
    //!!!ALWAYS CHANGE
    //2.2.1 find next edge
    /*ushort curr_intersection=trafficPersonVec[p].edgeNextInters;
    ushort end_intersection=trafficPersonVec[p].end_intersection;
    //2.1 check if end*/
    if (nextEdge == -1) { //if(curr_intersection==end_intersection)
      trafficPersonVec[p].active = 2; //finished
      return;
    }

    //if(trafficPersonVec[p].nextPathEdge>=nextEdgeM.size())printf("AAAAAAAAAAAAAAAAA\n");
    /////////////
    // update edge
    /*// stop
    if(noFirstInLane==false&&trafficLights[currentEdge+trafficPersonVec[p].numOfLaneInEdge]==0x0F){
    // first in lane and stop--> update to avoid to pass another car
    trafficLights[currentEdge+trafficPersonVec[p].numOfLaneInEdge]=0x00;
    }*/
    //trafficPersonVec[p].curEdgeLane=trafficPersonVec[p].nextEdge;
    trafficPersonVec[p].indexPathCurr++;
    trafficPersonVec[p].maxSpeedMperSec = trafficPersonVec[p].nextEdgemaxSpeedMperSec;
    trafficPersonVec[p].edgeNumLanes = trafficPersonVec[p].nextEdgeNumLanes;
    trafficPersonVec[p].edgeNextInters = trafficPersonVec[p].nextEdgeNextInters;
    trafficPersonVec[p].length = trafficPersonVec[p].nextEdgeLength;
    trafficPersonVec[p].posInLaneM = numMToMove;

    if (trafficPersonVec[p].numOfLaneInEdge >= trafficPersonVec[p].edgeNumLanes) {
      trafficPersonVec[p].numOfLaneInEdge = trafficPersonVec[p].edgeNumLanes - 1; //change line if there are less roads
    }

    ////////////
    // update next edge
    uint nextNEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];

    //trafficPersonVec[p].nextEdge=nextEdge;
    if (nextNEdge != -1) {
      //trafficPersonVec[p].nextPathEdge++;
      trafficPersonVec[p].LC_initOKLanes = 0xFF;
      trafficPersonVec[p].LC_endOKLanes = 0xFF;

      //2.2.3 update person edgeData
      //trafficPersonVec[p].nextEdge=nextEdge;
      trafficPersonVec[p].nextEdgemaxSpeedMperSec =
        edgesData[nextNEdge].maxSpeedMperSec;
      trafficPersonVec[p].nextEdgeNumLanes = edgesData[nextNEdge].numLines;
      trafficPersonVec[p].nextEdgeNextInters = edgesData[nextNEdge].nextIntersMapped;
      trafficPersonVec[p].nextEdgeLength = edgesData[nextNEdge].length;
    }

    trafficPersonVec[p].LC_stateofLaneChanging = 0;
    uchar vInMpS = (uchar) (trafficPersonVec[p].v * 3); //speed in m/s to fit in uchar
    ushort posInLineCells = (ushort) (trafficPersonVec[p].posInLaneM);

    // laneMap[mapToWriteShift + maxWidth * (nextEdge + trafficPersonVec[p].numOfLaneInEdge) + posInLineCells] = vInMpS;
    const uint posToSample = mapToWriteShift + kMaxMapWidthM * (nextEdge + (((int) (posInLineCells / kMaxMapWidthM)) * trafficPersonVec[p].edgeNumLanes) + trafficPersonVec[p].numOfLaneInEdge) + posInLineCells % kMaxMapWidthM;  // note the last % should not happen
    laneMap[posToSample] = vInMpS;
  }
}//

/*
__global__ void kernel_intersectionSTOPSimulation(
     uint numIntersections, 
     float currentTime, 
     LC::B18IntersectionData *intersections, 
     uchar *trafficLights,
     LC::B18EdgeData* edgesData,//for the length
     uchar* laneMap,//to check if there are cars
     uint mapToReadShift) {
     int i = blockIdx.x * blockDim.x + threadIdx.x;
     if (i<numIntersections) {//CUDA check (inside margins)

     const float deltaEvent = 0.0f; 

     //if(i==0)printf("i %d\n",i);
     if (currentTime > intersections[i].nextEvent && intersections[i].totalInOutEdges > 0) {
       uint edgeOT = intersections[i].edge[intersections[i].state];
       uchar numLinesO = edgeOT >> 24;
       uint edgeONum = edgeOT & kMaskLaneMap; // 0xFFFFF

       // red old traffic lights
       for (int nL = 0; nL < numLinesO; nL++) {
         trafficLights[edgeONum + nL] = 0x00; //red old traffic light
       }

       for (int iN = 0; iN <= intersections[i].totalInOutEdges + 1; iN++) { //to give a round
         intersections[i].state = (intersections[i].state + 1) %
           intersections[i].totalInOutEdges;//next light

         if ((intersections[i].edge[intersections[i].state] & kMaskInEdge) == kMaskInEdge) {  // 0x800000
           uint edgeIT = intersections[i].edge[intersections[i].state];
           uint edgeINum = edgeIT & kMaskLaneMap; //get edgeI 0xFFFFF
           uchar numLinesI = edgeIT >> 24;
           /// check if someone in this edge
           int rangeToCheck = 5.0f; //5m
           ushort firstPosToCheck = edgesData[edgeINum].length - intersectionClearance; //last po
           bool atLeastOneStopped = false;

           for (int posCheck = firstPosToCheck; rangeToCheck >= 0 && posCheck >= 0; posCheck--, rangeToCheck--) { //as many cells as the rangeToCheck says
             for (int nL = 0; nL < numLinesI; nL++) {
               //int cellNum = mapToReadShift + maxWidth * (edgeINum + nL) + posCheck;
               const uint posToSample = mapToReadShift + kMaxMapWidthM * (edgeINum + (((int) (posCheck / kMaxMapWidthM)) * numLinesI) + nL) + posCheck % kMaxMapWidthM;


               if (laneMap[posToSample] == 0) { //car stopped
                 trafficLights[edgeINum + nL] = 0x0F; // STOP SIGN 0x0F--> Let pass
                 atLeastOneStopped = true;
               }
             }
           }

           if (atLeastOneStopped == true) {
             intersections[i].nextEvent = currentTime + deltaEvent; //just move forward time if changed (otherwise check in next iteration)
             break;
           }
         }
       }
     }
     ///
   }
   
}//
*/

__global__ void kernel_intersectionOneSimulation(
      uint numIntersections,
      float currentTime,
      LC::B18IntersectionData *intersections,
      uchar *trafficLights) {

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<numIntersections){//CUDA check (inside margins)
    const float deltaEvent = 20.0f; /// !!!!
    if (currentTime > intersections[i].nextEvent && intersections[i].totalInOutEdges > 0) {

      uint edgeOT = intersections[i].edge[intersections[i].state];
      uchar numLinesO = edgeOT >> 24;
      uint edgeONum = edgeOT & kMaskLaneMap; // 0xFFFFF;

      // red old traffic lights
      if ((edgeOT&kMaskInEdge) == kMaskInEdge) {  // Just do it if we were in in
        for (int nL = 0; nL < numLinesO; nL++) {
          trafficLights[edgeONum + nL] = 0x00; //red old traffic light
        }
      }

      for (int iN = 0; iN <= intersections[i].totalInOutEdges + 1; iN++) { //to give a round
        intersections[i].state = (intersections[i].state + 1) % intersections[i].totalInOutEdges;//next light

        if ((intersections[i].edge[intersections[i].state] & kMaskInEdge) == kMaskInEdge) {  // 0x800000
          // green new traffic lights
          uint edgeIT = intersections[i].edge[intersections[i].state];
          uint edgeINum = edgeIT & kMaskLaneMap; //  0xFFFFF; //get edgeI
          uchar numLinesI = edgeIT >> 24;

          for (int nL = 0; nL < numLinesI; nL++) {
            trafficLights[edgeINum + nL] = 0xFF;
          }

          //trafficLights[edgeINum]=0xFF;
          break;
        }
      }//green new traffic light

      intersections[i].nextEvent = currentTime + deltaEvent;
    }
    //////////////////////////////////////////////////////
  }
   
 }//

// Kernel that executes on the CUDA device
__global__ void kernel_sampleTraffic(
  int numPeople,
  LC::B18TrafficPerson *trafficPersonVec,
  uint *indexPathVec,
  float *accSpeedPerLinePerTimeInterval,
  float *numVehPerLinePerTimeInterval, //this could have been int
  uint offset)
  {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p < numPeople) {//CUDA check (inside margins)
    if (trafficPersonVec[p].active == 1) { // just active
      int edgeNum = indexPathVec[trafficPersonVec[p].indexPathCurr];
      accSpeedPerLinePerTimeInterval[edgeNum + offset] += trafficPersonVec[p].v / 3.0f;
      numVehPerLinePerTimeInterval[edgeNum + offset]++;
    }
  }
}
__global__ void kernel_resetPeople(
  int numPeople,
  LC::B18TrafficPerson *trafficPersonVec) {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p < numPeople) {//CUDA check (inside margins)
    trafficPersonVec[p].active = 0;
  }
}

void b18GetSampleTrafficCUDA(std::vector<float>& accSpeedPerLinePerTimeInterval, std::vector<float>& numVehPerLinePerTimeInterval) {
  // copy back people
  size_t size = accSpeedPerLinePerTimeInterval.size() * sizeof(float);
  hipMemcpy(accSpeedPerLinePerTimeInterval.data(), accSpeedPerLinePerTimeInterval_d, size, hipMemcpyDeviceToHost);

  size_t sizeI = numVehPerLinePerTimeInterval.size() * sizeof(uchar);
  hipMemcpy(numVehPerLinePerTimeInterval.data(), numVehPerLinePerTimeInterval_d, sizeI, hipMemcpyDeviceToHost);
}

void b18ResetPeopleLanesCUDA(uint numPeople) {
  kernel_resetPeople << < ceil(numPeople / 1024.0f), 1024 >> > (numPeople, trafficPersonVec_d);
  hipMemset(&laneMap_d[0], -1, halfLaneMap*sizeof(unsigned char));
  hipMemset(&laneMap_d[halfLaneMap], -1, halfLaneMap*sizeof(unsigned char));
}

void b18SimulateTrafficCUDA(float currentTime,
  uint numPeople,
  uint numIntersections,
  float deltaTime,
  const parameters simParameters,
  int numBlocks,
  int threadsPerBlock) {
  intersectionBench.startMeasuring();
  const uint numStepsTogether = 12; //change also in density (10 per hour)
  ////////////////////////////////////////////////////////////
  // 1. CHANGE MAP: set map to use and clean the other
  if(readFirstMapC==true){
    mapToReadShift=0;
    mapToWriteShift=halfLaneMap;
    gpuErrchk(hipMemset(&laneMap_d[halfLaneMap], -1, halfLaneMap*sizeof(unsigned char)));//clean second half
  }else{
    mapToReadShift=halfLaneMap;
    mapToWriteShift=0;
    gpuErrchk(hipMemset(&laneMap_d[0], -1, halfLaneMap*sizeof(unsigned char)));//clean first half
  }
  readFirstMapC=!readFirstMapC;//next iteration invert use

  // Simulate intersections.
  kernel_intersectionOneSimulation << < ceil(numIntersections / 512.0f), 512 >> > (numIntersections, currentTime, intersections_d, trafficLights_d);
  gpuErrchk(hipPeekAtLastError());

  intersectionBench.stopMeasuring();
  
  peopleBench.startMeasuring();
  // Simulate people.
  kernel_trafficSimulation <<< numBlocks, threadsPerBlock>> > (numPeople, currentTime, mapToReadShift, mapToWriteShift, trafficPersonVec_d, indexPathVec_d, edgesData_d, laneMap_d, intersections_d, trafficLights_d, deltaTime, simParameters);
  gpuErrchk(hipPeekAtLastError());
  peopleBench.stopMeasuring();

  // Sample if necessary.
  if ((((float) ((int) currentTime)) == (currentTime)) &&
    ((int) currentTime % ((int) 30)) == 0) { //3min //(sample double each 3min)
    int samplingNumber = (currentTime - startTime) / (30 * numStepsTogether);
    uint offset = numIntersections * samplingNumber;
    //printf("Sample %d\n", samplingNumber);
    kernel_sampleTraffic << < ceil(numPeople / 1024.0f), 1024 >> > (numPeople, trafficPersonVec_d, indexPathVec_d, accSpeedPerLinePerTimeInterval_d, numVehPerLinePerTimeInterval_d, offset);
    gpuErrchk(hipPeekAtLastError());
  }
}//
