#include "hip/hip_runtime.h"
//CUDA CODE
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""

#include "b18TrafficPerson.h"
#include "b18EdgeData.h"
#include <vector>
#include <iostream>

#ifndef ushort
#define ushort uint16_t
#endif
#ifndef uint
#define uint uint32_t
#endif
#ifndef uchar
#define uchar uint8_t
#endif

///////////////////////////////
// CONSTANTS

__constant__ float intersectionClearance = 7.8f;
// `s_0` refers to the minimum spacing distance used in the Intelligent Driver Model (IDM)
__constant__ float s_0 = 7.0f;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}
inline void printMemoryUsage() {
  // show memory usage of GPU
  size_t free_byte;
  size_t total_byte;
  hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);
  if (hipSuccess != cuda_status) {
    printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status));
    exit(1);
  }
  double free_db = (double) free_byte;
  double total_db = (double) total_byte;
  double used_db = total_db - free_db;
  printf("GPU memory usage: used = %.0f, free = %.0f MB, total = %.0f MB\n", used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}
////////////////////////////////
// VARIABLES
LC::B18TrafficPerson *trafficPersonVec_d;
uint *indexPathVec_d;
LC::B18EdgeData *edgesData_d;

__constant__ bool calculatePollution = true;
__constant__ float cellSize = 1.0f;

__constant__ float deltaTime = 0.5f;
const float deltaTimeH = 0.5f;

const uint numStepsPerSample = 30.0f / deltaTimeH; //each min
const uint numStepsTogether = 12; //change also in density (10 per hour)

uchar *laneMap_d;
bool readFirstMapC=true;
uint mapToReadShift;
uint mapToWriteShift;
uint halfLaneMap;
float startTime;


LC::B18IntersectionData *intersections_d;
uchar *trafficLights_d;

float* accSpeedPerLinePerTimeInterval_d;
float* numVehPerLinePerTimeInterval_d;

LC::ConnectionsInfo *deviceConnections;
size_t amountOfConnections;

void b18InitCUDA(
    bool fistInitialization,
    std::vector<LC::B18TrafficPerson>& trafficPersonVec, 
    std::vector<uint> &indexPathVec, 
    std::vector<LC::B18EdgeData>& edgesData, 
    std::vector<uchar>& laneMap, 
    std::vector<uchar>& trafficLights, 
    std::vector<LC::B18IntersectionData>& intersections,
    float startTimeH, float endTimeH,
    std::vector<float>& accSpeedPerLinePerTimeInterval,
    std::vector<float>& numVehPerLinePerTimeInterval,
    const std::vector<LC::ConnectionsInfo> & hostConnections) {

  
  printf("Starting lane info\n");
  for (const auto & lane : hostConnections) {
    printf("{in: %d, out: %d, on: %d}", lane.in_lane_number, lane.out_lane_number, lane.enabled);
  }
  { // people
    amountOfConnections = hostConnections.size();
    size_t size = hostConnections.size() * sizeof(LC::ConnectionsInfo);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &deviceConnections, size));   // Allocate array on device
    gpuErrchk(hipMemcpy(deviceConnections, hostConnections.data(), size, hipMemcpyHostToDevice));
  }

  printMemoryUsage();
  { // people
    size_t size = trafficPersonVec.size() * sizeof(LC::B18TrafficPerson);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &trafficPersonVec_d, size));   // Allocate array on device
    gpuErrchk(hipMemcpy(trafficPersonVec_d, trafficPersonVec.data(), size, hipMemcpyHostToDevice));
  }
  
  { // indexPathVec
    size_t sizeIn = indexPathVec.size() * sizeof(uint);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &indexPathVec_d, sizeIn));   // Allocate array on device
    gpuErrchk(hipMemcpy(indexPathVec_d, indexPathVec.data(), sizeIn, hipMemcpyHostToDevice));
  }
  {//edgeData
    size_t sizeD = edgesData.size() * sizeof(LC::B18EdgeData);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &edgesData_d, sizeD));   // Allocate array on device
    gpuErrchk(hipMemcpy(edgesData_d, edgesData.data(), sizeD, hipMemcpyHostToDevice));
  }
  {//laneMap
    size_t sizeL = laneMap.size() * sizeof(uchar);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &laneMap_d, sizeL));   // Allocate array on device
    gpuErrchk(hipMemcpy(laneMap_d, laneMap.data(), sizeL, hipMemcpyHostToDevice));
    halfLaneMap = laneMap.size() / 2;
  }
  {// intersections
    size_t sizeI = intersections.size() * sizeof(LC::B18IntersectionData);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &intersections_d, sizeI));   // Allocate array on device
    gpuErrchk(hipMemcpy(intersections_d, intersections.data(), sizeI, hipMemcpyHostToDevice));
    size_t sizeT = trafficLights.size() * sizeof(uchar);//total number of lanes
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &trafficLights_d, sizeT));   // Allocate array on device
    gpuErrchk(hipMemcpy(trafficLights_d, trafficLights.data(), sizeT, hipMemcpyHostToDevice));
  }
  {
    startTime = startTimeH * 3600.0f;
    uint numSamples = ceil(((endTimeH*3600.0f - startTimeH*3600.0f) / (deltaTimeH * numStepsPerSample * numStepsTogether))) + 1; //!!!
    accSpeedPerLinePerTimeInterval.clear();
    numVehPerLinePerTimeInterval.clear();
    accSpeedPerLinePerTimeInterval.resize(numSamples * trafficLights.size());
    numVehPerLinePerTimeInterval.resize(numSamples * trafficLights.size());
    size_t sizeAcc = accSpeedPerLinePerTimeInterval.size() * sizeof(float);
    if (fistInitialization)gpuErrchk(hipMalloc((void **) &accSpeedPerLinePerTimeInterval_d, sizeAcc));   // Allocate array on device
    if (fistInitialization)gpuErrchk(hipMalloc((void **) &numVehPerLinePerTimeInterval_d, sizeAcc));   // Allocate array on device
    gpuErrchk(hipMemset(&accSpeedPerLinePerTimeInterval_d[0], 0, sizeAcc));
    gpuErrchk(hipMemset(&numVehPerLinePerTimeInterval_d[0], 0, sizeAcc));
  }
  printMemoryUsage();
}//

void b18FinishCUDA(void){
  //////////////////////////////
  // FINISH
  hipFree(trafficPersonVec_d);
  hipFree(indexPathVec_d);
  hipFree(edgesData_d);
  hipFree(laneMap_d);
  hipFree(intersections_d);
  hipFree(trafficLights_d);

  hipFree(accSpeedPerLinePerTimeInterval_d);
  hipFree(numVehPerLinePerTimeInterval_d);
}//

 void b18GetDataCUDA(std::vector<LC::B18TrafficPerson>& trafficPersonVec){
   // copy back people
   size_t size = trafficPersonVec.size() * sizeof(LC::B18TrafficPerson);
   hipMemcpy(trafficPersonVec.data(),trafficPersonVec_d,size,hipMemcpyDeviceToHost);//hipMemcpyHostToDevice
 }


 __device__ void calculateGapsLC(
   uint mapToReadShift,
   uchar* laneMap,
   uchar trafficLightState,
   uint laneToCheck,
   ushort numLinesEdge,
   float posInMToCheck,
   float length,
   uchar &v_a,
   uchar &v_b,
   float &gap_a,
   float &gap_b) {
   ushort numOfCells = ceil(length);
   ushort initShift = ceil(posInMToCheck);
   bool found = false;

   // CHECK FORWARD
   //printf("initShift %u numOfCells %u\n",initShift,numOfCells);
   for (ushort b = initShift - 1; (b < numOfCells) && (found == false); b++) { //NOTE -1 to make sure there is none in at the same level
     // laneChar = laneMap[mapToReadShift + maxWidth * (laneToCheck) + b];
     const uint posToSample = mapToReadShift + kMaxMapWidthM * (laneToCheck + (((int) (b / kMaxMapWidthM)) * numLinesEdge)) + b % kMaxMapWidthM;
     const uchar laneChar = laneMap[posToSample];

     if (laneChar != 0xFF) {
       gap_a = ((float) b - initShift); //m
       v_a = laneChar; //laneChar is in 3*ms (to save space in array)
       found = true;
       break;
     }
   }

   if (found == false) {
     if (trafficLightState == 0x00) { //red
       //found=true;
       gap_a = gap_b = 1000.0f; //force to change to the line without vehicle
       v_a = v_b = 0xFF;
       return;
     }
   }

   if (found == false) {
     gap_a = 1000.0f;
   }

   // CHECK BACKWARDS
   found = false;

   //printf("2initShift %u numOfCells %u\n",initShift,numOfCells);
   for (int b = initShift + 1; (b >= 0) && (found == false); b--) {  // NOTE +1 to make sure there is none in at the same level
     const uint posToSample = mapToReadShift + kMaxMapWidthM * (laneToCheck + (((int) (b / kMaxMapWidthM)) * numLinesEdge)) + b % kMaxMapWidthM;
     const uchar laneChar = laneMap[posToSample];
     if (laneChar != 0xFF) {
       gap_b = ((float) initShift - b); //m
       v_b = laneChar; //laneChar is in 3*ms (to save space in array)
       found = true;
       break;
     }
   }

   //printf("3initShift %u numOfCells %u\n",initShift,numOfCells);
   if (found == false) {
     gap_b = 1000.0f;
   }

  }//

 __device__ void calculateLaneCarShouldBe(
   uint curEdgeLane,
   uint nextEdge,
   LC::B18IntersectionData* intersections,
   uint edgeNextInters,
   ushort edgeNumLanes,
   ushort &initOKLanes,
   ushort &endOKLanes) {
   initOKLanes = 0;
   endOKLanes = edgeNumLanes;
   bool currentEdgeFound = false;
   bool exitFound = false;
   ushort numExitToTake = 0;
   ushort numExists = 0;

   for (int eN = intersections[edgeNextInters].totalInOutEdges - 1; eN >= 0; eN--) {  // clockwise
     uint procEdge = intersections[edgeNextInters].edge[eN];

     if ((procEdge & kMaskLaneMap) == curEdgeLane) { //current edge 0xFFFFF
       currentEdgeFound = true;
       if (exitFound == false) {
         numExitToTake = 0;
       }
       continue;
     }

     if ((procEdge & kMaskInEdge) == 0x0) { //out edge 0x800000
       numExists++;
       if (currentEdgeFound == true) {
         numExitToTake++;
       }
       if (currentEdgeFound == false && exitFound == false) {
         numExitToTake++;
       }
     }
     if ((procEdge & kMaskInEdge) == nextEdge) {
       exitFound = true;
       currentEdgeFound = false;
     }
   }

   if (edgeNumLanes == 0) {
     printf("ERRRROR\n");
   }

   switch (edgeNumLanes) {
     /// ONE LANE
   case 1:
     initOKLanes = 0;
     endOKLanes = 1;
     break;

     /// TWO LANE
   case 2:
     switch (numExists) {
     case 1:
     case 2://all okay
       initOKLanes = 0;
       endOKLanes = 2;
       break;

     case 3:
       if (numExitToTake > 2) { //left
         initOKLanes = 0;
         endOKLanes = 1;
         break;
       }

       initOKLanes = 1;
       endOKLanes = 2;
       break;

     default:

       if (numExitToTake >= numExists - 1) {
         initOKLanes = 0;
         endOKLanes = 1;
         break;
       }

       initOKLanes = 1;
       endOKLanes = 2;
       break;
     }

     break;

     /// THREE LANE
   case 3:
     switch (numExists) {
     case 1:
     case 2://all okay
       initOKLanes = 0;
       endOKLanes = 3;
       break;

     case 3:
       if (numExitToTake > 2) { //left
         initOKLanes = 0;
         endOKLanes = 1;
         break;
       }

       initOKLanes = 1;
       endOKLanes = 3;
       break;

     default:
       if (numExitToTake >= numExists - 1) {
         initOKLanes = 0;
         endOKLanes = 1;
         break;
       }

       initOKLanes = 1;
       endOKLanes = 2;
       break;
     }

     break;

   case 4:
     switch (numExists) {
     case 1:
     case 2://all okay
       initOKLanes = 0;
       endOKLanes = 4;
       break;

     case 3:
       if (numExitToTake == 1) { //right
         initOKLanes = 3;
         endOKLanes = 4;
       }

       if (numExitToTake > 3) { //left
         initOKLanes = 0;
         endOKLanes = 1;
         break;
       }

       initOKLanes = 1;
       endOKLanes = 4;
       break;

     default:
       if (numExitToTake == 1) { //right
         initOKLanes = edgeNumLanes - 1;
         endOKLanes = edgeNumLanes;
       }

       if (numExitToTake >= numExists - 2) {
         initOKLanes = 0;
         endOKLanes = 2;
         break;
       }

       initOKLanes = 1; //also lane 2
       endOKLanes = edgeNumLanes;
     }

     break;

   default:
     switch (numExists) {
     case 1:
     case 2://all okay
       initOKLanes = 0;
       endOKLanes = edgeNumLanes;
       break;

     case 3:
       if (numExitToTake == 1) { //right
         initOKLanes = edgeNumLanes - 1;
         endOKLanes = edgeNumLanes;
       }

       if (numExitToTake > edgeNumLanes - 2) { //left
         initOKLanes = 0;
         endOKLanes = 2;
         break;
       }

       initOKLanes = 1;
       endOKLanes = edgeNumLanes;
       break;

     default:
       if (numExitToTake < 2) { //right
         initOKLanes = edgeNumLanes - 2;
         endOKLanes = edgeNumLanes;
       }

       if (numExitToTake >= numExists - 2) {
         initOKLanes = 0;
         endOKLanes = 2;
         break;
       }

       initOKLanes = 1; //also lane 2
       endOKLanes = edgeNumLanes - 1;
     }

     break;
   }
  }//

 // Kernel that executes on the CUDA device
__global__ void kernel_trafficSimulation(
   const int numPeople,
   float currentTime,
   uint mapToReadShift,
   uint mapToWriteShift,
   LC::B18TrafficPerson *trafficPersonVec,
   uint *indexPathVec,
   LC::B18EdgeData* edgesData,
   uchar *laneMap,
   LC::B18IntersectionData *intersections,
   uchar *trafficLights,
   LC::ConnectionsInfo *connections,
   size_t amountOfConnections
   )
 {
   const int p = blockIdx.x * blockDim.x + threadIdx.x;
   // Only proceed if the computed index `p` is valid
   if (p < numPeople) {
     if (trafficPersonVec[p].active == 2) {
       // Return if this person has reached its destiny
       return;
     }

     if (trafficPersonVec[p].active == 0) {
       // Check what should be done if the persone is still inactive
       if (trafficPersonVec[p].time_departure > currentTime) {
         // Return if it's not yet the time for this person
         return;
       }
       else {
         // Else initialize this person's data
         trafficPersonVec[p].indexPathCurr = trafficPersonVec[p].indexPathInit;
         const uint firstEdge = indexPathVec[trafficPersonVec[p].indexPathCurr];
         if (firstEdge == -1) {
           // Return if this person's path has length zero
           trafficPersonVec[p].active = 2;
           return;
         }
         trafficPersonVec[p].edgeNumLanes = edgesData[firstEdge].numLines;
         trafficPersonVec[p].edgeNextInters = edgesData[firstEdge].nextInters;
         trafficPersonVec[p].length = edgesData[firstEdge].length;
         trafficPersonVec[p].maxSpeedMperSec = edgesData[firstEdge].maxSpeedMperSec;

         // Find the starting position of the current person
         // At least `requiredAmountOfEmptyCells` are needed before the position where the car will
         // be placed
         const ushort requiredAmountOfEmptyCells = s_0;
         const ushort numOfCells = ceil(trafficPersonVec[p].length);
         const ushort initShift = static_cast<ushort>(0.5f * numOfCells); //number of cells it should be placed (half of road)
         bool placed = false;
         ushort amountOfEmptySells = 0;
         for (ushort position = initShift; (position < numOfCells) && (placed == false); position++) {
           const ushort numberOfRightLane = trafficPersonVec[p].edgeNumLanes - 1;
           const uchar laneChar = laneMap[mapToReadShift + kMaxMapWidthM * (firstEdge + numberOfRightLane) + position];
           if (laneChar != 0xFF) {
             // If the cell is not empty reset the empty-cells counter
             amountOfEmptySells = 0;
             continue;
           }

           // Keep advancing until enough empty cells have been found
           amountOfEmptySells++;
           if (amountOfEmptySells < requiredAmountOfEmptyCells) { continue; }

           // If we get to this point we can place the car
           trafficPersonVec[p].numOfLaneInEdge = numberOfRightLane;
           trafficPersonVec[p].posInLaneM = position; //m
           const uchar vInMpS = (uchar) (trafficPersonVec[p].v * 3); //speed in m/s *3 (to keep more precision
           laneMap[mapToWriteShift + kMaxMapWidthM * (firstEdge + numberOfRightLane) + position] = vInMpS;
           placed = true;
           break;
         }

         if (placed == false) {
           // Return if the current road is too busy
           return;
         }

         trafficPersonVec[p].v = 0;
         trafficPersonVec[p].LC_stateofLaneChanging = 0;
         trafficPersonVec[p].active = 1;
         trafficPersonVec[p].isInIntersection = 0;
         trafficPersonVec[p].num_steps = 1;
         trafficPersonVec[p].co = 0.0f;
         trafficPersonVec[p].gas = 0.0f;

         const uint nextEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];
         if (nextEdge != -1) {
           trafficPersonVec[p].nextEdgemaxSpeedMperSec = edgesData[nextEdge].maxSpeedMperSec;
           trafficPersonVec[p].nextEdgeNumLanes = edgesData[nextEdge].numLines;
           trafficPersonVec[p].nextEdgeNextInters = edgesData[nextEdge].nextInters;
           trafficPersonVec[p].nextEdgeLength = edgesData[nextEdge].length;
           trafficPersonVec[p].LC_initOKLanes = 0xFF;
           trafficPersonVec[p].LC_endOKLanes = 0xFF;
         }
         return;
       }
     }

     ///////////////////////////////
     //2. it is moving
     if (float(currentTime) == int(currentTime)) { // assuming deltatime = 0.5f --> each second
       trafficPersonVec[p].num_steps++;
     }

     // Try to move current person's car.
     // Car movement is modeled using the Intelligent Driver Model (IDM)
     const uint currentEdge = indexPathVec[trafficPersonVec[p].indexPathCurr];
     const uint nextEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];
     float numMToMove;
     bool getToNextEdge = false;
     bool nextVehicleIsATrafficLight = false;
     float thirdTerm = 0;
     int remainingCellsToCheck = max(30.0f, trafficPersonVec[p].v * deltaTime * 2); //30 or double of the speed*time
     
     bool found = false;
     bool noFirstInLaneBeforeSign = false; //use for stop control (just let 1st to pass)
     bool noFirstInLaneAfterSign = false; //use for stop control (just let 1st to pass)
     float s;
     float delta_v;
     const ushort byteInLine = (ushort) floor(trafficPersonVec[p].posInLaneM);
     const ushort numOfCells = ceil((trafficPersonVec[p].length - intersectionClearance));

     // Check if there is another car in the same lane
     for (ushort b = byteInLine + 2; (b < numOfCells) && (found == false) && (remainingCellsToCheck > 0); b++, remainingCellsToCheck--) {
       const uint posToSample =
          mapToReadShift
          + kMaxMapWidthM * (
            indexPathVec[trafficPersonVec[p].indexPathCurr]
            + static_cast<int>(byteInLine / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
            + trafficPersonVec[p].numOfLaneInEdge)
          + b % kMaxMapWidthM;
       const uchar laneChar = laneMap[posToSample];

       if (laneChar != 0xFF) {
         s = ((float) (b - byteInLine)); //m
         delta_v = trafficPersonVec[p].v - (laneChar / 3.0f); //laneChar is in 3*ms (to save space in array)
         found = true;
         noFirstInLaneBeforeSign = true;
         break;
       }
     }

     // If no obstacle has yet been found, check if the next intersection's traffic light is 
     // available
     if (byteInLine < numOfCells && found == false && remainingCellsToCheck > 0) { //before traffic signaling (and not cell limited)
       // TODO: Here we should check if some lane of the needed edge is enabled
       if (trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge] == 0x00) { //red
         // If no connection to the needed edge is enabled, then that intersection will be treated
         // as a stopped car
         s = ((float) (numOfCells - byteInLine));  // In meters
         delta_v = trafficPersonVec[p].v - 0;
         nextVehicleIsATrafficLight = true;
         found = true;
       }
     }

     // Check if there is another car in the same lane after the traffic light
     // TODO: With the proposed changes it does not seem to make sense to check after the intersection
     for (ushort b = byteInLine + 2; (b < numOfCells) && (found == false) && (remainingCellsToCheck > 0); b++, remainingCellsToCheck--) {
       const uint posToSample =
         mapToReadShift
         + kMaxMapWidthM * (
           indexPathVec[trafficPersonVec[p].indexPathCurr]
           + static_cast<int>(byteInLine / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
           + trafficPersonVec[p].numOfLaneInEdge)
         + b % kMaxMapWidthM;
       const uchar laneChar = laneMap[posToSample];

       if (laneChar != 0xFF) {
         s = ((float) (b - byteInLine)); //m
         delta_v = trafficPersonVec[p].v - (laneChar /
           3.0f); //laneChar is in 3*ms (to save space in array)
         found = true;
         noFirstInLaneAfterSign = true;
         break;
       }
     }

     if (trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge] == 0x0F && remainingCellsToCheck > 0) { //stop 
       //check
       if (noFirstInLaneBeforeSign == false && byteInLine < numOfCells && //first before traffic
         trafficPersonVec[p].v == 0 && //stopped
         noFirstInLaneAfterSign == false) { // noone after the traffic light (otherwise wait before stop) !! Todo also check the beginning of next edge

         trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge] = 0x00; //reset stop
         trafficPersonVec[p].posInLaneM = ceilf(numOfCells) + 1; //move magicly after stop

       } else { //stop before STOP
         if (noFirstInLaneBeforeSign == false) { //just update this if it was the first one before sign
           s = ((float) (numOfCells - byteInLine)); //m
           delta_v = trafficPersonVec[p].v - 0; //it should be treated as an obstacle
           nextVehicleIsATrafficLight = true;
           found = true;
         }
       }
     }

     // NEXT LINE
     if (found == false && remainingCellsToCheck > 0) { //check if in next line
       if ((nextEdge != -1) && (trafficPersonVec[p].edgeNextInters != trafficPersonVec[p].end_intersection)) { // we haven't arrived to destination (check next line)
         ushort nextEdgeLaneToBe = trafficPersonVec[p].numOfLaneInEdge; //same lane

         //printf("trafficPersonVec[p].numOfLaneInEdge %u\n",trafficPersonVec[p].numOfLaneInEdge);
         if (nextEdgeLaneToBe >= trafficPersonVec[p].nextEdgeNumLanes) {
           nextEdgeLaneToBe = trafficPersonVec[p].nextEdgeNumLanes -
             1; //change line if there are less roads
         }

         //printf("2trafficPersonVec[p].numOfLaneInEdge %u\n",trafficPersonVec[p].numOfLaneInEdge);
         ushort numOfCells = ceil(trafficPersonVec[p].nextEdgeLength);

         for (ushort b = 0; (b < numOfCells) && (found == false) && (remainingCellsToCheck > 0); b++, remainingCellsToCheck--) {
           const uint posToSample = mapToReadShift + kMaxMapWidthM * (nextEdge + nextEdgeLaneToBe) + b; // b18 not changed since we check first width
           const uchar laneChar = laneMap[posToSample];

           if (laneChar != 0xFF) {
             s = ((float) (b)); //m
             delta_v = trafficPersonVec[p].v - (laneChar / 3.0f);  // laneChar is in 3*ms (to save space in array)
             found = true;
             break;
           }
         }
       }
     }

     float s_star;
     if (found == true) { //car in front and slower than us
       // 2.1.2 calculate dv_dt
       s_star = s_0 + max(0.0f,
         (trafficPersonVec[p].v * trafficPersonVec[p].T + (trafficPersonVec[p].v *
         delta_v) / (2 * sqrtf(trafficPersonVec[p].a * trafficPersonVec[p].b))));
       thirdTerm =powf(((s_star) / (s)), 2);
       //printf(">FOUND s_star %f thirdTerm %f!!!!\n",s_star,thirdTerm);
     }

     float dv_dt = trafficPersonVec[p].a * (1.0f - std::pow((
       trafficPersonVec[p].v / trafficPersonVec[p].maxSpeedMperSec), 4) - thirdTerm);

     // 2.1.3 update values
     numMToMove = max(0.0f, trafficPersonVec[p].v * deltaTime + 0.5f * (dv_dt) * deltaTime * deltaTime);

     trafficPersonVec[p].v += dv_dt * deltaTime;

     if (trafficPersonVec[p].v < 0) {
       //printf("p %d v %f v0 %f a %f dv_dt %f s %f s_star %f MOVE %f\n",p,trafficPersonVec[p].v,trafficPersonVec[p].maxSpeedMperSec,trafficPersonVec[p].a,dv_dt,s,s_star,numMToMove);
       trafficPersonVec[p].v = 0;
       dv_dt = 0.0f;
     }

     if (calculatePollution && ((float(currentTime) == int(currentTime)))) { // enabled and each second (assuming deltaTime 0.5f)
       // Note: compute CO and Gas values each second

       // CO Calculation
       const float speedMph = trafficPersonVec[p].v * 2.2369362920544; //mps to mph
       const float COStepPerSecond = -0.064 + 0.0056 * speedMph + 0.00026 * (speedMph - 50.0f) * (speedMph - 50.0f);
       if (COStepPerSecond > 0) {
         trafficPersonVec[p].co += COStepPerSecond;
       }

       // Gas Consumption
       const float a = dv_dt;
       const float v = trafficPersonVec[p].v; // in mps
       const float Pea = a > 0.0f ? (0.472f*1.680f*a*a*v) : 0.0f;
       const float gasStepPerSecond = 0.666f + 0.072f*(0.269f*v + 0.000672f*(v*v*v) + 0.0171f*(v*v) + 1.680f*a*v + Pea);
       trafficPersonVec[p].gas += gasStepPerSecond;
     }

     if (trafficPersonVec[p].v == 0) {
       const ushort posInLineCells = (ushort) (trafficPersonVec[p].posInLaneM);
       const uint posToSample =
         mapToWriteShift
         + kMaxMapWidthM * (
             currentEdge
             + static_cast<int>(posInLineCells / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
             + trafficPersonVec[p].numOfLaneInEdge)
         + posInLineCells % kMaxMapWidthM;
       laneMap[posToSample] = 0;

       return;
     }

     trafficPersonVec[p].color = p << 8;
     trafficPersonVec[p].posInLaneM = trafficPersonVec[p].posInLaneM + numMToMove;

     if (trafficPersonVec[p].posInLaneM > trafficPersonVec[p].length) { //reach intersection
       numMToMove = trafficPersonVec[p].posInLaneM - trafficPersonVec[p].length;
       getToNextEdge = true;
     } else { //does not research next intersection
       // If the intersection has not been reached try to changed lane if:
       //   - The car is going at least 10 km per hour
       //   - 5 seconds have happened since the last lane change
       if (trafficPersonVec[p].v > 3.0f && trafficPersonVec[p].num_steps % 5 == 0) {
         // next thing is not a traffic light
         // skip if there is one lane (avoid to do this)
         // skip if it is the last edge
         if (nextVehicleIsATrafficLight == false && trafficPersonVec[p].edgeNumLanes > 1 && nextEdge != -1) {
           ////////////////////////////////////////////////////
           // LC 1 update lane changing status
           if (trafficPersonVec[p].LC_stateofLaneChanging == 0) {
             // 2.2-exp((x-1)^2)
             const float x = trafficPersonVec[p].posInLaneM / trafficPersonVec[p].length;
             if (x > 0.4f) { //just after 40% of the road
               float probabiltyMandatoryState = 2.2 - exp((x - 1) * (x - 1));
               if ((((int) (x * 100) % 100) / 100.0f) < probabiltyMandatoryState) { // pseudo random number
                 trafficPersonVec[p].LC_stateofLaneChanging = 1;
               }
             }
           }

           ////////////////////////////////////////////////////
           // LC 2 NOT MANDATORY STATE
           if (trafficPersonVec[p].LC_stateofLaneChanging == 0) {
             //if(p==40)printf("LC v %f v0 %f a %f\n",trafficPersonVec[p].v,trafficPersonVec[p].maxSpeedMperSec*0.5f,dv_dt);
             // discretionary change: v slower than the current road limit and deccelerating and moving
             if ((trafficPersonVec[p].v < (trafficPersonVec[p].maxSpeedMperSec * 0.7f)) &&
               (dv_dt < 0) && trafficPersonVec[p].v > 3.0f) {
               bool leftLane = trafficPersonVec[p].numOfLaneInEdge > 0; //at least one lane on the left
               bool rightLane = trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].edgeNumLanes - 1; //at least one lane

               if (leftLane == true && rightLane == true) {
                 if (int(trafficPersonVec[p].v) % 2 == 0) { // pseudo random
                   leftLane = false;
                 } else {
                   rightLane = false;
                 }
               }
               ushort laneToCheck;
               if (leftLane == true) {
                 laneToCheck = trafficPersonVec[p].numOfLaneInEdge - 1;
               } else {
                 laneToCheck = trafficPersonVec[p].numOfLaneInEdge + 1;
               }

               uchar v_a, v_b;
               float gap_a, gap_b;
               uchar trafficLightState = trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge];
               calculateGapsLC(mapToReadShift, laneMap, trafficLightState,
                 currentEdge + laneToCheck, trafficPersonVec[p].edgeNumLanes, trafficPersonVec[p].posInLaneM,
                 trafficPersonVec[p].length, v_a, v_b, gap_a, gap_b);

               if (gap_a == 1000.0f && gap_b == 1000.0f) { //lag and lead car very far
                 trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
               } else { // NOT ALONE
                 float b1A = 0.05f, b2A = 0.15f;
                 float b1B = 0.15f, b2B = 0.40f;
                 // s_0-> critical lead gap
                 float g_na_D, g_bn_D;
                 bool acceptLC = true;

                 if (gap_a != 1000.0f) {
                   g_na_D = max(s_0, s_0 + b1A * trafficPersonVec[p].v + b2A *
                     (trafficPersonVec[p].v - v_a * 3.0f));

                   if (gap_a < g_na_D) { //gap smaller than critical gap
                     acceptLC = false;
                   }
                 }

                 if (acceptLC == true && gap_b != 1000.0f) {
                   g_bn_D = max(s_0, s_0 + b1B * v_b * 3.0f + b2B * (v_b * 3.0f - trafficPersonVec[p].v));

                   if (gap_b < g_bn_D) { //gap smaller than critical gap
                     acceptLC = false;
                   }
                 }

                 if (acceptLC == true) {
                   trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
                 }
               }
             }
           }// Discretionary

           ////////////////////////////////////////////////////
           // LC 3 *MANDATORY* STATE
           if (trafficPersonVec[p].LC_stateofLaneChanging == 1) {
             // LC 3.1 Calculate the correct lanes
             if (trafficPersonVec[p].LC_endOKLanes == 0xFF) {
               calculateLaneCarShouldBe(currentEdge, nextEdge, intersections,
                 trafficPersonVec[p].edgeNextInters, trafficPersonVec[p].edgeNumLanes,
                 trafficPersonVec[p].LC_initOKLanes, trafficPersonVec[p].LC_endOKLanes);

               //printf("p%u num lanes %u min %u max %u\n",p,trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);
               if (trafficPersonVec[p].LC_initOKLanes == 0 &&
                 trafficPersonVec[p].LC_endOKLanes == 0) {
                 //exit(0);
               }
             }


             //printf(">>LANE CHANGE\n");
             //printf("LC 0 %u\n",trafficPersonVec[p].numOfLaneInEdge);
             bool leftLane = false, rightLane = false;

             // LC 3.2 CORRECT LANES--> DICRETIONARY LC WITHIN
             if (trafficPersonVec[p].numOfLaneInEdge >= trafficPersonVec[p].LC_initOKLanes &&
               trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].LC_endOKLanes) {
               // for discretionary it should be under some circustances
               if ((trafficPersonVec[p].v < (trafficPersonVec[p].maxSpeedMperSec * 0.7f)) &&
                 (dv_dt < 0) && trafficPersonVec[p].v > 3.0f) {
                 leftLane =
                   (trafficPersonVec[p].numOfLaneInEdge > 0) && //at least one lane on the left
                   (trafficPersonVec[p].numOfLaneInEdge - 1 >= trafficPersonVec[p].LC_initOKLanes)
                   &&
                   (trafficPersonVec[p].numOfLaneInEdge - 1 < trafficPersonVec[p].LC_endOKLanes);
                 rightLane =
                   (trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].edgeNumLanes - 1) &&
                   //at least one lane
                   (trafficPersonVec[p].numOfLaneInEdge + 1 >= trafficPersonVec[p].LC_initOKLanes)
                   &&
                   (trafficPersonVec[p].numOfLaneInEdge + 1 < trafficPersonVec[p].LC_endOKLanes);
                 //printf("D\n");
               }
             }
             // LC 3.3 INCORRECT LANES--> MANDATORY LC
             else {
               //printf("num lanes %u min %u max %u\n",trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);
               //printf("p%u num lanes %u min %u max %u\n",p,trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);

               if (trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].LC_initOKLanes) {
                 rightLane = true;
               } else {
                 leftLane = true;
               }

               if (rightLane == true &&
                 trafficPersonVec[p].numOfLaneInEdge + 1 >= trafficPersonVec[p].edgeNumLanes) {
                 printf("ERROR: RT laneToCheck>=trafficPersonVec[p].edgeNumLanes\n");
               }

               if (leftLane == true && trafficPersonVec[p].numOfLaneInEdge == 0) {
                 printf("ERROR %u: LT laneToCheck>=trafficPersonVec[p].edgeNumLanes OK %u-%u NE %u\n",
                   p, trafficPersonVec[p].LC_initOKLanes, trafficPersonVec[p].LC_endOKLanes,
                   nextEdge);
                 //exit(0);
               }

               //printf("M L %d R %d nL %u\n",leftLane,rightLane,trafficPersonVec[p].numOfLaneInEdge);
             }

             if (leftLane == true || rightLane == true) {
               // choose lane (if necessary)
               if (leftLane == true && rightLane == true) {
                 if ((int) (trafficPersonVec[p].posInLaneM) % 2 == 0) { //pseudo random
                   leftLane = false;
                 } else {
                   rightLane = false;
                 }
               }
               ushort laneToCheck;
               if (leftLane == true) {
                 laneToCheck = trafficPersonVec[p].numOfLaneInEdge - 1;
               } else {
                 laneToCheck = trafficPersonVec[p].numOfLaneInEdge + 1;
               }

               if (laneToCheck >= trafficPersonVec[p].edgeNumLanes) {
                 printf("ERROR: laneToCheck>=trafficPersonVec[p].edgeNumLanes %u %u\n",
                   laneToCheck, trafficPersonVec[p].edgeNumLanes);
               }

               uchar v_a, v_b;
               float gap_a, gap_b;
               //printf("p %u LC 1 %u\n",p,laneToCheck);
               uchar trafficLightState = trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge];
               calculateGapsLC(mapToReadShift, laneMap, trafficLightState,
                 currentEdge + laneToCheck, trafficPersonVec[p].edgeNumLanes, trafficPersonVec[p].posInLaneM,
                 trafficPersonVec[p].length, v_a, v_b, gap_a, gap_b);

               //printf("LC 2 %u %u %f %f\n",v_a,v_b,gap_a,gap_b);
               if (gap_a == 1000.0f && gap_b == 1000.0f) { //lag and lead car very far
                 trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
               } else { // NOT ALONE
                 float b1A = 0.05f, b2A = 0.15f;
                 float b1B = 0.15f, b2B = 0.40f;
                 float gamma = 0.000025;
                 // s_0-> critical lead gap
                 float distEnd = trafficPersonVec[p].length - trafficPersonVec[p].posInLaneM;
                 float expTerm = (1 - exp(-gamma * distEnd * distEnd));

                 float g_na_M, g_bn_M;
                 bool acceptLC = true;

                 if (gap_a != 1000.0f) {
                   g_na_M = max(s_0, s_0 + (b1A * trafficPersonVec[p].v + b2A *
                     (trafficPersonVec[p].v - v_a * 3.0f)));

                   if (gap_a < g_na_M) { //gap smaller than critical gap
                     acceptLC = false;
                   }
                 }

                 if (acceptLC == true && gap_b != 1000.0f) {
                   g_bn_M = max(s_0, s_0 + (b1B * v_b * 3.0f + b2B * (v_b * 3.0f -
                     trafficPersonVec[p].v)));

                   if (gap_b < g_bn_M) { //gap smaller than critical gap
                     acceptLC = false;
                   }
                 }

                 if (acceptLC == true) {
                   trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
                 }
               }
             }
           }// Mandatory
         }//at least two lanes and not stopped by traffic light
       }

       // Update person' speed
       const uchar vInMpS = (uchar) (trafficPersonVec[p].v * 3); //speed in m/s to fit in uchar
       const ushort posInLineCells = (ushort) (trafficPersonVec[p].posInLaneM);
       const uint posToSample =
         mapToWriteShift
         + kMaxMapWidthM * (
             currentEdge
             + static_cast<int>(posInLineCells / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
             + trafficPersonVec[p].numOfLaneInEdge)
         + posInLineCells % kMaxMapWidthM;
       laneMap[posToSample] = vInMpS;
       return;
     }

     if (nextEdge == -1) {
       trafficPersonVec[p].active = 2;
       return;
     }

     // Update current edge information
     trafficPersonVec[p].indexPathCurr++;
     trafficPersonVec[p].maxSpeedMperSec = trafficPersonVec[p].nextEdgemaxSpeedMperSec;
     trafficPersonVec[p].edgeNumLanes = trafficPersonVec[p].nextEdgeNumLanes;
     trafficPersonVec[p].edgeNextInters = trafficPersonVec[p].nextEdgeNextInters;
     trafficPersonVec[p].length = trafficPersonVec[p].nextEdgeLength;
     trafficPersonVec[p].posInLaneM = numMToMove;

     if (trafficPersonVec[p].numOfLaneInEdge >= trafficPersonVec[p].edgeNumLanes) {
       trafficPersonVec[p].numOfLaneInEdge = trafficPersonVec[p].edgeNumLanes - 1; //change line if there are less roads
     }

     // Update person's next edge
     const uint nextEdgeIdx = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];

     if (nextEdgeIdx != -1) {
       trafficPersonVec[p].LC_initOKLanes = 0xFF;
       trafficPersonVec[p].LC_endOKLanes = 0xFF;

       trafficPersonVec[p].nextEdgemaxSpeedMperSec = edgesData[nextEdgeIdx].maxSpeedMperSec;
       trafficPersonVec[p].nextEdgeNumLanes = edgesData[nextEdgeIdx].numLines;
       trafficPersonVec[p].nextEdgeNextInters = edgesData[nextEdgeIdx].nextInters;
       trafficPersonVec[p].nextEdgeLength = edgesData[nextEdgeIdx].length;
     }

     trafficPersonVec[p].LC_stateofLaneChanging = 0;
     uchar vInMpS = (uchar) (trafficPersonVec[p].v * 3); //speed in m/s to fit in uchar
     ushort posInLineCells = (ushort) (trafficPersonVec[p].posInLaneM);

     const uint posToSample =
       mapToWriteShift
       + kMaxMapWidthM * (
           nextEdge
           + static_cast<int>(posInLineCells / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
           + trafficPersonVec[p].numOfLaneInEdge)
       + posInLineCells % kMaxMapWidthM;  // note the last % should not happen
     laneMap[posToSample] = vInMpS;
   }
}

__global__ void kernel_intersectionOneSimulation(
    uint numIntersections,
    float currentTime,
    LC::B18IntersectionData *intersections,
    uchar *trafficLights) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<numIntersections){
    const float deltaEvent = 20.0f;  // 20 seconds between each change in the traffic lights
    if (currentTime > intersections[i].nextEvent && intersections[i].totalInOutEdges > 0) {
      uint edgeOT = intersections[i].edge[intersections[i].state];
      uchar numLinesO = edgeOT >> 24;
      uint edgeONum = edgeOT & kMaskLaneMap; // 0xFFFFF;

      // red old traffic lights
      if ((edgeOT&kMaskInEdge) == kMaskInEdge) {  // Just do it if we were in in
        for (int nL = 0; nL < numLinesO; nL++) {
          trafficLights[edgeONum + nL] = 0x00; //red old traffic light
        }
      }

      for (int iN = 0; iN <= intersections[i].totalInOutEdges + 1; iN++) { //to give a round
        intersections[i].state = (intersections[i].state + 1) % intersections[i].totalInOutEdges;//next light
        if ((intersections[i].edge[intersections[i].state] & kMaskInEdge) == kMaskInEdge) {  // 0x800000
          // green new traffic lights
          uint edgeIT = intersections[i].edge[intersections[i].state];
          uint edgeINum = edgeIT & kMaskLaneMap; //  0xFFFFF; //get edgeI
          uchar numLinesI = edgeIT >> 24;
          
          for (int nL = 0; nL < numLinesI; nL++) {
            trafficLights[edgeINum + nL] = 0xFF;
          }
          
          //trafficLights[edgeINum]=0xFF;
          break;
        }
      }//green new traffic light
      intersections[i].nextEvent = currentTime + deltaEvent;
    }
  } 
}

__global__ void kernel_sampleTraffic(
  int numPeople,
  LC::B18TrafficPerson *trafficPersonVec,
  uint *indexPathVec,
  float *accSpeedPerLinePerTimeInterval,
  float *numVehPerLinePerTimeInterval, //this could have been int
  uint offset
  ) {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p < numPeople) {//CUDA check (inside margins)
    if (trafficPersonVec[p].active == 1) { // just active
      int edgeNum = indexPathVec[trafficPersonVec[p].indexPathCurr];
      accSpeedPerLinePerTimeInterval[edgeNum + offset] += trafficPersonVec[p].v / 3.0f;
      numVehPerLinePerTimeInterval[edgeNum + offset]++;
    }
  }
}
__global__ void kernel_resetPeople(
  int numPeople,
  LC::B18TrafficPerson *trafficPersonVec) {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p < numPeople) {//CUDA check (inside margins)
    trafficPersonVec[p].active = 0;
  }
}

void b18GetSampleTrafficCUDA(std::vector<float>& accSpeedPerLinePerTimeInterval, std::vector<float>& numVehPerLinePerTimeInterval) {
  // copy back people
  size_t size = accSpeedPerLinePerTimeInterval.size() * sizeof(float);
  hipMemcpy(accSpeedPerLinePerTimeInterval.data(), accSpeedPerLinePerTimeInterval_d, size, hipMemcpyDeviceToHost);

  size_t sizeI = numVehPerLinePerTimeInterval.size() * sizeof(uchar);
  hipMemcpy(numVehPerLinePerTimeInterval.data(), numVehPerLinePerTimeInterval_d, sizeI, hipMemcpyDeviceToHost);
}

void b18ResetPeopleLanesCUDA(uint numPeople) {
  kernel_resetPeople << < ceil(numPeople / 1024.0f), 1024 >> > (numPeople, trafficPersonVec_d);
  hipMemset(&laneMap_d[0], -1, halfLaneMap*sizeof(unsigned char));
  hipMemset(&laneMap_d[halfLaneMap], -1, halfLaneMap*sizeof(unsigned char));
}

void b18SimulateTrafficCUDA(float currentTime, uint numPeople, uint numIntersections) {

  ////////////////////////////////////////////////////////////
  // 1. CHANGE MAP: set map to use and clean the other
  if(readFirstMapC==true){
    mapToReadShift=0;
    mapToWriteShift=halfLaneMap;
    gpuErrchk(hipMemset(&laneMap_d[halfLaneMap], -1, halfLaneMap*sizeof(unsigned char)));//clean second half
  }else{
    mapToReadShift=halfLaneMap;
    mapToWriteShift=0;
    gpuErrchk(hipMemset(&laneMap_d[0], -1, halfLaneMap*sizeof(unsigned char)));//clean first half
  }
  readFirstMapC=!readFirstMapC;//next iteration invert use

  // Simulate intersections.
  kernel_intersectionOneSimulation << < ceil(numIntersections / 512.0f), 512 >> > (numIntersections, currentTime, intersections_d, trafficLights_d);
  gpuErrchk(hipPeekAtLastError());

  // Simulate people.
  kernel_trafficSimulation <<< ceil(numPeople / 384.0f), 384>> > (
    numPeople,
    currentTime,
    mapToReadShift,
    mapToWriteShift,
    trafficPersonVec_d,
    indexPathVec_d,
    edgesData_d,
    laneMap_d,
    intersections_d,
    trafficLights_d,
    deviceConnections,
    amountOfConnections);
  gpuErrchk(hipPeekAtLastError());

  // Sample if necessary.
  if ((((float) ((int) currentTime)) == (currentTime)) &&
    ((int) currentTime % ((int) 30)) == 0) { //3min //(sample double each 3min)
    int samplingNumber = (currentTime - startTime) / (30 * numStepsTogether);
    uint offset = numIntersections * samplingNumber;
    //printf("Sample %d\n", samplingNumber);
    kernel_sampleTraffic << < ceil(numPeople / 1024.0f), 1024 >> > (numPeople, trafficPersonVec_d, indexPathVec_d, accSpeedPerLinePerTimeInterval_d, numVehPerLinePerTimeInterval_d, offset);
    gpuErrchk(hipPeekAtLastError());
  }
}//
