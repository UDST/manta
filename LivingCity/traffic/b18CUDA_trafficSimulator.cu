#include "hip/hip_runtime.h"
//CUDA CODE
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""

#include "b18TrafficPerson.h"
#include "b18EdgeData.h"
#include <vector>
#include <iostream>

#ifndef ushort
#define ushort uint16_t
#endif
#ifndef uint
#define uint uint32_t
#endif
#ifndef uchar
#define uchar uint8_t
#endif

///////////////////////////////
// CONSTANTS

__constant__ float intersectionClearance = 7.8f;
// `s_0` refers to the minimum spacing distance used in the Intelligent Driver Model (IDM)
__constant__ float s_0 = 7.0f;

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
    if (abort) exit(code);
  }
}
inline void printMemoryUsage() {
  // show memory usage of GPU
  size_t free_byte;
  size_t total_byte;
  hipError_t cuda_status = hipMemGetInfo(&free_byte, &total_byte);
  if (hipSuccess != cuda_status) {
    printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status));
    exit(1);
  }
  double free_db = (double) free_byte;
  double total_db = (double) total_byte;
  double used_db = total_db - free_db;
  printf("GPU memory usage: used = %.0f, free = %.0f MB, total = %.0f MB\n", used_db / 1024.0 / 1024.0, free_db / 1024.0 / 1024.0, total_db / 1024.0 / 1024.0);
}
////////////////////////////////
// VARIABLES
LC::B18TrafficPerson *trafficPersonVec_d;
uint *indexPathVec_d;
LC::B18EdgeData *edgesData_d;

__constant__ bool calculatePollution = true;
__constant__ float cellSize = 1.0f;

__constant__ float deltaTime = 0.5f;
const float deltaTimeH = 0.5f;

const uint numStepsPerSample = 30.0f / deltaTimeH; //each min
const uint numStepsTogether = 12; //change also in density (10 per hour)

uchar *laneMap_d;
bool readFirstMapC=true;
uint mapToReadShift;
uint mapToWriteShift;
uint halfLaneMap;
float startTime;


LC::B18IntersectionData *intersections_d;
uchar *trafficLights_d;

float* accSpeedPerLinePerTimeInterval_d;
float* numVehPerLinePerTimeInterval_d;

LC::Connection *deviceConnections;
size_t amountOfConnections;
LC::Intersection *deviceIntersections;
size_t amountOfIntersections;

void b18InitCUDA(
    bool fistInitialization,
    std::vector<LC::B18TrafficPerson>& trafficPersonVec,
    std::vector<uint> &indexPathVec,
    std::vector<LC::B18EdgeData>& edgesData,
    std::vector<uchar>& laneMap,
    std::vector<uchar>& trafficLights,
    std::vector<LC::B18IntersectionData>& b18Intersections,
    float startTimeH, float endTimeH,
    std::vector<float>& accSpeedPerLinePerTimeInterval,
    std::vector<float>& numVehPerLinePerTimeInterval,
    const std::vector<LC::Connection> & hostConnections,
    const std::vector<LC::Intersection> & hostIntersections) {

  { // Connections
    amountOfConnections = hostConnections.size();
    size_t size = hostConnections.size() * sizeof(LC::Connection);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &deviceConnections, size));   // Allocate array on device
    gpuErrchk(hipMemcpy(deviceConnections, hostConnections.data(), size, hipMemcpyHostToDevice));
  }

  { // Intersections
    amountOfIntersections = hostIntersections.size();
    size_t size = hostIntersections.size() * sizeof(LC::Intersection);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &deviceIntersections, size));   // Allocate array on device
    gpuErrchk(hipMemcpy(deviceIntersections, hostIntersections.data(), size, hipMemcpyHostToDevice));
  }

  printMemoryUsage();
  { // people
    size_t size = trafficPersonVec.size() * sizeof(LC::B18TrafficPerson);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &trafficPersonVec_d, size));   // Allocate array on device
    gpuErrchk(hipMemcpy(trafficPersonVec_d, trafficPersonVec.data(), size, hipMemcpyHostToDevice));
  }

  { // indexPathVec
    size_t sizeIn = indexPathVec.size() * sizeof(uint);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &indexPathVec_d, sizeIn));   // Allocate array on device
    gpuErrchk(hipMemcpy(indexPathVec_d, indexPathVec.data(), sizeIn, hipMemcpyHostToDevice));
  }
  {//edgeData
    size_t sizeD = edgesData.size() * sizeof(LC::B18EdgeData);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &edgesData_d, sizeD));   // Allocate array on device
    gpuErrchk(hipMemcpy(edgesData_d, edgesData.data(), sizeD, hipMemcpyHostToDevice));
  }
  {//laneMap
    size_t sizeL = laneMap.size() * sizeof(uchar);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &laneMap_d, sizeL));   // Allocate array on device
    gpuErrchk(hipMemcpy(laneMap_d, laneMap.data(), sizeL, hipMemcpyHostToDevice));
    halfLaneMap = laneMap.size() / 2;
  }
  {// b18Intersections
    size_t sizeI = b18Intersections.size() * sizeof(LC::B18IntersectionData);
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &intersections_d, sizeI));   // Allocate array on device
    gpuErrchk(hipMemcpy(intersections_d, b18Intersections.data(), sizeI, hipMemcpyHostToDevice));
    size_t sizeT = trafficLights.size() * sizeof(uchar);//total number of lanes
    if (fistInitialization) gpuErrchk(hipMalloc((void **) &trafficLights_d, sizeT));   // Allocate array on device
    gpuErrchk(hipMemcpy(trafficLights_d, trafficLights.data(), sizeT, hipMemcpyHostToDevice));
  }
  {
    startTime = startTimeH * 3600.0f;
    uint numSamples = ceil(((endTimeH*3600.0f - startTimeH*3600.0f) / (deltaTimeH * numStepsPerSample * numStepsTogether))) + 1; //!!!
    accSpeedPerLinePerTimeInterval.clear();
    numVehPerLinePerTimeInterval.clear();
    accSpeedPerLinePerTimeInterval.resize(numSamples * trafficLights.size());
    numVehPerLinePerTimeInterval.resize(numSamples * trafficLights.size());
    size_t sizeAcc = accSpeedPerLinePerTimeInterval.size() * sizeof(float);
    if (fistInitialization)gpuErrchk(hipMalloc((void **) &accSpeedPerLinePerTimeInterval_d, sizeAcc));   // Allocate array on device
    if (fistInitialization)gpuErrchk(hipMalloc((void **) &numVehPerLinePerTimeInterval_d, sizeAcc));   // Allocate array on device
    gpuErrchk(hipMemset(&accSpeedPerLinePerTimeInterval_d[0], 0, sizeAcc));
    gpuErrchk(hipMemset(&numVehPerLinePerTimeInterval_d[0], 0, sizeAcc));
  }
  printMemoryUsage();
}//

void b18FinishCUDA(void){
  hipFree(deviceConnections);
  hipFree(deviceIntersections);
  hipFree(trafficPersonVec_d);
  hipFree(indexPathVec_d);
  hipFree(edgesData_d);
  hipFree(laneMap_d);
  hipFree(intersections_d);
  hipFree(trafficLights_d);
  hipFree(accSpeedPerLinePerTimeInterval_d);
  hipFree(numVehPerLinePerTimeInterval_d);
}//

 void b18GetDataCUDA(std::vector<LC::B18TrafficPerson>& trafficPersonVec){
   // copy back people
   size_t size = trafficPersonVec.size() * sizeof(LC::B18TrafficPerson);
   hipMemcpy(trafficPersonVec.data(),trafficPersonVec_d,size,hipMemcpyDeviceToHost);//hipMemcpyHostToDevice
 }


 __device__ void calculateGapsLC(
     uint mapToReadShift,
     uchar* laneMap,
     uchar trafficLightState,
     uint laneToCheck,
     ushort numLinesEdge,
     float posInMToCheck,
     float length,
     uchar &v_a,
     uchar &v_b,
     float &gap_a,
     float &gap_b) {
   ushort numOfCells = ceil(length);
   ushort initShift = ceil(posInMToCheck);
   bool found = false;

   // CHECK FORWARD
   //printf("initShift %u numOfCells %u\n",initShift,numOfCells);
   for (ushort b = initShift - 1; (b < numOfCells) && (found == false); b++) { //NOTE -1 to make sure there is none in at the same level
     // laneChar = laneMap[mapToReadShift + maxWidth * (laneToCheck) + b];
     const uint posToSample = mapToReadShift + kMaxMapWidthM * (laneToCheck + (((int) (b / kMaxMapWidthM)) * numLinesEdge)) + b % kMaxMapWidthM;
     const uchar laneChar = laneMap[posToSample];

     if (laneChar != 0xFF) {
       gap_a = ((float) b - initShift); //m
       v_a = laneChar; //laneChar is in 3*ms (to save space in array)
       found = true;
       break;
     }
   }

   if (found == false) {
     if (trafficLightState == 0x00) { //red
       //found=true;
       gap_a = gap_b = 1000.0f; //force to change to the line without vehicle
       v_a = v_b = 0xFF;
       return;
     }
   }

   if (found == false) {
     gap_a = 1000.0f;
   }

   // CHECK BACKWARDS
   found = false;

   //printf("2initShift %u numOfCells %u\n",initShift,numOfCells);
   for (int b = initShift + 1; (b >= 0) && (found == false); b--) {  // NOTE +1 to make sure there is none in at the same level
     const uint posToSample = mapToReadShift + kMaxMapWidthM * (laneToCheck + (((int) (b / kMaxMapWidthM)) * numLinesEdge)) + b % kMaxMapWidthM;
     const uchar laneChar = laneMap[posToSample];
     if (laneChar != 0xFF) {
       gap_b = ((float) initShift - b); //m
       v_b = laneChar; //laneChar is in 3*ms (to save space in array)
       found = true;
       break;
     }
   }

   //printf("3initShift %u numOfCells %u\n",initShift,numOfCells);
   if (found == false) {
     gap_b = 1000.0f;
   }

  }//

 __device__ void calculateLaneCarShouldBe(
   uint curEdgeLane,
   uint nextEdge,
   LC::B18IntersectionData* b18Intersections,
   uint edgeNextInters,
   ushort edgeNumLanes,
   ushort &initOKLanes,
   ushort &endOKLanes) {

   initOKLanes = 0;
   endOKLanes = edgeNumLanes;
   bool currentEdgeFound = false;
   bool exitFound = false;
   ushort numExitToTake = 0;
   ushort numExists = 0;

   for (int eN = b18Intersections[edgeNextInters].totalInOutEdges - 1; eN >= 0; eN--) {  // clockwise
     // retrieve
     uint procEdge = b18Intersections[edgeNextInters].edge[eN];

     if ((procEdge & kMaskLaneMap) == curEdgeLane) { //current edge 0xFFFFF
       currentEdgeFound = true;
       if (exitFound == false) {
         numExitToTake = 0;
       }
       continue;
     }

     if ((procEdge & kMaskInEdge) == 0x0) { //out edge 0x800000
       numExists++;
       if (currentEdgeFound == true) {
         numExitToTake++;
       }
       if (currentEdgeFound == false && exitFound == false) {
         numExitToTake++;
       }
     }
     if ((procEdge & kMaskInEdge) == nextEdge) {
       exitFound = true;
       currentEdgeFound = false;
     }
   }

   if (edgeNumLanes == 0) {
     printf("ERRRROR\n");
   }

   switch (edgeNumLanes) {
     /// ONE LANE
   case 1:
     initOKLanes = 0;
     endOKLanes = 1;
     break;

     /// TWO LANE
   case 2:
     switch (numExists) {
     case 1:
     case 2://all okay
       initOKLanes = 0;
       endOKLanes = 2;
       break;

     case 3:
       if (numExitToTake > 2) { //left
         initOKLanes = 0;
         endOKLanes = 1;
         break;
       }

       initOKLanes = 1;
       endOKLanes = 2;
       break;

     default:

       if (numExitToTake >= numExists - 1) {
         initOKLanes = 0;
         endOKLanes = 1;
         break;
       }

       initOKLanes = 1;
       endOKLanes = 2;
       break;
     }

     break;

     /// THREE LANE
   case 3:
     switch (numExists) {
     case 1:
     case 2://all okay
       initOKLanes = 0;
       endOKLanes = 3;
       break;

     case 3:
       if (numExitToTake > 2) { //left
         initOKLanes = 0;
         endOKLanes = 1;
         break;
       }

       initOKLanes = 1;
       endOKLanes = 3;
       break;

     default:
       if (numExitToTake >= numExists - 1) {
         initOKLanes = 0;
         endOKLanes = 1;
         break;
       }

       initOKLanes = 1;
       endOKLanes = 2;
       break;
     }

     break;

   case 4:
     switch (numExists) {
     case 1:
     case 2://all okay
       initOKLanes = 0;
       endOKLanes = 4;
       break;

     case 3:
       if (numExitToTake == 1) { //right
         initOKLanes = 3;
         endOKLanes = 4;
       }

       if (numExitToTake > 3) { //left
         initOKLanes = 0;
         endOKLanes = 1;
         break;
       }

       initOKLanes = 1;
       endOKLanes = 4;
       break;

     default:
       if (numExitToTake == 1) { //right
         initOKLanes = edgeNumLanes - 1;
         endOKLanes = edgeNumLanes;
       }

       if (numExitToTake >= numExists - 2) {
         initOKLanes = 0;
         endOKLanes = 2;
         break;
       }

       initOKLanes = 1; //also lane 2
       endOKLanes = edgeNumLanes;
     }

     break;

   default:
     switch (numExists) {
     case 1:
     case 2://all okay
       initOKLanes = 0;
       endOKLanes = edgeNumLanes;
       break;

     case 3:
       if (numExitToTake == 1) { //right
         initOKLanes = edgeNumLanes - 1;
         endOKLanes = edgeNumLanes;
       }

       if (numExitToTake > edgeNumLanes - 2) { //left
         initOKLanes = 0;
         endOKLanes = 2;
         break;
       }

       initOKLanes = 1;
       endOKLanes = edgeNumLanes;
       break;

     default:
       if (numExitToTake < 2) { //right
         initOKLanes = edgeNumLanes - 2;
         endOKLanes = edgeNumLanes;
       }

       if (numExitToTake >= numExists - 2) {
         initOKLanes = 0;
         endOKLanes = 2;
         break;
       }

       initOKLanes = 1; //also lane 2
       endOKLanes = edgeNumLanes - 1;
     }

     break;
   }
  }//

 // Kernel that executes on the CUDA device
__global__ void kernel_trafficSimulation(
   const int numPeople,
   float currentTime,
   uint mapToReadShift,
   uint mapToWriteShift,
   LC::B18TrafficPerson *trafficPersonVec,
   uint *indexPathVec,
   LC::B18EdgeData* edgesData,
   uchar *laneMap,

   LC::B18IntersectionData *b18Intersections,
   // TODO: Remove usage of this data
   uchar *trafficLights,

   // TODO: Use only this data for computations
   LC::Connection *connections,
   size_t amountOfConnections,
   LC::Intersection *intersections,
   size_t amountOfIntersections)
 {
   const int p = blockIdx.x * blockDim.x + threadIdx.x;
   // Only proceed if the computed index `p` is valid
   if (p < numPeople) {
     /**
      * First ensure this person's car's info is initialized and whether is it active or not
      */
     if (trafficPersonVec[p].active == 2) {
       // Return if this person has reached its destiny
       return;
     }

     if (trafficPersonVec[p].active == 0){
       if (trafficPersonVec[p].time_departure > currentTime) {
       // Return if it's not yet the time for this person
         return;
       }

       const uint firstEdge = indexPathVec[trafficPersonVec[p].indexPathInit];
       if (firstEdge == -1) {
         // Return if this person's path has length zero
         trafficPersonVec[p].active = 2;
         return;
       }

       // Else initialize this person's data
       trafficPersonVec[p].indexPathCurr = trafficPersonVec[p].indexPathInit;
       trafficPersonVec[p].edgeNumLanes = edgesData[firstEdge].numLines;
       trafficPersonVec[p].edgeNextInters = edgesData[firstEdge].nextInters;
       trafficPersonVec[p].length = edgesData[firstEdge].length;
       trafficPersonVec[p].maxSpeedMperSec = edgesData[firstEdge].maxSpeedMperSec;

       // Find the starting position of the current person
       // At least `requiredAmountOfEmptyCells` are needed before the position where the car will be placed
       const ushort requiredAmountOfEmptyCells = s_0;
       const ushort startingRoadAmountOfCells = ceil(trafficPersonVec[p].length);
       // We will start to search from the middle of the starting road
       const ushort initShift = static_cast<ushort>(0.5f * startingRoadAmountOfCells);
       bool placed = false;
       ushort amountOfEmptySells = 0;
       for (ushort position = initShift; (position < startingRoadAmountOfCells) && (placed == false); position++) {
         const ushort numberOfRightLane = trafficPersonVec[p].edgeNumLanes - 1;
         const uchar laneChar = laneMap[mapToReadShift + kMaxMapWidthM * (firstEdge + numberOfRightLane) + position];
         if (laneChar != 0xFF) {
           // If the cell is not empty reset the empty-cells counter
           amountOfEmptySells = 0;
           continue;
         }

         // Keep advancing until enough empty cells have been found
         amountOfEmptySells++;
         if (amountOfEmptySells < requiredAmountOfEmptyCells) { continue; }

         // If we get to this point we can place the car
         trafficPersonVec[p].numOfLaneInEdge = numberOfRightLane;
         trafficPersonVec[p].posInLaneM = position; //m
         const uchar vInMpS = (uchar) (trafficPersonVec[p].v * 3); //speed in m/s *3 (to keep more precision
         laneMap[mapToWriteShift + kMaxMapWidthM * (firstEdge + numberOfRightLane) + position] = vInMpS;
         placed = true;
         break;
       }

       if (!placed) {
         // Return if the current road is too busy
         return;
       }

       trafficPersonVec[p].v = 0;
       trafficPersonVec[p].LC_stateofLaneChanging = 0;
       trafficPersonVec[p].active = 1;
       trafficPersonVec[p].isInIntersection = 0;
       trafficPersonVec[p].num_steps = 1;
       trafficPersonVec[p].co = 0.0f;
       trafficPersonVec[p].gas = 0.0f;

       const uint nextEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];
       if (nextEdge != -1) {
         // TODO: Storing a reference to the next edge data would be safer (instead of plainly copying it)
         trafficPersonVec[p].nextEdgemaxSpeedMperSec = edgesData[nextEdge].maxSpeedMperSec;
         trafficPersonVec[p].nextEdgeNumLanes = edgesData[nextEdge].numLines;
         trafficPersonVec[p].nextEdgeNextInters = edgesData[nextEdge].nextInters;
         trafficPersonVec[p].nextEdgeLength = edgesData[nextEdge].length;
         trafficPersonVec[p].LC_initOKLanes = 0xFF;
         trafficPersonVec[p].LC_endOKLanes = 0xFF;
       }
       return;
     }

     // At this point we can assume trafficPersonVec[p].active == 1
     if (float(currentTime) == int(currentTime)) { // assuming deltatime = 0.5f --> each second
       trafficPersonVec[p].num_steps++;
     }

     /**
      * Gather enough information to know how the current car should be updated, using the Intelligent Driver Model (IDM)
      */
     const uint currentEdge = indexPathVec[trafficPersonVec[p].indexPathCurr];
     const uint nextEdge = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];
     float numMToMove;
     bool getToNextEdge = false;
     bool nextVehicleIsATrafficLight = false;
     float thirdTerm = 0;
     int remainingCellsToCheck = max(30.0f, trafficPersonVec[p].v * deltaTime * 2); //30 or double of the speed*time

     bool obstacleFound = false;
     bool noFirstInLaneBeforeIntersection = false; //use for stop control (just let 1st to pass)
     bool noFirstInLaneAfterSign = false; //use for stop control (just let 1st to pass)
     float s;
     float delta_v;
     const ushort byteInLine = (ushort) floor(trafficPersonVec[p].posInLaneM);
     const ushort numOfCells = ceil((trafficPersonVec[p].length - intersectionClearance));

     // Check if there is another car in the same lane
     for (ushort b = byteInLine + 2; (b < numOfCells) && (obstacleFound == false) && (remainingCellsToCheck > 0); b++, remainingCellsToCheck--) {
       const uint posToSample =
          mapToReadShift
          + kMaxMapWidthM * (
            indexPathVec[trafficPersonVec[p].indexPathCurr]
            + static_cast<int>(byteInLine / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
            + trafficPersonVec[p].numOfLaneInEdge)
          + b % kMaxMapWidthM;
       const uchar laneChar = laneMap[posToSample];

       if (laneChar != 0xFF) {
         s = ((float) (b - byteInLine)); //m
         delta_v = trafficPersonVec[p].v - (laneChar / 3.0f); //laneChar is in 3*ms (to save space in array)
         obstacleFound = true;
         noFirstInLaneBeforeIntersection = true;
         break;
       }
     }

     // At this point we found an obstacle or we reached the end of the current edge
     // If we are at the end of the current edge, check if this car's lane's connections are enabled
     if (byteInLine < numOfCells && !obstacleFound && remainingCellsToCheck > 0) {
       const int dstVertexNumber = edgesData[currentEdge].originalTargetVertexIndex;
       const auto currentLaneNumber = currentEdge + trafficPersonVec[p].numOfLaneInEdge;
       const auto nextEdgeNumber = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];
       bool atLeastOneEnabledConnection = false;
       for (int connectionIdx = intersections[dstVertexNumber].connectionGraphStart; connectionIdx < intersections[dstVertexNumber].connectionGraphEnd; ++connectionIdx) {
         // Check if a least one connection is enabled between the current edge and the following one
         const LC::Connection & connection = connections[connectionIdx];
         if (connection.inLaneNumber == currentLaneNumber
             && connection.outEdgeNumber == nextEdgeNumber
             && connection.enabled) {
           // TODO: Here I could store the available connection so that I don't need to make this cycle again later on
           atLeastOneEnabledConnection = true;
           break;
         }
       }

       // If no connection to the needed edge is enabled, then that intersection will be treated as an obstacle
       if (!atLeastOneEnabledConnection && nextEdgeNumber != -1) {
         s = ((float) (numOfCells - byteInLine));  // In meters
         delta_v = trafficPersonVec[p].v - 0;
         nextVehicleIsATrafficLight = true;
         obstacleFound = true;
       }
     }

     // Check if there is another car in the same lane after the intersection
     // TODO: At this point we need the information about which lane would be taken in case the intersection needs to be crossed
     for (ushort b = byteInLine + 2; (b < numOfCells) && (obstacleFound == false) && (remainingCellsToCheck > 0); b++, remainingCellsToCheck--) {
       const uint posToSample =
         mapToReadShift
         + kMaxMapWidthM * (
           indexPathVec[trafficPersonVec[p].indexPathCurr]
           + static_cast<int>(byteInLine / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
           + trafficPersonVec[p].numOfLaneInEdge)
         + b % kMaxMapWidthM;
       const uchar laneChar = laneMap[posToSample];

       if (laneChar != 0xFF) {
         s = ((float) (b - byteInLine)); //m
         delta_v = trafficPersonVec[p].v - (laneChar / 3.0f); //laneChar is in 3*ms (to save space in array)
         obstacleFound = true;
         noFirstInLaneAfterSign = true;
         break;
       }
     }

     // TODO: Confirm if all this if statement can be removed
     if (trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge] == 0x0F && remainingCellsToCheck > 0) { //stop
       //check
       if (!noFirstInLaneBeforeIntersection
           && byteInLine < numOfCells //first before traffic
           && trafficPersonVec[p].v == 0 //stopped
           && !noFirstInLaneAfterSign) { // noone after the traffic light (otherwise wait before stop) !! Todo also check the beginning of next edge
         trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge] = 0x00; //reset stop
         trafficPersonVec[p].posInLaneM = ceilf(numOfCells) + 1; //move magicly after stop
       } else { //stop before STOP
         if (noFirstInLaneBeforeIntersection == false) { //just update this if it was the first one before sign
           s = ((float) (numOfCells - byteInLine)); //m
           delta_v = trafficPersonVec[p].v - 0; //it should be treated as an obstacle
           nextVehicleIsATrafficLight = true;
           obstacleFound = true;
         }
       }
     }

     // NEXT LINE
     if (obstacleFound == false && remainingCellsToCheck > 0) { //check if in next line
       if ((nextEdge != -1) && (trafficPersonVec[p].edgeNextInters != trafficPersonVec[p].end_intersection)) { // we haven't arrived to destination (check next line)
         ushort nextEdgeLaneToBe = trafficPersonVec[p].numOfLaneInEdge; //same lane

         if (nextEdgeLaneToBe >= trafficPersonVec[p].nextEdgeNumLanes) {
           nextEdgeLaneToBe = trafficPersonVec[p].nextEdgeNumLanes - 1; //change line if there are less roads
         }

         ushort numOfCells = ceil(trafficPersonVec[p].nextEdgeLength);

         for (ushort b = 0; (b < numOfCells) && (obstacleFound == false) && (remainingCellsToCheck > 0); b++, remainingCellsToCheck--) {
           const uint posToSample = mapToReadShift + kMaxMapWidthM * (nextEdge + nextEdgeLaneToBe) + b; // b18 not changed since we check first width
           const uchar laneChar = laneMap[posToSample];

           if (laneChar != 0xFF) {
             s = ((float) (b)); //m
             delta_v = trafficPersonVec[p].v - (laneChar / 3.0f);  // laneChar is in 3*ms (to save space in array)
             obstacleFound = true;
             break;
           }
         }
       }
     }

     /**
      * Update car's information
      */
     float s_star;
     if (obstacleFound) {
       s_star = s_0 + max(
         0.0f,
         trafficPersonVec[p].v * trafficPersonVec[p].T + (trafficPersonVec[p].v * delta_v) / (2 * sqrtf(trafficPersonVec[p].a * trafficPersonVec[p].b))
       );
       thirdTerm = powf(s_star / s, 2);
     }
     float dv_dt = trafficPersonVec[p].a * (1.0f - std::pow((trafficPersonVec[p].v / trafficPersonVec[p].maxSpeedMperSec), 4) - thirdTerm);

     numMToMove = max(0.0f, trafficPersonVec[p].v * deltaTime + 0.5f * (dv_dt) * deltaTime * deltaTime);
     trafficPersonVec[p].v += dv_dt * deltaTime;
     if (trafficPersonVec[p].v < 0) {
       trafficPersonVec[p].v = 0;
       dv_dt = 0.0f;
     }

     if (calculatePollution && ((float(currentTime) == int(currentTime)))) { // enabled and each second (assuming deltaTime 0.5f)
       // Note: compute CO and Gas values each second

       // CO Calculation
       const float speedMph = trafficPersonVec[p].v * 2.2369362920544; //mps to mph
       const float COStepPerSecond = -0.064 + 0.0056 * speedMph + 0.00026 * (speedMph - 50.0f) * (speedMph - 50.0f);
       if (COStepPerSecond > 0) { trafficPersonVec[p].co += COStepPerSecond; }

       // Gas Consumption
       const float a = dv_dt;
       const float v = trafficPersonVec[p].v; // in mps
       const float Pea = a > 0.0f ? (0.472f*1.680f*a*a*v) : 0.0f;
       const float gasStepPerSecond = 0.666f + 0.072f*(0.269f*v + 0.000672f*(v*v*v) + 0.0171f*(v*v) + 1.680f*a*v + Pea);
       trafficPersonVec[p].gas += gasStepPerSecond;
     }

     if (trafficPersonVec[p].v == 0) {
       const ushort posInLineCells = static_cast<ushort>(trafficPersonVec[p].posInLaneM);
       const uint posToSample =
         mapToWriteShift
         + kMaxMapWidthM * (
             currentEdge
             + static_cast<int>(posInLineCells / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
             + trafficPersonVec[p].numOfLaneInEdge)
         + posInLineCells % kMaxMapWidthM;
       laneMap[posToSample] = 0;

       return;
     }

     trafficPersonVec[p].color = p << 8;
     trafficPersonVec[p].posInLaneM = trafficPersonVec[p].posInLaneM + numMToMove;

     if (trafficPersonVec[p].posInLaneM > trafficPersonVec[p].length) { //reach intersection
       numMToMove = trafficPersonVec[p].posInLaneM - trafficPersonVec[p].length;
       getToNextEdge = true;
     } else { //does not research next intersection
       // If the intersection has not been reached try to changed lane if:
       //   - The car is going at least 10 km per hour
       //   - 5 seconds have happened since the last lane change
       if (trafficPersonVec[p].v > 3.0f && trafficPersonVec[p].num_steps % 5 == 0) {
         // next thing is not a traffic light
         // skip if there is one lane (avoid to do this)
         // skip if it is the last edge
         if (nextVehicleIsATrafficLight == false && trafficPersonVec[p].edgeNumLanes > 1 && nextEdge != -1) {
           ////////////////////////////////////////////////////
           // LC 1 update lane changing status
           if (trafficPersonVec[p].LC_stateofLaneChanging == 0) {
             // 2.2-exp((x-1)^2)
             const float x = trafficPersonVec[p].posInLaneM / trafficPersonVec[p].length;
             if (x > 0.4f) { //just after 40% of the road
               float probabiltyMandatoryState = 2.2 - exp((x - 1) * (x - 1));
               if ((((int) (x * 100) % 100) / 100.0f) < probabiltyMandatoryState) { // pseudo random number
                 trafficPersonVec[p].LC_stateofLaneChanging = 1;
               }
             }
           }

           ////////////////////////////////////////////////////
           // LC 2 NOT MANDATORY STATE
           if (trafficPersonVec[p].LC_stateofLaneChanging == 0) {
             //if(p==40)printf("LC v %f v0 %f a %f\n",trafficPersonVec[p].v,trafficPersonVec[p].maxSpeedMperSec*0.5f,dv_dt);
             // discretionary change: v slower than the current road limit and deccelerating and moving
             if ((trafficPersonVec[p].v < (trafficPersonVec[p].maxSpeedMperSec * 0.7f)) &&
               (dv_dt < 0) && trafficPersonVec[p].v > 3.0f) {
               bool leftLane = trafficPersonVec[p].numOfLaneInEdge > 0; //at least one lane on the left
               bool rightLane = trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].edgeNumLanes - 1; //at least one lane

               if (leftLane == true && rightLane == true) {
                 if (int(trafficPersonVec[p].v) % 2 == 0) { // pseudo random
                   leftLane = false;
                 } else {
                   rightLane = false;
                 }
               }
               ushort laneToCheck;
               if (leftLane == true) {
                 laneToCheck = trafficPersonVec[p].numOfLaneInEdge - 1;
               } else {
                 laneToCheck = trafficPersonVec[p].numOfLaneInEdge + 1;
               }

               uchar v_a, v_b;
               float gap_a, gap_b;
               // TODO: Replace the following line by the a value indicating whether the corresponding intersection is enabled
               uchar trafficLightState = trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge];
               calculateGapsLC(mapToReadShift, laneMap, trafficLightState,
                 currentEdge + laneToCheck, trafficPersonVec[p].edgeNumLanes, trafficPersonVec[p].posInLaneM,
                 trafficPersonVec[p].length, v_a, v_b, gap_a, gap_b);

               if (gap_a == 1000.0f && gap_b == 1000.0f) { //lag and lead car very far
                 trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
               } else { // NOT ALONE
                 float b1A = 0.05f, b2A = 0.15f;
                 float b1B = 0.15f, b2B = 0.40f;
                 // s_0-> critical lead gap
                 float g_na_D, g_bn_D;
                 bool acceptLC = true;

                 if (gap_a != 1000.0f) {
                   g_na_D = max(s_0, s_0 + b1A * trafficPersonVec[p].v + b2A *
                     (trafficPersonVec[p].v - v_a * 3.0f));

                   if (gap_a < g_na_D) { //gap smaller than critical gap
                     acceptLC = false;
                   }
                 }

                 if (acceptLC == true && gap_b != 1000.0f) {
                   g_bn_D = max(s_0, s_0 + b1B * v_b * 3.0f + b2B * (v_b * 3.0f - trafficPersonVec[p].v));

                   if (gap_b < g_bn_D) { //gap smaller than critical gap
                     acceptLC = false;
                   }
                 }

                 if (acceptLC == true) {
                   trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
                 }
               }
             }
           }// Discretionary

           ////////////////////////////////////////////////////
           // LC 3 *MANDATORY* STATE
           if (trafficPersonVec[p].LC_stateofLaneChanging == 1) {
             // LC 3.1 Calculate the correct lanes
             if (trafficPersonVec[p].LC_endOKLanes == 0xFF) {
               calculateLaneCarShouldBe(currentEdge, nextEdge, b18Intersections,
                 trafficPersonVec[p].edgeNextInters, trafficPersonVec[p].edgeNumLanes,
                 trafficPersonVec[p].LC_initOKLanes, trafficPersonVec[p].LC_endOKLanes);

               //printf("p%u num lanes %u min %u max %u\n",p,trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);
               if (trafficPersonVec[p].LC_initOKLanes == 0 &&
                 trafficPersonVec[p].LC_endOKLanes == 0) {
                 //exit(0);
               }
             }


             //printf(">>LANE CHANGE\n");
             //printf("LC 0 %u\n",trafficPersonVec[p].numOfLaneInEdge);
             bool leftLane = false, rightLane = false;

             // LC 3.2 CORRECT LANES--> DICRETIONARY LC WITHIN
             if (trafficPersonVec[p].numOfLaneInEdge >= trafficPersonVec[p].LC_initOKLanes &&
               trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].LC_endOKLanes) {
               // for discretionary it should be under some circustances
               if ((trafficPersonVec[p].v < (trafficPersonVec[p].maxSpeedMperSec * 0.7f)) &&
                 (dv_dt < 0) && trafficPersonVec[p].v > 3.0f) {
                 leftLane =
                   (trafficPersonVec[p].numOfLaneInEdge > 0) && //at least one lane on the left
                   (trafficPersonVec[p].numOfLaneInEdge - 1 >= trafficPersonVec[p].LC_initOKLanes)
                   &&
                   (trafficPersonVec[p].numOfLaneInEdge - 1 < trafficPersonVec[p].LC_endOKLanes);
                 rightLane =
                   (trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].edgeNumLanes - 1) &&
                   //at least one lane
                   (trafficPersonVec[p].numOfLaneInEdge + 1 >= trafficPersonVec[p].LC_initOKLanes)
                   &&
                   (trafficPersonVec[p].numOfLaneInEdge + 1 < trafficPersonVec[p].LC_endOKLanes);
                 //printf("D\n");
               }
             }
             // LC 3.3 INCORRECT LANES--> MANDATORY LC
             else {
               //printf("num lanes %u min %u max %u\n",trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);
               //printf("p%u num lanes %u min %u max %u\n",p,trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);

               if (trafficPersonVec[p].numOfLaneInEdge < trafficPersonVec[p].LC_initOKLanes) {
                 rightLane = true;
               } else {
                 leftLane = true;
               }

               if (rightLane == true &&
                 trafficPersonVec[p].numOfLaneInEdge + 1 >= trafficPersonVec[p].edgeNumLanes) {
                 printf("ERROR: RT laneToCheck>=trafficPersonVec[p].edgeNumLanes\n");
               }

               if (leftLane == true && trafficPersonVec[p].numOfLaneInEdge == 0) {
                 printf("ERROR %u: LT laneToCheck>=trafficPersonVec[p].edgeNumLanes OK %u-%u NE %u\n",
                   p, trafficPersonVec[p].LC_initOKLanes, trafficPersonVec[p].LC_endOKLanes,
                   nextEdge);
                 //exit(0);
               }

               //printf("M L %d R %d nL %u\n",leftLane,rightLane,trafficPersonVec[p].numOfLaneInEdge);
             }

             if (leftLane == true || rightLane == true) {
               // choose lane (if necessary)
               if (leftLane == true && rightLane == true) {
                 if ((int) (trafficPersonVec[p].posInLaneM) % 2 == 0) { //pseudo random
                   leftLane = false;
                 } else {
                   rightLane = false;
                 }
               }
               ushort laneToCheck;
               if (leftLane == true) {
                 laneToCheck = trafficPersonVec[p].numOfLaneInEdge - 1;
               } else {
                 laneToCheck = trafficPersonVec[p].numOfLaneInEdge + 1;
               }

               if (laneToCheck >= trafficPersonVec[p].edgeNumLanes) {
                 printf("ERROR: laneToCheck>=trafficPersonVec[p].edgeNumLanes %u %u\n",
                   laneToCheck, trafficPersonVec[p].edgeNumLanes);
               }

               uchar v_a, v_b;
               float gap_a, gap_b;
               // TODO: Replace the following line by the a value indicating whether the corresponding intersection is enabled
               uchar trafficLightState = trafficLights[currentEdge + trafficPersonVec[p].numOfLaneInEdge];
               calculateGapsLC(mapToReadShift, laneMap, trafficLightState,
                 currentEdge + laneToCheck, trafficPersonVec[p].edgeNumLanes, trafficPersonVec[p].posInLaneM,
                 trafficPersonVec[p].length, v_a, v_b, gap_a, gap_b);

               //printf("LC 2 %u %u %f %f\n",v_a,v_b,gap_a,gap_b);
               if (gap_a == 1000.0f && gap_b == 1000.0f) { //lag and lead car very far
                 trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
               } else { // NOT ALONE
                 float b1A = 0.05f, b2A = 0.15f;
                 float b1B = 0.15f, b2B = 0.40f;
                 float gamma = 0.000025;
                 // s_0-> critical lead gap
                 float distEnd = trafficPersonVec[p].length - trafficPersonVec[p].posInLaneM;
                 float expTerm = (1 - exp(-gamma * distEnd * distEnd));

                 float g_na_M, g_bn_M;
                 bool acceptLC = true;

                 if (gap_a != 1000.0f) {
                   g_na_M = max(s_0, s_0 + (b1A * trafficPersonVec[p].v + b2A *
                     (trafficPersonVec[p].v - v_a * 3.0f)));

                   if (gap_a < g_na_M) { //gap smaller than critical gap
                     acceptLC = false;
                   }
                 }

                 if (acceptLC == true && gap_b != 1000.0f) {
                   g_bn_M = max(s_0, s_0 + (b1B * v_b * 3.0f + b2B * (v_b * 3.0f -
                     trafficPersonVec[p].v)));

                   if (gap_b < g_bn_M) { //gap smaller than critical gap
                     acceptLC = false;
                   }
                 }

                 if (acceptLC == true) {
                   trafficPersonVec[p].numOfLaneInEdge = laneToCheck; // CHANGE LINE
                 }
               }
             }
           }// Mandatory
         }//at least two lanes and not stopped by traffic light
       }

       // Update person' speed
       const uchar vInMpS = (uchar) (trafficPersonVec[p].v * 3); //speed in m/s to fit in uchar
       const ushort posInLineCells = (ushort) (trafficPersonVec[p].posInLaneM);
       const uint posToSample =
         mapToWriteShift
         + kMaxMapWidthM * (
             currentEdge
             + static_cast<int>(posInLineCells / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
             + trafficPersonVec[p].numOfLaneInEdge)
         + posInLineCells % kMaxMapWidthM;
       laneMap[posToSample] = vInMpS;
       return;
     }

     if (nextEdge == -1) {
       trafficPersonVec[p].active = 2;
       return;
     }

     // Update current edge information
     trafficPersonVec[p].indexPathCurr++;
     trafficPersonVec[p].maxSpeedMperSec = trafficPersonVec[p].nextEdgemaxSpeedMperSec;
     trafficPersonVec[p].edgeNumLanes = trafficPersonVec[p].nextEdgeNumLanes;
     trafficPersonVec[p].edgeNextInters = trafficPersonVec[p].nextEdgeNextInters;
     trafficPersonVec[p].length = trafficPersonVec[p].nextEdgeLength;
     trafficPersonVec[p].posInLaneM = numMToMove;

     if (trafficPersonVec[p].numOfLaneInEdge >= trafficPersonVec[p].edgeNumLanes) {
       trafficPersonVec[p].numOfLaneInEdge = trafficPersonVec[p].edgeNumLanes - 1; //change line if there are less roads
     }

     // Update person's next edge
     const uint nextEdgeIdx = indexPathVec[trafficPersonVec[p].indexPathCurr + 1];

     if (nextEdgeIdx != -1) {
       trafficPersonVec[p].LC_initOKLanes = 0xFF;
       trafficPersonVec[p].LC_endOKLanes = 0xFF;

       trafficPersonVec[p].nextEdgemaxSpeedMperSec = edgesData[nextEdgeIdx].maxSpeedMperSec;
       trafficPersonVec[p].nextEdgeNumLanes = edgesData[nextEdgeIdx].numLines;
       trafficPersonVec[p].nextEdgeNextInters = edgesData[nextEdgeIdx].nextInters;
       trafficPersonVec[p].nextEdgeLength = edgesData[nextEdgeIdx].length;
     }

     trafficPersonVec[p].LC_stateofLaneChanging = 0;
     const uchar vInMpS = static_cast<uchar>(trafficPersonVec[p].v * 3); //speed in m/s to fit in uchar
     const ushort posInLineCells = static_cast<ushort>(trafficPersonVec[p].posInLaneM);

     const uint posToSample =
       mapToWriteShift
       + kMaxMapWidthM * (
           nextEdge
           + static_cast<int>(posInLineCells / kMaxMapWidthM) * trafficPersonVec[p].edgeNumLanes
           + trafficPersonVec[p].numOfLaneInEdge)
       + posInLineCells % kMaxMapWidthM;  // note the last % should not happen
     laneMap[posToSample] = vInMpS;
   }
}

__global__ void kernel_intersectionOneSimulation(
    uint numIntersections,
    float currentTime,
    LC::B18IntersectionData *b18Intersections,
    uchar *trafficLights) {
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  if(i<numIntersections){
    const float deltaEvent = 20.0f;  // 20 seconds between each change in the traffic lights
    if (currentTime > b18Intersections[i].nextEvent && b18Intersections[i].totalInOutEdges > 0) {
      uint edgeOT = b18Intersections[i].edge[b18Intersections[i].state];
      uchar numLinesO = edgeOT >> 24;
      uint edgeONum = edgeOT & kMaskLaneMap; // 0xFFFFF;

      // red old traffic lights
      if ((edgeOT&kMaskInEdge) == kMaskInEdge) {  // Just do it if we were in in
        for (int nL = 0; nL < numLinesO; nL++) {
          trafficLights[edgeONum + nL] = 0x00; //red old traffic light
        }
      }

      for (int iN = 0; iN <= b18Intersections[i].totalInOutEdges + 1; iN++) { //to give a round
        b18Intersections[i].state = (b18Intersections[i].state + 1) % b18Intersections[i].totalInOutEdges;//next light
        if ((b18Intersections[i].edge[b18Intersections[i].state] & kMaskInEdge) == kMaskInEdge) {  // 0x800000
          // green new traffic lights
          uint edgeIT = b18Intersections[i].edge[b18Intersections[i].state];
          uint edgeINum = edgeIT & kMaskLaneMap; //  0xFFFFF; //get edgeI
          uchar numLinesI = edgeIT >> 24;

          for (int nL = 0; nL < numLinesI; nL++) {
            trafficLights[edgeINum + nL] = 0xFF;
          }

          //trafficLights[edgeINum]=0xFF;
          break;
        }
      }//green new traffic light
      b18Intersections[i].nextEvent = currentTime + deltaEvent;
    }
  }
}

__global__ void kernel_sampleTraffic(
  int numPeople,
  LC::B18TrafficPerson *trafficPersonVec,
  uint *indexPathVec,
  float *accSpeedPerLinePerTimeInterval,
  float *numVehPerLinePerTimeInterval, //this could have been int
  uint offset
  ) {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p < numPeople) {//CUDA check (inside margins)
    if (trafficPersonVec[p].active == 1) { // just active
      int edgeNum = indexPathVec[trafficPersonVec[p].indexPathCurr];
      accSpeedPerLinePerTimeInterval[edgeNum + offset] += trafficPersonVec[p].v / 3.0f;
      numVehPerLinePerTimeInterval[edgeNum + offset]++;
    }
  }
}
__global__ void kernel_resetPeople(
  int numPeople,
  LC::B18TrafficPerson *trafficPersonVec) {
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  if (p < numPeople) {//CUDA check (inside margins)
    trafficPersonVec[p].active = 0;
  }
}

void b18GetSampleTrafficCUDA(std::vector<float>& accSpeedPerLinePerTimeInterval, std::vector<float>& numVehPerLinePerTimeInterval) {
  // copy back people
  size_t size = accSpeedPerLinePerTimeInterval.size() * sizeof(float);
  hipMemcpy(accSpeedPerLinePerTimeInterval.data(), accSpeedPerLinePerTimeInterval_d, size, hipMemcpyDeviceToHost);

  size_t sizeI = numVehPerLinePerTimeInterval.size() * sizeof(uchar);
  hipMemcpy(numVehPerLinePerTimeInterval.data(), numVehPerLinePerTimeInterval_d, sizeI, hipMemcpyDeviceToHost);
}

void b18ResetPeopleLanesCUDA(uint numPeople) {
  kernel_resetPeople << < ceil(numPeople / 1024.0f), 1024 >> > (numPeople, trafficPersonVec_d);
  hipMemset(&laneMap_d[0], -1, halfLaneMap*sizeof(unsigned char));
  hipMemset(&laneMap_d[halfLaneMap], -1, halfLaneMap*sizeof(unsigned char));
}

void b18SimulateTrafficCUDA(float currentTime, uint numPeople, uint numIntersections) {

  ////////////////////////////////////////////////////////////
  // 1. CHANGE MAP: set map to use and clean the other
  if(readFirstMapC==true){
    mapToReadShift=0;
    mapToWriteShift=halfLaneMap;
    gpuErrchk(hipMemset(&laneMap_d[halfLaneMap], -1, halfLaneMap*sizeof(unsigned char)));//clean second half
  }else{
    mapToReadShift=halfLaneMap;
    mapToWriteShift=0;
    gpuErrchk(hipMemset(&laneMap_d[0], -1, halfLaneMap*sizeof(unsigned char)));//clean first half
  }
  readFirstMapC=!readFirstMapC;//next iteration invert use

  // Update intersections.
  kernel_intersectionOneSimulation<<<ceil(numIntersections / 512.0f), 512>>>(
    numIntersections,
    currentTime,
    intersections_d,
    trafficLights_d);
  gpuErrchk(hipPeekAtLastError());

  // Simulate people.
  kernel_trafficSimulation<<<ceil(numPeople / 384.0f), 384>>>(
    numPeople,
    currentTime,
    mapToReadShift,
    mapToWriteShift,
    trafficPersonVec_d,
    indexPathVec_d,
    edgesData_d,
    laneMap_d,
    intersections_d,
    trafficLights_d,
    deviceConnections,
    amountOfConnections,
    deviceIntersections,
    amountOfIntersections);
  gpuErrchk(hipPeekAtLastError());

  // Sample if necessary.
  if ((((float) ((int) currentTime)) == (currentTime)) &&
    ((int) currentTime % ((int) 30)) == 0) { //3min //(sample double each 3min)
    int samplingNumber = (currentTime - startTime) / (30 * numStepsTogether);
    uint offset = numIntersections * samplingNumber;
    //printf("Sample %d\n", samplingNumber);
    kernel_sampleTraffic << < ceil(numPeople / 1024.0f), 1024 >> > (numPeople, trafficPersonVec_d, indexPathVec_d, accSpeedPerLinePerTimeInterval_d, numVehPerLinePerTimeInterval_d, offset);
    gpuErrchk(hipPeekAtLastError());
  }
}//
