#include "hip/hip_runtime.h"
//CUDA CODE
#include <stdio.h>
#include <vector>
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include ""

#include "cudaTrafficPerson.h"
#include "cudaEdgeData.h"

#define ushort unsigned short
#define uint unsigned int
#define uchar unsigned char

///////////////////////////////
// CONSTANTS
//__constant__ float deltaTimeG=1.0f*(1.0f/3600.0f);
__constant__ float s_0=7.0f;
//const float deltaTimeC=1.0f*(1.0f/3600.0f);
////////////////////////////////
// VARIABLES
LC::CUDATrafficPerson *trafficPersonVec_d;
//ushort *nextEdgeM_d;
LC::edgeData *edgesData_d;
float cellSize;
float deltaTime;
uchar *laneMap_d;
bool readFirstMapC=true;
uint mapToReadShift;
uint mapToWriteShift;
uint halfLaneMap;
ushort maxWidth;

LC::intersectionData *intersections_d;
uchar *trafficLights_d;

void initCUDA(float _cellSize,float _deltaTime,ushort _maxWidth,std::vector<LC::CUDATrafficPerson>& trafficPersonVec,std::vector<LC::edgeData>& edgesData,std::vector<uchar>& laneMap,std::vector<LC::intersectionData>& intersections){
	cellSize=_cellSize;
	deltaTime=_deltaTime;
	maxWidth=_maxWidth;
	// people
	size_t size = trafficPersonVec.size() * sizeof(LC::CUDATrafficPerson);
	hipError_t err ;
	err=hipMalloc((void **) &trafficPersonVec_d, size);   // Allocate array on device
	if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
	err =hipMemcpy(trafficPersonVec_d,trafficPersonVec.data(),size,hipMemcpyHostToDevice);
	if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
	
	/*//nextEdge
	size_t sizeE = nextEdgeM.size() * sizeof(ushort);
	hipMalloc((void **) &nextEdgeM_d, sizeE);   // Allocate array on device
	hipMemcpy(nextEdgeM_d,nextEdgeM.data(),sizeE,hipMemcpyHostToDevice);*/
	//edgeData
	size_t sizeD = edgesData.size() * sizeof(LC::edgeData);
	err=hipMalloc((void **) &edgesData_d, sizeD);   // Allocate array on device
	if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
	err=hipMemcpy(edgesData_d,edgesData.data(),sizeD,hipMemcpyHostToDevice);
	if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
	//laneMap
	size_t sizeL = laneMap.size() * sizeof(uchar);
	err=hipMalloc((void **) &laneMap_d, sizeL);   // Allocate array on device
	if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
	err=hipMemcpy(laneMap_d,laneMap.data(),sizeL,hipMemcpyHostToDevice);
	if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
	halfLaneMap=laneMap.size()/2;
	// intersections
	size_t sizeI = intersections.size() * sizeof(LC::intersectionData);
	err=hipMalloc((void **) &intersections_d, sizeI);   // Allocate array on device
	if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
	err=hipMemcpy(intersections_d,intersections.data(),sizeI,hipMemcpyHostToDevice);
	if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
	size_t sizeT = (halfLaneMap/_maxWidth) * sizeof(uchar);//total number of lanes
	err=hipMalloc((void **) &trafficLights_d, sizeT);   // Allocate array on device
	if ( hipSuccess != err )fprintf( stderr, "Cuda error: %s.\n",hipGetErrorString( err) );
}//

void finishCUDA(void){
	//////////////////////////////
	// FINISH
	hipFree(trafficPersonVec_d);
	//hipFree(nextEdgeM_d);
	hipFree(edgesData_d);
	hipFree(laneMap_d);
	hipFree(intersections_d);
	hipFree(trafficLights_d);

}//

 void getDataCUDA(std::vector<LC::CUDATrafficPerson>& trafficPersonVec,std::vector<uchar>& trafficLights){
	 // copy back people
	 size_t size = trafficPersonVec.size() * sizeof(LC::CUDATrafficPerson);
	 hipMemcpy(trafficPersonVec.data(),trafficPersonVec_d,size,hipMemcpyDeviceToHost);//hipMemcpyHostToDevice

	 size_t sizeI = trafficLights.size() * sizeof(uchar);
	 hipMemcpy(trafficLights.data(),trafficLights_d,sizeI,hipMemcpyDeviceToHost);
 }


 __device__ void calculateGapsLC(
		float cellSize,
		ushort maxWidth,
		uint mapToReadShift,
		uchar *laneMap,
		uchar trafficLightState,
		ushort laneToCheck,
		float posInMToCheck,
		float length,
		uchar &v_a,
		uchar &v_b,
		float &gap_a,
		float &gap_b){
		ushort numOfCells=ceil(length/cellSize);
		ushort initShift=ceil(posInMToCheck/cellSize);
		uchar laneChar;
		bool found=false;
		// CHECK FORWARD
		//printf("initShift %u numOfCells %u\n",initShift,numOfCells);
		for(ushort b=initShift-1;(b<numOfCells)&&(found==false);b++){//NOTE -1 to make sure there is none in at the same level
			laneChar=laneMap[mapToReadShift+maxWidth*(laneToCheck)+b];
			if(laneChar!=0xFF){
				gap_a=((float)b-initShift)*cellSize;//m
				v_a=laneChar;//laneChar is in 3*ms (to save space in array)
				found=true;
				break;
			}
		}
		if(found==false){
			if(trafficLightState==0x00){//red
				//gap_a=((float)numOfCells-initShift)*cellSize;
				//found=true;
				gap_a=gap_b=1000.0f;//force to change to the line without vehicle
				v_a=v_b=0xFF;
				return;
			}
		}
		if(found==false){
			gap_a=1000.0f;
		}
		// CHECK BACKWARDS
		found=false;
		//printf("2initShift %u numOfCells %u\n",initShift,numOfCells);
		for(int b=initShift+1;(b>=0)&&(found==false);b--){//NOTE +1 to make sure there is none in at the same level
			laneChar=laneMap[mapToReadShift+maxWidth*(laneToCheck)+b];
			if(laneChar!=0xFF){
				gap_b=((float)initShift-b)*cellSize;//m
				v_b=laneChar;//laneChar is in 3*ms (to save space in array)
				found=true;
				break;
			}
		}
		//printf("3initShift %u numOfCells %u\n",initShift,numOfCells);
		if(found==false){
			gap_b=1000.0f;
		}

	}//

 __device__ void calculateLaneCarShouldBe(
		ushort curEdgeLane,
		ushort nextEdge,
		LC::intersectionData *intersections,
		ushort edgeNextInters,
		ushort edgeNumLanes,
		ushort &initOKLanes,
		ushort &endOKLanes){
		initOKLanes=0;
		endOKLanes=edgeNumLanes;
		/*if(DEBUG_TRAFFIC==1)printf("curEdgeLane %05x nextEdge %05x\n",curEdgeLane,nextEdge);
		if(DEBUG_TRAFFIC==1){
			for(int eN=0;eN<intersections[edgeNextInters].totalInOutEdges;eN++){
				printf("* procEdge %05x\n",intersections[edgeNextInters].edge[eN]);
			}
		}*/
		bool currentEdgeFound=false;
		bool exitFound=false;
		ushort numExitToTake=0;
		ushort numExists=0;
		for(int eN=intersections[edgeNextInters].totalInOutEdges-1;eN>=0;eN--){//clockwise
			uint procEdge=intersections[edgeNextInters].edge[eN];
			if((procEdge&0xFFFF)==curEdgeLane){//current edge
				//if(DEBUG_TRAFFIC==1)printf("CE procEdge %05x\n",procEdge);
				currentEdgeFound=true;
				if(exitFound==false)
					numExitToTake=0;
				continue;
			}
			
			
			if((procEdge&0x010000)==0x0){//out edge
				//if(DEBUG_TRAFFIC==1)printf("   procEdge %05x\n",procEdge);
				numExists++;
				if(currentEdgeFound==true){
					numExitToTake++;
				}
				if(currentEdgeFound==false&&exitFound==false){
					numExitToTake++;
				}
			}
			if((procEdge&0xFFFF)==nextEdge){
				exitFound=true;
				currentEdgeFound=false;
				//if(DEBUG_TRAFFIC==1)printf("NE procEdge %05x\n",procEdge);
			}
		}
		//if(DEBUG_TRAFFIC==1)printf("Num extis %u Num exit to take %u%\n",numExists,numExitToTake);
		if(edgeNumLanes==0){
			return;//printf("ERRRROR\n");
		}
		switch(edgeNumLanes){
			/// ONE LANE
		case 1:
			initOKLanes=0;
			endOKLanes=1;
			break;
			/// TWO LANE
		case 2:
			switch(numExists){
			case 1:
			case 2://all okay
				initOKLanes=0;
				endOKLanes=2;
				break;
			case 3:
				if(numExitToTake>2){//left
					initOKLanes=0;
					endOKLanes=1;
					break;
				}
				initOKLanes=1;
				endOKLanes=2;
				break;
			default:
				if(numExitToTake>=numExists-1){
					initOKLanes=0;
					endOKLanes=1;
					break;
				}
				initOKLanes=1;
				endOKLanes=2;
				break;
			}
			break;
			/// THREE LANE
		case 3:
			switch(numExists){
			case 1:
			case 2://all okay
				initOKLanes=0;
				endOKLanes=3;
				break;
			case 3:
				if(numExitToTake>2){//left
					initOKLanes=0;
					endOKLanes=1;
					break;
				}
				initOKLanes=1;
				endOKLanes=3;
				break;
			default:
				if(numExitToTake>=numExists-1){
					initOKLanes=0;
					endOKLanes=1;
					break;
				}
				initOKLanes=1;
				endOKLanes=2;
				break;
			}
			break;
		case 4:
			switch(numExists){
			case 1:
			case 2://all okay
				initOKLanes=0;
				endOKLanes=4;
				break;
			case 3:
				if(numExitToTake==1){//right
					initOKLanes=3;
					endOKLanes=4;
				}
				if(numExitToTake>3){//left
					initOKLanes=0;
					endOKLanes=1;
					break;
				}
				initOKLanes=1;
				endOKLanes=4;
				break;
			default:
				if(numExitToTake==1){//right
					initOKLanes=edgeNumLanes-1;
					endOKLanes=edgeNumLanes;
				}
				if(numExitToTake>=numExists-2){
					initOKLanes=0;
					endOKLanes=2;
					break;
				}
				initOKLanes=1;//also lane 2
				endOKLanes=edgeNumLanes;
			}
			break;
		default:
			switch(numExists){
			case 1:
			case 2://all okay
				initOKLanes=0;
				endOKLanes=edgeNumLanes;
				break;
			case 3:
				if(numExitToTake==1){//right
					initOKLanes=edgeNumLanes-1;
					endOKLanes=edgeNumLanes;
				}
				if(numExitToTake>edgeNumLanes-2){//left
					initOKLanes=0;
					endOKLanes=2;
					break;
				}
				initOKLanes=1;
				endOKLanes=edgeNumLanes;
				break;
			default:
				if(numExitToTake<2){//right
					initOKLanes=edgeNumLanes-2;
					endOKLanes=edgeNumLanes;
				}
				if(numExitToTake>=numExists-2){
					initOKLanes=0;
					endOKLanes=2;
					break;
				}
				initOKLanes=1;//also lane 2
				endOKLanes=edgeNumLanes-1;
			}
			break;

		}
	}//

__device__ int cuda_qrand(){
	return 10;
}

 // Kernel that executes on the CUDA device
 __global__ void kernel_trafficSimulation(
	 int numPeople,
	 float currentTime,
	 float cellSize,
	 float deltaTime,
	 LC::CUDATrafficPerson *trafficPersonVec,
	 //ushort *nextEdgeM,
	 LC::edgeData* edgesData,
	 uchar *laneMap,
	 LC::intersectionData *intersections,
	 uchar *trafficLights,
	 uint mapToReadShift,
	 uint mapToWriteShift,
	 ushort maxWidth)
 {
	 const bool DEBUG_TRAFFIC=0;
	 int p = blockIdx.x * blockDim.x + threadIdx.x;
	 //printf("p %d Numpe %d\n",p,numPeople);
	 if(p<numPeople){//CUDA check (inside margins)
		 ///
		 ///////////////////////////////
		//2.0. check if finished
		if(trafficPersonVec[p].active==2){
				return;
			}
			///////////////////////////////
			//2.1. check if person should still wait or should start
			if(trafficPersonVec[p].active==0){

				//printf("  1. Person: %d active==0\n",p);
				if(trafficPersonVec[p].time_departure>currentTime){//wait
					//1.1 just continue waiting
					//printf("   1.1 Person: %d wait\n",p);
					return;
				}else{//start
					//1.2 find first edge
					trafficPersonVec[p].currPathEdge=0;
					ushort firstEdge=trafficPersonVec[p].personPath[0];
					if(firstEdge==0xFFFF){
						trafficPersonVec[p].active=2;
						//printf("0xFFFF\n");
						return;
					}

					
					//1.3 update person edgeData
					//if(DEBUG_TRAFFIC==1)printf("   1.3 Person: %d put in first edge %u\n",p,firstEdge);
					//printf("edgesData %d\n",edgesData);

					// COPY DATA FROM EDGE TO PERSON
					trafficPersonVec[p].edgeNumLanes=edgesData[firstEdge].numLines;
					trafficPersonVec[p].edgeNextInters=edgesData[firstEdge].nextInters;

					trafficPersonVec[p].length=edgesData[firstEdge].length;
					trafficPersonVec[p].maxSpeedMperSec=edgesData[firstEdge].maxSpeedMperSec;
					//printf("edgesData %.10f\n",edgesData[firstEdge].maxSpeedCellsPerDeltaTime);
					//1.4 try to place it in middle of edge
					ushort numOfCells=ceil(trafficPersonVec[p].length/cellSize);
					ushort initShift=(ushort)(0.5f*numOfCells);//number of cells it should be placed (half of road)

					uchar laneChar;
					bool placed=false;

					ushort numCellsEmptyToBePlaced=s_0/cellSize;
					ushort countEmptyCells=0;
					for(ushort b=initShift;(b<numOfCells)&&(placed==false);b++){
						//for(int lN=trafficPersonVec[p].edgeNumLanes-1;lN>=0;lN--){
						//ushort lN=0;//just first LANE !!!!!!!
						ushort lN=trafficPersonVec[p].edgeNumLanes-1;//just right LANE !!!!!!!
						laneChar=laneMap[mapToReadShift+maxWidth*(firstEdge+lN)+b];//get byte of edge (proper line)
						if(laneChar!=0xFF){
							countEmptyCells=0;
							continue;
						}
						countEmptyCells++;// ensure there is enough room to place the car
						if(countEmptyCells<numCellsEmptyToBePlaced){
							continue;
						}
						trafficPersonVec[p].numOfLaneInEdge=lN;
						trafficPersonVec[p].posInLaneM=b*cellSize;//m
						uchar vInMpS=(uchar)(trafficPersonVec[p].v*3);//speed in m/s *3 (to keep more precision
						laneMap[mapToWriteShift+maxWidth*(firstEdge+lN)+b]=vInMpS;
						placed=true;
						//printf("Placed\n");
						break;
						//}
					}
					if(placed==false){//not posible to start now
						return;
					}
					trafficPersonVec[p].v=0;//trafficPersonVec[p].maxSpeedCellsPerDeltaTime;//(20000.0f*deltaTime)/cellSize;//20km/h-->cell/delta time
					trafficPersonVec[p].LC_stateofLaneChanging=0;
					//1.5 active car
					
					trafficPersonVec[p].active=1;
					trafficPersonVec[p].num_steps=1;
					trafficPersonVec[p].gas=0;
					//trafficPersonVec[p].nextPathEdge++;//incremet so it continues in next edge
					// set up next edge info
					ushort nextEdge=trafficPersonVec[p].personPath[1];
					//trafficPersonVec[p].nextEdge=nextEdge;
					if(nextEdge!=0xFFFF){
						trafficPersonVec[p].nextEdgemaxSpeedMperSec=edgesData[nextEdge].maxSpeedMperSec;
						trafficPersonVec[p].nextEdgeNumLanes=edgesData[nextEdge].numLines;
						trafficPersonVec[p].nextEdgeNextInters=edgesData[nextEdge].nextInters;
						trafficPersonVec[p].nextEdgeLength=edgesData[nextEdge].length;
						//trafficPersonVec[p].nextPathEdge++;
						trafficPersonVec[p].LC_initOKLanes=0xFF;
						trafficPersonVec[p].LC_endOKLanes=0xFF;
					}
					return;
				}
			}
			
			///////////////////////////////
			//2. it is moving
			trafficPersonVec[p].num_steps++;
			//2.1 try to move
			float numMToMove;
			bool getToNextEdge=false;
			bool nextVehicleIsATrafficLight=false;
			ushort currentEdge=trafficPersonVec[p].personPath[trafficPersonVec[p].currPathEdge];
			ushort nextEdge=trafficPersonVec[p].personPath[trafficPersonVec[p].currPathEdge+1];
			//if(trafficPersonVec[p].posInLaneM<trafficPersonVec[p].length){
			// www.vwi.tu-dresden.de/~treiber/MicroApplet/IDM.html
			// IDM
			float thirdTerm=0;
			///////////////////////////////////////////////////
			// 2.1.1 Find front car
			//int numCellsCheck=fmax<float>(15.0f,trafficPersonVec[p].v*deltaTime*2);//15 or double of the speed*time
			// SAME LINE
			bool found=false;
			float s;
			float delta_v;
			uchar laneChar;
			ushort byteInLine=(ushort)floor(trafficPersonVec[p].posInLaneM/cellSize);
			ushort numOfCells=ceil(trafficPersonVec[p].length/cellSize);
			for(ushort b=byteInLine+2;(b<numOfCells)&&(found==false)/*&&(numCellsCheck>0)*/;b++/*,numCellsCheck--*/){
				laneChar=laneMap[mapToReadShift+maxWidth*(trafficPersonVec[p].personPath[trafficPersonVec[p].currPathEdge]+trafficPersonVec[p].numOfLaneInEdge)+b];
				if(laneChar!=0xFF){
					s=((float)(b-byteInLine)*cellSize);//m
					delta_v=trafficPersonVec[p].v-(laneChar/3.0f);//laneChar is in 3*ms (to save space in array)
					found=true;
					//printf("\nFOUND Car Same Lane s %f delta_v %f!!!!\n",s,delta_v);
					break;
				}
			}
			// TRAFFIC LIGHT
			if(found==false){//check if traffic light is red
				if(trafficLights[currentEdge]==0x00){//red
					s=((float)(numOfCells-byteInLine)*cellSize);//m
					delta_v=trafficPersonVec[p].v-0;//it should be treated as an obstacle
					nextVehicleIsATrafficLight=true;
					//printf("\nFOUND TL\n",s,delta_v);
					found=true;
				}
			}
			// NEXT LINE
			if(found==false){//check if in next line
				if((nextEdge!=0xFFFF)&&(trafficPersonVec[p].edgeNextInters!=trafficPersonVec[p].end_intersection)){// we haven't arrived to destination (check next line)

					ushort nextEdgeLaneToBe=trafficPersonVec[p].numOfLaneInEdge;//same lane
					//printf("trafficPersonVec[p].numOfLaneInEdge %u\n",trafficPersonVec[p].numOfLaneInEdge);
					if(nextEdgeLaneToBe>=trafficPersonVec[p].nextEdgeNumLanes){
						nextEdgeLaneToBe=trafficPersonVec[p].nextEdgeNumLanes-1;//change line if there are less roads
					}
					//printf("2trafficPersonVec[p].numOfLaneInEdge %u\n",trafficPersonVec[p].numOfLaneInEdge);
					ushort numOfCells=ceil(trafficPersonVec[p].nextEdgeLength/cellSize);
					for(ushort b=0;(b<numOfCells)&&(found==false)/*&&(numCellsCheck>0)*/;b++/*,numCellsCheck--*/){
						laneChar=laneMap[mapToReadShift+maxWidth*(nextEdge+nextEdgeLaneToBe)+b];
						if(laneChar!=0xFF){
							s=((float)(b)*cellSize);//m
							delta_v=trafficPersonVec[p].v-(laneChar/3.0f);//laneChar is in 3*ms (to save space in array)
							found=true;
							//printf("\FOUND Car Next Lane s %f delta_v %f!!!!\n",s,delta_v);
							//printf("\FOUND Car Next Lane next Edge %u NumLanes %u Be in Lane %u!!!!\n",trafficPersonVec[p].nextEdge,trafficPersonVec[p].nextEdgeNumLanes,nextEdgeLaneToBe);
							break;
						}
					}
				}

			}
			float s_star;
			if(found==true){//car in front and slower than us
				// 2.1.2 calculate dv_dt
				s_star=s_0+fmax(0.0f,(trafficPersonVec[p].v*trafficPersonVec[p].T+(trafficPersonVec[p].v*delta_v)/(2*std::sqrt(trafficPersonVec[p].a*trafficPersonVec[p].b))));
				thirdTerm=std::pow(((s_star)/(s)),2);
				//printf(">FOUND s_star %f thirdTerm %f!!!!\n",s_star,thirdTerm);
			}

			float dv_dt= trafficPersonVec[p].a*(1.0f-std::pow((trafficPersonVec[p].v/trafficPersonVec[p].maxSpeedMperSec),4)-thirdTerm);

			// 2.1.3 update values
			numMToMove=fmax(0.0f,trafficPersonVec[p].v*deltaTime+0.5f*(dv_dt)*deltaTime*deltaTime);

			
			//printf("v %.10f v d %.10f\n",trafficPersonVec[p].v,trafficPersonVec[p].v+((dv_dt/(deltaTime)/deltaTime)));
			trafficPersonVec[p].v+=dv_dt*deltaTime;
			if(trafficPersonVec[p].v<0){
				//printf("p %d v %f v0 %f a %f dv_dt %f s %f s_star %f MOVE %f\n",p,trafficPersonVec[p].v,trafficPersonVec[p].maxSpeedMperSec,trafficPersonVec[p].a,dv_dt,s,s_star,numMToMove);
				trafficPersonVec[p].v=0;
			}
			/////
			//CO2
			//if(trafficPersonVec[p].v>0)
			{
				float speedMph=trafficPersonVec[p].v*2.2369362920544;//mps to mph
				float gasStep=-0.064+0.0056*speedMph+0.00026*(speedMph-50.0f)*(speedMph-50.0f);
				if(gasStep>0){
					gasStep*=deltaTime;
					trafficPersonVec[p].gas+=gasStep;
				}
			}
			//trafficPersonVec[p].gas+=numMToMove/1000.0f;
			//////////////////////////////////////////////

			if(trafficPersonVec[p].v==0){//if not moving not do anything else
				ushort posInLineCells=(ushort)(trafficPersonVec[p].posInLaneM/cellSize);
				laneMap[mapToWriteShift+maxWidth*(currentEdge+trafficPersonVec[p].numOfLaneInEdge)+posInLineCells]=0;
				return;
			}
			//////////

			///////////////////////////////
			// COLOR
			////////////////////////////////

			
			//numCellsToMove=trafficPersonVec[p].v;
			trafficPersonVec[p].posInLaneM=trafficPersonVec[p].posInLaneM+numMToMove;
			if(trafficPersonVec[p].posInLaneM>trafficPersonVec[p].length){//research intersection
				numMToMove=trafficPersonVec[p].posInLaneM-trafficPersonVec[p].length;
				getToNextEdge=true;
			}else{//does not research next intersection
				////////////////////////////////////////////////////////
				// LANE CHANGING (happens when we are not reached the intersection)
				if(trafficPersonVec[p].v>3.0f&&//at least 10km/h to try to change lane
					trafficPersonVec[p].num_steps%10==0//just check every (10 steps) 5 seconds	
					){
					//next thing is not a traffic light 
					// skip if there is one lane (avoid to do this)
					// skip if it is the last edge
					if(nextVehicleIsATrafficLight==false&&trafficPersonVec[p].edgeNumLanes>1&&nextEdge!=0xFFFF){

						////////////////////////////////////////////////////
						// LC 1 update lane changing status
						if(trafficPersonVec[p].LC_stateofLaneChanging==0){
							// 2.2-exp((x-1)^2)
							float x=trafficPersonVec[p].posInLaneM/trafficPersonVec[p].length;
							if(x>0.4f){//just after 40% of the road
								float probabiltyMandatoryState=2.2-exp((x-1)*(x-1));
								{
									trafficPersonVec[p].LC_stateofLaneChanging=1;
								}
							}

						}

						////////////////////////////////////////////////////
						// LC 2 NOT MANDATORY STATE
						if(trafficPersonVec[p].LC_stateofLaneChanging==0){
							//if(p==40)printf("LC v %f v0 %f a %f\n",trafficPersonVec[p].v,trafficPersonVec[p].maxSpeedMperSec*0.5f,dv_dt);
							// discretionary change: v slower than the current road limit and deccelerating and moving
							if((trafficPersonVec[p].v<(trafficPersonVec[p].maxSpeedMperSec*0.7f))&&(dv_dt<0)&&trafficPersonVec[p].v>3.0f){
								//printf(">>LANE CHANGE\n");
								ushort laneToCheck;//!!!!
								//printf("LC 0 %u\n",trafficPersonVec[p].numOfLaneInEdge);
								bool leftLane=trafficPersonVec[p].numOfLaneInEdge>0;//at least one lane on the left
								bool rightLane=trafficPersonVec[p].numOfLaneInEdge<trafficPersonVec[p].edgeNumLanes-1;//at least one lane
								if(leftLane==true&&rightLane==true){
									if((int(currentTime))%2==0){
										leftLane=false;
									}else{
										rightLane=false;
									}
								}
								if(leftLane==true){
									laneToCheck=trafficPersonVec[p].numOfLaneInEdge-1;
								}else{
									laneToCheck=trafficPersonVec[p].numOfLaneInEdge+1;
								}

								uchar v_a,v_b;float gap_a,gap_b;
								//printf("p %u LC 1 %u\n",p,laneToCheck);
								uchar trafficLightState=trafficLights[currentEdge];
								calculateGapsLC(cellSize,maxWidth,mapToReadShift,laneMap,trafficLightState,currentEdge+laneToCheck,trafficPersonVec[p].posInLaneM,trafficPersonVec[p].length,v_a,v_b,gap_a,gap_b);
								//printf("LC 2 %u %u %f %f\n",v_a,v_b,gap_a,gap_b);
								if(gap_a==1000.0f&&gap_b==1000.0f){//lag and lead car very far
									trafficPersonVec[p].numOfLaneInEdge=laneToCheck;// CHANGE LINE

								}else{// NOT ALONE
									float b1A=0.05f,b2A=0.15f;
									float b1B=0.15f,b2B=0.40f;
									// s_0-> critical lead gap
									float g_na_D,g_bn_D;
									bool acceptLC=true;
									if(gap_a!=1000.0f){
										g_na_D=fmax(s_0,s_0+b1A*trafficPersonVec[p].v+b2A*(trafficPersonVec[p].v-v_a*3.0f));
										if(gap_a<g_na_D)//gap smaller than critical gap
											acceptLC=false;
									}
									if(acceptLC==true&&gap_b!=1000.0f){
										g_bn_D=fmax(s_0,s_0+b1B*v_b*3.0f+b2B*(v_b*3.0f-trafficPersonVec[p].v));
										if(gap_b<g_bn_D)//gap smaller than critical gap
											acceptLC=false;
									}
									if(acceptLC==true){
										trafficPersonVec[p].numOfLaneInEdge=laneToCheck;// CHANGE LINE
									}
								}
								//printf("<<LANE CHANGE\n");
							}


						}// Discretionary

						////////////////////////////////////////////////////
						// LC 3 *MANDATORY* STATE
						if(trafficPersonVec[p].LC_stateofLaneChanging==1){
							// LC 3.1 Calculate the correct lanes
							if(trafficPersonVec[p].LC_endOKLanes==0xFF){
								calculateLaneCarShouldBe(currentEdge,nextEdge,intersections,trafficPersonVec[p].edgeNextInters,trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);
								//printf("p%u num lanes %u min %u max %u\n",p,trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);
								if(trafficPersonVec[p].LC_initOKLanes==0&&trafficPersonVec[p].LC_endOKLanes==0)
									return;
							}


							//printf(">>LANE CHANGE\n");
							ushort laneToCheck;//!!!!
							//printf("LC 0 %u\n",trafficPersonVec[p].numOfLaneInEdge);
							bool leftLane=false,rightLane=false;
							// LC 3.2 CORRECT LANES--> DICRETIONARY LC WITHIN
							if(trafficPersonVec[p].numOfLaneInEdge>=trafficPersonVec[p].LC_initOKLanes&&trafficPersonVec[p].numOfLaneInEdge<trafficPersonVec[p].LC_endOKLanes){
								// for discretionary it should be under some circustances
								if((trafficPersonVec[p].v<(trafficPersonVec[p].maxSpeedMperSec*0.7f))&&(dv_dt<0)&&trafficPersonVec[p].v>3.0f){
									leftLane=
										(trafficPersonVec[p].numOfLaneInEdge>0) &&//at least one lane on the left
										(trafficPersonVec[p].numOfLaneInEdge-1>=trafficPersonVec[p].LC_initOKLanes)&&
										(trafficPersonVec[p].numOfLaneInEdge-1<trafficPersonVec[p].LC_endOKLanes);
									rightLane=
										(trafficPersonVec[p].numOfLaneInEdge<trafficPersonVec[p].edgeNumLanes-1)&&//at least one lane
										(trafficPersonVec[p].numOfLaneInEdge+1>=trafficPersonVec[p].LC_initOKLanes)&&
										(trafficPersonVec[p].numOfLaneInEdge+1<trafficPersonVec[p].LC_endOKLanes);
									//printf("D\n");
								}
							}
							// LC 3.3 INCORRECT LANES--> MANDATORY LC
							else{
								//printf("num lanes %u min %u max %u\n",trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);
								//printf("p%u num lanes %u min %u max %u\n",p,trafficPersonVec[p].edgeNumLanes,trafficPersonVec[p].LC_initOKLanes,trafficPersonVec[p].LC_endOKLanes);

								if(trafficPersonVec[p].numOfLaneInEdge<trafficPersonVec[p].LC_initOKLanes){
									rightLane=true;
								}else{
									leftLane=true;
								}
								if(rightLane==true&&trafficPersonVec[p].numOfLaneInEdge+1>=trafficPersonVec[p].edgeNumLanes){
									
								}
								if(leftLane==true&&trafficPersonVec[p].numOfLaneInEdge==0){
									
									return;
								}
								//printf("M L %d R %d nL %u\n",leftLane,rightLane,trafficPersonVec[p].numOfLaneInEdge);
							}
							if(leftLane==true||rightLane==true){

								// choose lane (if necessary)
								if(leftLane==true&&rightLane==true){
									if((int(currentTime))%2==0){
										leftLane=false;
									}else{
										rightLane=false;
									}
								}
								if(leftLane==true){
									laneToCheck=trafficPersonVec[p].numOfLaneInEdge-1;
								}else{
									laneToCheck=trafficPersonVec[p].numOfLaneInEdge+1;
								}
								if(laneToCheck>=trafficPersonVec[p].edgeNumLanes){
									
								}
								uchar v_a,v_b;float gap_a,gap_b;
								//printf("p %u LC 1 %u\n",p,laneToCheck);
								uchar trafficLightState=trafficLights[currentEdge];
								calculateGapsLC(cellSize,maxWidth,mapToReadShift,laneMap,trafficLightState,currentEdge+laneToCheck,trafficPersonVec[p].posInLaneM,trafficPersonVec[p].length,v_a,v_b,gap_a,gap_b);
								//printf("LC 2 %u %u %f %f\n",v_a,v_b,gap_a,gap_b);
								if(gap_a==1000.0f&&gap_b==1000.0f){//lag and lead car very far
									trafficPersonVec[p].numOfLaneInEdge=laneToCheck;// CHANGE LINE

								}else{// NOT ALONE
									float b1A=0.05f,b2A=0.15f;
									float b1B=0.15f,b2B=0.40f;
									float gamma=0.000025;
									// s_0-> critical lead gap
									float distEnd=trafficPersonVec[p].length-trafficPersonVec[p].posInLaneM;
									float expTerm=(1-exp(-gamma*distEnd*distEnd));

									float g_na_M,g_bn_M;
									bool acceptLC=true;
									if(gap_a!=1000.0f){
										g_na_M=fmax(s_0,s_0+(b1A*trafficPersonVec[p].v+b2A*(trafficPersonVec[p].v-v_a*3.0f)));
										if(gap_a<g_na_M)//gap smaller than critical gap
											acceptLC=false;
									}
									if(acceptLC==true&&gap_b!=1000.0f){
										g_bn_M=fmax(s_0,s_0+(b1B*v_b*3.0f+b2B*(v_b*3.0f-trafficPersonVec[p].v)));
										if(gap_b<g_bn_M)//gap smaller than critical gap
											acceptLC=false;
									}
									if(acceptLC==true){
										trafficPersonVec[p].numOfLaneInEdge=laneToCheck;// CHANGE LINE
									}
								}


							}

						}// Mandatory

					}//at least two lanes and not stopped by traffic light

				}
				///////////////////////////////////////////////////////
				
				uchar vInMpS=(uchar)(trafficPersonVec[p].v*3);//speed in m/s to fit in uchar
				ushort posInLineCells=(ushort)(trafficPersonVec[p].posInLaneM/cellSize);
				laneMap[mapToWriteShift+maxWidth*(currentEdge+trafficPersonVec[p].numOfLaneInEdge)+posInLineCells]=vInMpS;
				//printf("2<<LANE CHANGE\n");
				return;
			}
			//}
			//2.2 close to intersection

			//2.2 check if change intersection
			//!!!ALWAYS CHANGE
			//2.2.1 find next edge
			/*ushort curr_intersection=trafficPersonVec[p].edgeNextInters;
			ushort end_intersection=trafficPersonVec[p].end_intersection;
			//2.1 check if end*/
			if(nextEdge==0xFFFF){//if(curr_intersection==end_intersection){
				
				trafficPersonVec[p].active=2;//finished
				return;
			}
			//if(trafficPersonVec[p].nextPathEdge>=nextEdgeM.size())printf("AAAAAAAAAAAAAAAAA\n");
			/////////////
			// update edge
			//trafficPersonVec[p].curEdgeLane=trafficPersonVec[p].nextEdge;
			trafficPersonVec[p].currPathEdge++;
			trafficPersonVec[p].maxSpeedMperSec=trafficPersonVec[p].nextEdgemaxSpeedMperSec;
			trafficPersonVec[p].edgeNumLanes=trafficPersonVec[p].nextEdgeNumLanes;
			trafficPersonVec[p].edgeNextInters=trafficPersonVec[p].nextEdgeNextInters;
			trafficPersonVec[p].length=trafficPersonVec[p].nextEdgeLength;
			trafficPersonVec[p].posInLaneM=numMToMove;
			if(trafficPersonVec[p].numOfLaneInEdge>=trafficPersonVec[p].edgeNumLanes){
				trafficPersonVec[p].numOfLaneInEdge=trafficPersonVec[p].edgeNumLanes-1;//change line if there are less roads
			}

			////////////
			// update next edge
			ushort nextNEdge=trafficPersonVec[p].personPath[trafficPersonVec[p].currPathEdge+1];
			//trafficPersonVec[p].nextEdge=nextEdge;
			if(nextNEdge!=0xFFFF){
				//trafficPersonVec[p].nextPathEdge++;
				trafficPersonVec[p].LC_initOKLanes=0xFF;
				trafficPersonVec[p].LC_endOKLanes=0xFF;
				
				//2.2.3 update person edgeData
				//trafficPersonVec[p].nextEdge=nextEdge;
				trafficPersonVec[p].nextEdgemaxSpeedMperSec=edgesData[nextNEdge].maxSpeedMperSec;
				trafficPersonVec[p].nextEdgeNumLanes=edgesData[nextNEdge].numLines;
				trafficPersonVec[p].nextEdgeNextInters=edgesData[nextNEdge].nextInters;
				trafficPersonVec[p].nextEdgeLength=edgesData[nextNEdge].length;
			}

			trafficPersonVec[p].LC_stateofLaneChanging=0;
			uchar vInMpS=(uchar)(trafficPersonVec[p].v*3);//speed in m/s to fit in uchar
			ushort posInLineCells=(ushort)(trafficPersonVec[p].posInLaneM/cellSize);
			laneMap[mapToWriteShift+maxWidth*(nextEdge+trafficPersonVec[p].numOfLaneInEdge)+posInLineCells]=vInMpS;

		 ///
	 }
	 
}//

__global__ void kernel_intersectionSimulation(uint numIntersections,float currentTime,LC::intersectionData *intersections,uchar *trafficLights) {
	 int i = blockIdx.x * blockDim.x + threadIdx.x;
	 if(i<numIntersections){//CUDA check (inside margins)

		 //////////////////////////////////////////////////////
		 const float deltaEvent=20.0f;
		 //if(i==0)printf("i %d\n",i);
		 if(currentTime>intersections[i].nextEvent&&intersections[i].totalInOutEdges>0){


			 uint edgeOT=intersections[i].edge[intersections[i].state];
			 uchar numLinesO=edgeOT>>24;
			 ushort edgeONum=edgeOT&0xFFFF;
			 // red old traffic lights
			 for(int nL=0;nL<numLinesO;nL++){
				 trafficLights[edgeONum+nL]=0x00;//red old traffic light
			 }
			 for(int iN=0;iN<=intersections[i].totalInOutEdges;iN++){//to give a round
				 intersections[i].state=(intersections[i].state+1)%intersections[i].totalInOutEdges;//next light
				 if((intersections[i].edge[intersections[i].state]&0x010000)==0x010000){
					 // green new traffic lights
					 uint edgeIT=intersections[i].edge[intersections[i].state];
					 ushort edgeINum=edgeIT&0xFFFF;//get edgeI
					 uchar numLinesI=edgeIT>>24;
					 for(int nL=0;nL<numLinesI;nL++){
						 trafficLights[edgeINum+nL]=0xFF;
					 }
					 break;
				 }
			 }//green new traffic light
			 //printf("i %d CHANGE state %u of %d (Old edge %u New Edge %u)\n",i,intersections[i].state,intersections[i].totalInOutEdges,edgeO,edgeI);
			 ////
			 intersections[i].nextEvent=currentTime+deltaEvent;
		 }
	 

		 //////////////////////////////////////////////////////
	 }
	 
 }//


void simulateTrafficCUDA(float currentTime,uint numPeople,uint numIntersections){

	////////////////////////////////////////////////////////////
	// 1. CHANGE MAP: set map to use and clean the other
	if(readFirstMapC==true){
		mapToReadShift=0;
		mapToWriteShift=halfLaneMap;
		hipMemset(&laneMap_d[halfLaneMap],-1,halfLaneMap*sizeof (unsigned char));//clean second half
	}else{
		mapToReadShift=halfLaneMap;
		mapToWriteShift=0;
		hipMemset(&laneMap_d[0],-1,halfLaneMap*sizeof (unsigned char));//clean first half
	}
	readFirstMapC=!readFirstMapC;//next iteration invert use

	//float currentTime=3.0f;
	//printf("!! >>Call kernel\n");
	//while(currentTime<24.0f){

	kernel_intersectionSimulation <<< numIntersections,1 >>> (numIntersections, currentTime,intersections_d,trafficLights_d);
	//int blocks=1;
	//if(numPeople>10000){
		//blocks=1024;
		//numPeople=ceil(numPeople/1024.0f);
	//}
	kernel_trafficSimulation <<< ceil(numPeople/1024.0f),1024 >>> (numPeople,currentTime,cellSize,deltaTime,trafficPersonVec_d,edgesData_d,laneMap_d,intersections_d,trafficLights_d, mapToReadShift,mapToWriteShift,maxWidth);// n_blocks, block_size >>> (a_d, N);
		//currentTime+=deltaTimeC;
	//}
	//printf("!! <<End kernel\n");

	

	//exampleCUDA();
	//const float deltaTime=1.0f*(1.0f/3600.0f);//1*(1hours/3600 seconds)
	//float currentTime=3.0f;//lets start at 300 AM
	// 0. put data as references to simplified code

	/*
	int numVertex=cudaTrafficRoutes.numVertex;
	int numPeople=trafficPersonVec.size();

	printf(">>Start Simulation\n");
	QTime timer;
	timer.start();
	while(currentTime<24.0f){
		bool anyActive=false;
		//printf("time %f\n",currentTime);
		for(int p=0;p<numPeople;p++){
			//printf(" 0. Person: %d\n",p);
			//0. check if finished
			

		}
		currentTime+=deltaTime;
		if(anyActive){
			clientMain->mGLWidget_3D->updateGL();
			Sleep(1);
			QApplication::processEvents();
		}
	}*/
	

	//printf("<<simulate\n");

}//
